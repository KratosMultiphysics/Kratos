/*
==============================================================================
KratosGPUApplication 
A library based on:
Kratos
A General Purpose Software for Multi-Physics Finite Element Analysis
Version 1.0 (Released on march 05, 2007).

Copyright 2009
Pooyan Dadvand, Riccardo Rossi, Isaac Gallego, Farshid Mossaiby 
pooyan@cimne.upc.edu 
rrossi@cimne.upc.edu
isaac.gallego.pla@gmail.com
mossaiby@yahoo.com
- CIMNE (International Center for Numerical Methods in Engineering),
Gran Capita' s/n, 08034 Barcelona, Spain

Permission is hereby granted, free  of charge, to any person obtaining
a  copy  of this  software  and  associated  documentation files  (the
"Software"), to  deal in  the Software without  restriction, including
without limitation  the rights to  use, copy, modify,  merge, publish,
distribute,  sublicense and/or  sell copies  of the  Software,  and to
permit persons to whom the Software  is furnished to do so, subject to
the following condition:

Distribution of this code for  any  commercial purpose  is permissible
ONLY BY DIRECT ARRANGEMENT WITH THE COPYRIGHT OWNERS.

The  above  copyright  notice  and  this permission  notice  shall  be
included in all copies or substantial portions of the Software.

THE  SOFTWARE IS  PROVIDED  "AS  IS", WITHOUT  WARRANTY  OF ANY  KIND,
EXPRESS OR  IMPLIED, INCLUDING  BUT NOT LIMITED  TO THE  WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT  SHALL THE AUTHORS OR COPYRIGHT HOLDERS  BE LIABLE FOR ANY
CLAIM, DAMAGES OR  OTHER LIABILITY, WHETHER IN AN  ACTION OF CONTRACT,
TORT  OR OTHERWISE, ARISING  FROM, OUT  OF OR  IN CONNECTION  WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

==============================================================================
 */

#include "AMGpreconditioner.h"
#include <cstdio>

AMGpreconditioner::AMGpreconditioner(double _W, size_t _numLevelsRoh, bool _assumeZerosForEachStep, size_t _numMaxHierarchyLevels, size_t _minimumSizeAllowed, size_t* _preSweeps, size_t* _postSweeps, bool actAsPreconditioner) {
        W = _W;
        numLevelsRoh = _numLevelsRoh;
	assumeZerosForEachStep = _assumeZerosForEachStep;
	numMaxHierarchyLevels = _numMaxHierarchyLevels;
	minimumSizeAllowed = _minimumSizeAllowed;

	preSweeps = _preSweeps;
	postSweeps = _postSweeps;
	isPreconditioner = actAsPreconditioner;
	numFinalLevels = 0;
        //printf("Minimum size allowed set in constructor: %lu\n", minimumSizeAllowed);
}

AMGpreconditioner::AMGpreconditioner(double _W, size_t _numLevelsRoh, bool _assumeZerosForEachStep, size_t _numMaxHierarchyLevels, size_t _minimumSizeAllowed, bool actAsPreconditioner){
	W = _W;
        numLevelsRoh = _numLevelsRoh;
	assumeZerosForEachStep = _assumeZerosForEachStep;
	numMaxHierarchyLevels = _numMaxHierarchyLevels;
	minimumSizeAllowed = _minimumSizeAllowed;
	isPreconditioner = actAsPreconditioner;
	numFinalLevels = 0;
}

AMGpreconditioner::AMGpreconditioner(){
	numFinalLevels = 0;
}

AMGpreconditioner::~AMGpreconditioner() {

}

void AMGpreconditioner::cleanPreconditioner(){
	if(numFinalLevels > 0){
		//P
		delete[] P[0].indices_cpu;
		delete[] P[0].values_cpu;
		delete[] P[0].ptr_cpu;
		deletingStuff(P[0].indices_gpu);
		deletingStuff(P[0].values_gpu);
		deletingStuff(P[0].ptr_gpu);
		//R
		delete[] R[0].indices_cpu;
		delete[] R[0].values_cpu;
		delete[] R[0].ptr_cpu;
		deletingStuff(R[0].indices_gpu);
		deletingStuff(R[0].values_gpu);
		deletingStuff(R[0].ptr_gpu);
		//G
		delete[] G[0].indices_cpu;
		delete[] G[0].values_cpu;
		delete[] G[0].ptr_cpu;
		deletingStuff(G[0].indices_gpu);
		deletingStuff(G[0].values_gpu);
		deletingStuff(G[0].ptr_gpu);
		for(size_t i = 1; i < numFinalLevels; i++){
		    //P
		    delete[] P[i].indices_cpu;
		    delete[] P[i].values_cpu;
		    delete[] P[i].ptr_cpu;
		    deletingStuff(P[i].indices_gpu);
		    deletingStuff(P[i].values_gpu);
		    deletingStuff(P[i].ptr_gpu);
		    //R
		    delete[] R[i].indices_cpu;
		    delete[] R[i].values_cpu;
		    delete[] R[i].ptr_cpu;
		    deletingStuff(R[i].indices_gpu);
		    deletingStuff(R[i].values_gpu);
		    deletingStuff(R[i].ptr_gpu);
		    //G
		    delete[] G[i].indices_cpu;
		    delete[] G[i].values_cpu;
		    delete[] G[i].ptr_cpu;
		    deletingStuff(G[i].indices_gpu);
		    deletingStuff(G[i].values_gpu);
		    deletingStuff(G[i].ptr_gpu);
		    //A
		    delete[] Matrices[i].indices_cpu;
		    delete[] Matrices[i].values_cpu;
		    delete[] Matrices[i].ptr_cpu;
		    deletingStuff(Matrices[i].indices_gpu);
		    deletingStuff(Matrices[i].values_gpu);
		    deletingStuff(Matrices[i].ptr_gpu);
		}
    	}
	delete[] Matrices[numFinalLevels].matAuxValues;
	delete[] P;
	delete[] R;
	delete[] G;
	delete[] Matrices;
}

void AMGpreconditioner::initialize(size_t* ptr_cpu, size_t* indices_cpu, double* values_cpu,
        size_t* ptr_gpu, size_t* indices_gpu, double* values_gpu,
        size_t numRows, size_t numCols, size_t numNNZ, bool dataIsChanged, bool structureIsChanged){

	/*printf("PRINTING from AMGpreconditioner initialize, variable values:\n W = %f, Roh = %u, Zeros = %s, HierarchyLevels = %u, minimumSize = %u, firstPre = %u, secondPre = %u\n", W, numLevelsRoh, (assumeZerosForEachStep)?"true":"false", numMaxHierarchyLevels, minimumSizeAllowed, preSweeps[0], preSweeps[1]);*/

	//printf("El valor de minimumSizeAllowed es: %u\n", minimumSizeAllowed);

    Matrices = new _Matrix[numMaxHierarchyLevels];
    P = new _Matrix[numMaxHierarchyLevels];
    R = new _Matrix[numMaxHierarchyLevels];
    G = new _Matrix[numMaxHierarchyLevels];
    b.numElems = numCols;

    Matrices[0].numRows = numRows;
    Matrices[0].numCols = numCols;
    Matrices[0].numNNZ = numNNZ;
    Matrices[0].indices_cpu = indices_cpu;
    Matrices[0].ptr_cpu = ptr_cpu;
    Matrices[0].values_cpu = values_cpu;

    Matrices[0].indices_gpu = indices_gpu;
    Matrices[0].ptr_gpu = ptr_gpu;
    Matrices[0].values_gpu = values_gpu;

        /** Generating hierarchy **/
    numFinalLevels = generateHierarchy(Matrices, P, R, G, W, numLevelsRoh, numMaxHierarchyLevels, minimumSizeAllowed);
	printf("Initialize finalized, with numFinalLevels %u\n", numFinalLevels);
}

void AMGpreconditioner::singleStep(double* b_gpu, double* x_gpu){

	_Vector u;
	u.numElems = Matrices[0].numCols;
	u.values_cpu = new double[u.numElems];
	if(isPreconditioner){
		GPU_fillWithZeros(u.numElems, x_gpu);
	}

	u.values_gpu = x_gpu;
	b.values_gpu = b_gpu;
	b.values_cpu = new double[u.numElems];
	multilevel(Matrices, P, R, G, b, u, 0, numFinalLevels, preSweeps, postSweeps, assumeZerosForEachStep);
	delete[] u.values_cpu;
	delete[] b.values_cpu;
}

size_t AMGpreconditioner::solve(double* b_gpu, double* b_cpu, double* x_gpu, double* x_cpu, double _precision, size_t maxIters){
    threshold = _precision;
    _Vector u;
    double residual;
    u.numElems = Matrices[0].numCols;
    u.values_cpu = x_cpu;
    u.values_gpu = x_gpu;
    b.values_cpu = b_cpu;
    b.values_gpu = b_gpu;

    residual = checkResidual(u, b, Matrices[0]);

    bool done = false;
    size_t iterations = 0;

    while(!done && iterations < maxIters){
        iterations++;
        //multilevel(Matrices, P, R, G, b, u, 0, numFinalLevels);
        singleStep(b_gpu, x_gpu);        
        done = checkConvergence(u, b, Matrices[0], residual, threshold);
    }
    copyMem(u.values_gpu, u.values_cpu, u.numElems, 1);
    if(iterations == maxIters)
        printf("It haven't converged!!\n");
    else
        printf("Convergence achieved int %u iterations\n", iterations);

    return iterations;
}

void AMGpreconditioner::setPreSweeps( size_t* _preSweeps) {
	preSweeps = _preSweeps;
}

void AMGpreconditioner::setPostSweeps(size_t* _postSweeps) {
	postSweeps = _postSweeps;

}

