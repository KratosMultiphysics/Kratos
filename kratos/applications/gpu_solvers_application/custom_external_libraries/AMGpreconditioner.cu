/*
==============================================================================
KratosGPUApplication 
A library based on:
Kratos
A General Purpose Software for Multi-Physics Finite Element Analysis
Version 1.0 (Released on march 05, 2007).

Copyright 2009
Pooyan Dadvand, Riccardo Rossi, Isaac Gallego, Farshid Mossaiby 
pooyan@cimne.upc.edu 
rrossi@cimne.upc.edu
isaac.gallego.pla@gmail.com
mossaiby@yahoo.com
- CIMNE (International Center for Numerical Methods in Engineering),
Gran Capita' s/n, 08034 Barcelona, Spain

Permission is hereby granted, free  of charge, to any person obtaining
a  copy  of this  software  and  associated  documentation files  (the
"Software"), to  deal in  the Software without  restriction, including
without limitation  the rights to  use, copy, modify,  merge, publish,
distribute,  sublicense and/or  sell copies  of the  Software,  and to
permit persons to whom the Software  is furnished to do so, subject to
the following condition:

Distribution of this code for  any  commercial purpose  is permissible
ONLY BY DIRECT ARRANGEMENT WITH THE COPYRIGHT OWNERS.

The  above  copyright  notice  and  this permission  notice  shall  be
included in all copies or substantial portions of the Software.

THE  SOFTWARE IS  PROVIDED  "AS  IS", WITHOUT  WARRANTY  OF ANY  KIND,
EXPRESS OR  IMPLIED, INCLUDING  BUT NOT LIMITED  TO THE  WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT  SHALL THE AUTHORS OR COPYRIGHT HOLDERS  BE LIABLE FOR ANY
CLAIM, DAMAGES OR  OTHER LIABILITY, WHETHER IN AN  ACTION OF CONTRACT,
TORT  OR OTHERWISE, ARISING  FROM, OUT  OF OR  IN CONNECTION  WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

==============================================================================
 */

#include "AMGpreconditioner.h"
#include <stdio.h>

AMGpreconditioner::AMGpreconditioner(double _W, size_t _numLevelsRoh, bool _assumeZerosForEachStep, size_t _numMaxHierarchyLevels, size_t _minimumSizeAllowed, size_t* _preSweeps, size_t* _postSweeps, bool actAsPreconditioner) {
        W = _W;
        numLevelsRoh = _numLevelsRoh;
	assumeZerosForEachStep = _assumeZerosForEachStep;
	numMaxHierarchyLevels = _numMaxHierarchyLevels;
	minimumSizeAllowed = _minimumSizeAllowed;

	preSweeps = _preSweeps;
	postSweeps = _postSweeps;
	isPreconditioner = actAsPreconditioner;
	numFinalLevels = 0;
        //printf("Minimum size allowed set in constructor: %lu\n", minimumSizeAllowed);
}

AMGpreconditioner::AMGpreconditioner(double _W, size_t _numLevelsRoh, bool _assumeZerosForEachStep, size_t _numMaxHierarchyLevels, size_t _minimumSizeAllowed, bool actAsPreconditioner){
	W = _W;
        numLevelsRoh = _numLevelsRoh;
	assumeZerosForEachStep = _assumeZerosForEachStep;
	numMaxHierarchyLevels = _numMaxHierarchyLevels;
	minimumSizeAllowed = _minimumSizeAllowed;
	isPreconditioner = actAsPreconditioner;
	numFinalLevels = 0;
}

AMGpreconditioner::AMGpreconditioner(){
	numFinalLevels = 0;
}

AMGpreconditioner::~AMGpreconditioner() {

}

void AMGpreconditioner::cleanPreconditioner(){
	if(numFinalLevels > 0){
		for(size_t i = 0; i < numFinalLevels; i++){
		    //P
		    delete P[i];
		    //R
		    delete R[i];
		    //G
		    delete G[i];
		    //A
		    delete Matrices[i];
		}
    	}
	delete Matrices[numFinalLevels];
	delete[] P;
	delete[] R;
	delete[] G;
	delete[] Matrices;
}

void AMGpreconditioner::initialize(size_t* ptr_cpu, size_t* indices_cpu, double* values_cpu,
        size_t* ptr_gpu, size_t* indices_gpu, double* values_gpu,
        size_t numRows, size_t numCols, size_t numNNZ, bool dataIsChanged, bool structureIsChanged){

	/*printf("PRINTING from AMGpreconditioner initialize, variable values:\n W = %f, Roh = %u, Zeros = %s, HierarchyLevels = %u, minimumSize = %u, firstPre = %u, secondPre = %u\n", W, numLevelsRoh, (assumeZerosForEachStep)?"true":"false", numMaxHierarchyLevels, minimumSizeAllowed, preSweeps[0], preSweeps[1]);*/

	//printf("El valor de minimumSizeAllowed es: %u\n", minimumSizeAllowed);

    Matrices = new GPUCSRMatrix*[numMaxHierarchyLevels];
    P = new GPUCSRMatrix*[numMaxHierarchyLevels];
    R = new GPUCSRMatrix*[numMaxHierarchyLevels];
    G = new GPUCSRMatrix*[numMaxHierarchyLevels];
    b = new GPUVector(numCols);

    Matrices[0] = new GPUCSRMatrix(numNNZ, numRows, numCols, indices_cpu, ptr_cpu, values_cpu, false);
    Matrices[0]->GPU_Allocate();
    Matrices[0]->Copy(CPU_GPU, false);


        /** Generating hierarchy **/
    numFinalLevels = generateHierarchy(Matrices, P, R, G, W, numLevelsRoh, numMaxHierarchyLevels, minimumSizeAllowed);
	printf("Initialize finalized, with numFinalLevels %u\n", numFinalLevels);
}

void AMGpreconditioner::singleStep(double* b_gpu, double* x_gpu){

	GPUVector u(Matrices[0]->Size2);
	u.CPU_Values = new double[u.Size];
	if(isPreconditioner){
		GPU_fillWithZeros(u.Size, x_gpu);
	}

	u.GPU_Values = x_gpu;
	u.Allocated = true;
	b->GPU_Values = b_gpu;
	b->CPU_Values = new double[u.Size];
	b->Allocated = true;
	multilevel(Matrices, P, R, G, *b, u, 0, numFinalLevels, preSweeps, postSweeps, assumeZerosForEachStep);
	u.Allocated = false;
	b->Allocated = false;
	delete[] u.CPU_Values;
	delete[] b->CPU_Values;
}

size_t AMGpreconditioner::solve(double* b_gpu, double* b_cpu, double* x_gpu, double* x_cpu, double _precision, size_t maxIters){
    threshold = _precision;
    GPUVector u(Matrices[0]->Size2, x_cpu);
    double residual;
    u.GPU_Values = x_gpu;
    b->CPU_Values = b_cpu;
    b->GPU_Values = b_gpu;

    residual = checkResidual(u, *b, *Matrices[0]);

    bool done = false;
    size_t iterations = 0;

    while(!done && iterations < maxIters){
        iterations++;
        //multilevel(Matrices, P, R, G, b, u, 0, numFinalLevels);
        singleStep(b_gpu, x_gpu);        
        done = checkConvergence(u, *b, *Matrices[0], residual, threshold);
    }
    copyMem(u.GPU_Values, u.CPU_Values, u.Size, 1);
    if(iterations == maxIters)
        printf("It haven't converged!!\n");
    else
        printf("Convergence achieved int %u iterations\n", iterations);

    return iterations;
}

void AMGpreconditioner::setPreSweeps( size_t* _preSweeps) {
	preSweeps = _preSweeps;
}

void AMGpreconditioner::setPostSweeps(size_t* _postSweeps) {
	postSweeps = _postSweeps;

}

