#include "hip/hip_runtime.h"
/*
==============================================================================
Kratos
A General Purpose Software for Multi-Physics Finite Element Analysis
Version 1.0 (Released on march 05, 2007).

Copyright 2007
Pooyan Dadvand, Riccardo Rossi, Farshid Mossaiby
pooyan@cimne.upc.edu
rrossi@cimne.upc.edu
mossaiby@yahoo.com
CIMNE (International Center for Numerical Methods in Engineering),
Gran Capita' s/n, 08034 Barcelona, Spain

Permission is hereby granted, free  of charge, to any person obtaining
a  copy  of this  software  and  associated  documentation files  (the
"Software"), to  deal in  the Software without  restriction, including
without limitation  the rights to  use, copy, modify,  merge, publish,
distribute,  sublicense and/or  sell copies  of the  Software,  and to
permit persons to whom the Software  is furnished to do so, subject to
the following condition:

Distribution of this code for  any  commercial purpose  is permissible
ONLY BY DIRECT ARRANGEMENT WITH THE COPYRIGHT OWNER.

The  above  copyright  notice  and  this permission  notice  shall  be
included in all copies or substantial portions of the Software.

THE  SOFTWARE IS  PROVIDED  "AS  IS", WITHOUT  WARRANTY  OF ANY  KIND,
EXPRESS OR  IMPLIED, INCLUDING  BUT NOT LIMITED  TO THE  WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT  SHALL THE AUTHORS OR COPYRIGHT HOLDERS  BE LIABLE FOR ANY
CLAIM, DAMAGES OR  OTHER LIABILITY, WHETHER IN AN  ACTION OF CONTRACT,
TORT  OR OTHERWISE, ARISING  FROM, OUT  OF OR  IN CONNECTION  WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

==============================================================================
*/

//
// Sparse matrix and vector operations on GPU

//
// Notes!
//
//   * In case of VectorVectorMultiply and VectorNorm2, cuBlas library has been used, as it is much faster than a code without proper optimizations.
//
//   * In case of VectorScaleAndAdd, as there is no direct way of using cuBlas for this, we had to write our own kernel. It was 2 or 3 times faster
//     than using cuBlas indirectly.
//
//   * For compilation command, simply issue a 'make' command.
//
//   * Removed templates to be able to link to Kratos.
//
//   * Modified GPUCSRMatrix to keep no. of non-zeros per row constant and equal to HALF_WARP_SIZE (16) and used same no. of threads to multiply a row

// More notes!
//
//	* Checks for error in Bind_X() and Unbind_X() have been removed for more consistent error checking after kernel calls; can they fail?
//
//	* Added an optional parameter to GPUCSRMatrix constructor, so that user can optionally avoid making non-zeros in a row a multiple of HALF_WARP_SIZE (16)
//
//	* Added a make file; in the command line use emu=1 for emulation mode and dbg=1 for a debug version

// Includes, system

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

// Includes, project

#include "gpu_sparse.h"
#include "gpu_sparse_utils.h"
#include "gpu_sparse_kernels.h"

//Includes, preconditioner
#include <lapackd.h>
#include <laslv.h>
#include <gmd.h>
#include <gmf.h>


using namespace std;

namespace Kratos
{

namespace GPUSparse
{

// GPUVector class definition

GPUVector::GPUVector(size_t _Size, double *_CPU_Values): Size(_Size), CPU_Values(_CPU_Values), GPU_Values(0), Allocated(false)
{
	// Nothing to do!
}

GPUVector::GPUVector(size_t _Size): Size(_Size), CPU_Values(0), GPU_Values(0), Allocated(false)
{
	// Nothing to do!
}

GPUVector::~GPUVector()
{
	if (Allocated)
		GPU_Free();
}

bool GPUVector::GPU_Allocate()
{
	if (Allocated)
		return false;

	Allocated = true;

	return CUDA_Success(hipMalloc(reinterpret_cast <void **> (&GPU_Values), Size * sizeof(double)));
}

bool GPUVector::GPU_Free()
{
	if (!Allocated)
		return false;

	Allocated = false;

	return CUDA_Success(hipFree(GPU_Values));
}

bool GPUVector::Copy(CopyDirection Direction)
{
	if (!Allocated)
		return false;

	switch (Direction)
	{
		case CPU_GPU:

			return CUDA_Success(hipMemcpy(GPU_Values, CPU_Values, Size * sizeof(double), hipMemcpyHostToDevice));

		case GPU_CPU:

			return CUDA_Success(hipMemcpy(CPU_Values, GPU_Values, Size * sizeof(double), hipMemcpyDeviceToHost));
	}

	// We should never get here!
	return false;
}

bool GPUVector::CopyFromGPU(GPUVector &V)
{
	if (V.Size != Size || !Allocated || !V.Allocated)
		return false;

	return CUDA_Success(hipMemcpy(GPU_Values, V.GPU_Values, Size * sizeof(double), hipMemcpyDeviceToDevice));
}

// GPUCSRMatrix class definition

GPUCSRMatrix::GPUCSRMatrix(size_t _NNZ, size_t _Size1, size_t _Size2, size_t *_CPU_Columns, size_t *_CPU_RowIndices, double *_CPU_Values, bool _NZMultiple16): NNZ(_NNZ), Size1(_Size1), Size2(_Size2), CPU_Columns(0), CPU_RowIndices(0), CPU_Values(0), GPU_Columns(0), GPU_RowIndices(0), GPU_Values(0), Allocated(false)

{
	haveDenseRepresentation = false;

	if (_NZMultiple16)
	{

		NNZ = 0;
		// Temporary RowIndices vector
		size_t *Temp_CPU_RowIndices = new size_t[Size1 + 1];

		Temp_CPU_RowIndices[0] = 0;

		// Find out how many non-zeros are needed to pad all rows to 16 while building the RowIndices
		for (size_t i = 0; i < Size1; i++)
		{
			size_t NZ = _CPU_RowIndices[i + 1] - _CPU_RowIndices[i];

			size_t R = NZ & HALF_WARP_SIZE_MASK;

			if (R != 0)
				NZ += HALF_WARP_SIZE - R;

			NNZ += NZ;
			Temp_CPU_RowIndices[i + 1] = Temp_CPU_RowIndices[i] + NZ;
		}

		// Allocate CPU memory for CSR structure using only one chunk of page-locked memory to speed up data transfer between CPU and GPU
		void *CSR_Data;


		if (!CUDA_Success(hipHostMalloc(&CSR_Data, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t))))  // TODO: What should be done?!
			CSR_Data = 0;

		// We are sure that using this order, the memory alignment conditions will be satisfied as NNZ is a multiple of HALF_WARP_SIZE (16) and sizeof(double) = 8
		// TODO: Check this!
/*		CPU_Values = reinterpret_cast <double *> (CSR_Data);  // Size: NNZ * sizeof(double)
		CPU_Columns = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * sizeof(double));  // Size: NNZ * sizeof(size_t)
		CPU_RowIndices = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * (sizeof(double) + sizeof(size_t)));  // Size: (Size1 + 1) * sizeof(size_t)*/
CPU_Values = (double*)CSR_Data;
CPU_Columns = (size_t*)((double*) CSR_Data + NNZ);
CPU_RowIndices = (size_t*)((size_t*)((double*) CSR_Data + NNZ) + NNZ);
		// Move temporary data

		memcpy(CPU_RowIndices, Temp_CPU_RowIndices, (Size1 + 1) * sizeof(size_t));


		delete[] Temp_CPU_RowIndices;

		// Build ECSR structure from given CSR
		for (size_t i = 0; i < Size1; i++)
		{
			size_t _Start = _CPU_RowIndices[i], Start = CPU_RowIndices[i];

			for (size_t j = 0; j < _CPU_RowIndices[i + 1] - _CPU_RowIndices[i]; j++)
			{
				CPU_Columns[Start + j] = _CPU_Columns[_Start + j];
				CPU_Values[Start + j] = _CPU_Values[_Start + j];
			}

			size_t LastCol = _CPU_Columns[_CPU_RowIndices[i + 1] - 1];

			for (size_t j = _CPU_RowIndices[i + 1] - _CPU_RowIndices[i]; j < CPU_RowIndices[i + 1] - CPU_RowIndices[i]; j++)
			{
				CPU_Columns[Start + j] = LastCol;  // To maintain coalescing as much as possible
				CPU_Values[Start + j] = 0.00;
			}
		}
	}

	else
	{
		// Allocate CPU memory for CSR structure using only one chunk of page-locked memory to speed up data transfer between CPU and GPU
		void *CSR_Data;

		if (!CUDA_Success(hipHostMalloc(&CSR_Data, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t))))  // TODO: What should be done?!
			CSR_Data = 0;

		// We are sure that using this order, the memory alignment conditions will be satisfied as NNZ is a multiple of HALF_WARP_SIZE (16) and sizeof(double) = 8
		// TODO: Check this!
/*		CPU_Values = reinterpret_cast <double *> (CSR_Data);  // Size: NNZ * sizeof(double)
		CPU_Columns = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * sizeof(double));  // Size: NNZ * sizeof(size_t)
		CPU_RowIndices = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * (sizeof(double) + sizeof(size_t)));  // Size: (Size1 + 1) * sizeof(size_t)*/
CPU_Values = (double*)CSR_Data;
CPU_Columns = (size_t*)((double*) CSR_Data + NNZ);
CPU_RowIndices = (size_t*)((size_t*)((double*) CSR_Data + NNZ) + NNZ);

		// Move temporary data
		memcpy(CPU_Values, _CPU_Values, NNZ * sizeof(double));
		memcpy(CPU_Columns, _CPU_Columns, NNZ * sizeof(size_t));
		memcpy(CPU_RowIndices, _CPU_RowIndices, (Size1 + 1) * sizeof(size_t));
	}
}

GPUCSRMatrix::GPUCSRMatrix(size_t _NNZ, size_t _Size1, size_t _Size2): NNZ(_NNZ), Size1(_Size1), Size2(_Size2), CPU_Columns(0), CPU_RowIndices(0), CPU_Values(0), GPU_Columns(0), GPU_RowIndices(0), GPU_Values(0), Allocated(false), haveDenseRepresentation(false)
{
	// Nothing to do!
}

GPUCSRMatrix::~GPUCSRMatrix()
{
	// Free CSR data; as it is allocated in one chunk of memory, we need only to free the begining address
	hipHostFree(CPU_Values);

	if (Allocated)
		GPU_Free();
	if (haveDenseRepresentation)
		delete[] matAuxValues;
}

bool GPUCSRMatrix::GPU_Allocate()
{
	if (Allocated)
		return false;

	Allocated = true;

	// Allocate GPU memory for CSR structure using only one chunk of memory to speed up data transfer between CPU and GPU

	void *CSR_Data;


	if (CUDA_Success(hipMalloc(&CSR_Data, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t))))
	{
/*		GPU_Values = reinterpret_cast <double *> (CSR_Data);  // Size: NNZ * sizeof(double)
		GPU_Columns = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * sizeof(double));  // Size: NNZ * sizeof(size_t)
		GPU_RowIndices = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * (sizeof(double) + sizeof(size_t)));  // Size: (Size1 + 1) * sizeof(size_t)*/
GPU_Values = (double*)CSR_Data;
		GPU_Columns = (size_t*)((double*) CSR_Data + NNZ);
		GPU_RowIndices = (size_t*)((size_t*)((double*) CSR_Data + NNZ) + NNZ);

		return true;
	}

	else
		return false;
}

bool GPUCSRMatrix::GPU_Free()
{
	if (!Allocated)
		return false;

	Allocated = false;
	
	return CUDA_Success(hipFree(GPU_Values));
}

bool GPUCSRMatrix::Copy(CopyDirection Direction, bool CopyValuesOnly)
{
	if (!Allocated)
		return false;

	switch (Direction)
	{
		case CPU_GPU:
			if (CopyValuesOnly)
				return CUDA_Success(hipMemcpy(GPU_Values, CPU_Values, NNZ * sizeof(double), hipMemcpyHostToDevice));
			else
				return CUDA_Success(hipMemcpy(GPU_Values, CPU_Values, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t), hipMemcpyHostToDevice));

		case GPU_CPU: 
			if (CopyValuesOnly)
				return CUDA_Success(hipMemcpy(CPU_Values, GPU_Values, NNZ * sizeof(double), hipMemcpyDeviceToHost));
			else
				return CUDA_Success(hipMemcpy(CPU_Values, GPU_Values, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t), hipMemcpyDeviceToHost));

	}

	// We should never get here!
	return false;
}

bool GPUCSRMatrix::CopyFromGPU(GPUCSRMatrix &M, bool CopyStructure, bool CopyValues)
{
	if (M.Size1 != Size1 || M.Size2 != Size2 || M.NNZ != NNZ || !Allocated || !M.Allocated)
		return false;

	size_t CopyLength;
	void *CopyFrom, *CopyTo;

	if (CopyStructure && CopyValues)
	{
		CopyTo = GPU_Values;
		CopyFrom = M.GPU_Values;
		CopyLength = NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t);
	}
	else if (CopyStructure && !CopyValues)
	{
		CopyTo = reinterpret_cast <void *> (reinterpret_cast <size_t> (GPU_Values) + NNZ * sizeof(double));
		CopyFrom = reinterpret_cast <void *> (reinterpret_cast <size_t> (M.GPU_Values) + NNZ * sizeof(double));
		CopyLength = (NNZ + Size1 + 1) * sizeof(size_t);
	}
	else if (!CopyStructure && CopyValues)
	{
		CopyTo = GPU_Values;
		CopyFrom = M.GPU_Values;
		CopyLength = NNZ * sizeof(double);
	}
	else if (!CopyStructure && !CopyValues)
	{
		CopyTo = 0;
		CopyFrom = 0;
		CopyLength = 0;
	}

	if (CopyLength != 0)
		return CUDA_Success(hipMemcpy(CopyTo, CopyFrom, CopyLength, hipMemcpyDeviceToDevice));
	else
		return true;
}

bool GPUCSRMatrix::GenerateDenseRepresentation(bool FortranRep){
	haveDenseRepresentation = true;
	numValuesDenseRep = Size2 * Size1;
	matAuxValues = new double[Size2 * Size1];
	size_t pointer = 0;
	size_t currentIndice = 0;
	for(size_t i = 0; i < Size1; i++){
		size_t nonZeros = CPU_RowIndices[i+1] - CPU_RowIndices[i];
		
		for(size_t j = 0; j < Size2; j++){

		    if(nonZeros > 0  && CPU_Columns[currentIndice] == j){
			matAuxValues[pointer] = (double)CPU_Values[currentIndice];
			currentIndice++;
			nonZeros--;
		    }else{
			matAuxValues[pointer] = 0.0;
		    }
		    pointer++;
		}
		
	}
	return true;
}

// Operations defined on GPUCSRMatrix and GPUVector

//
// CPUGPUCSRMatrixVectorMultiply
// Matrix-Vector multiply on CPU

bool CPUGPUCSRMatrixVectorMultiply(GPUCSRMatrix &A, GPUVector &X, GPUVector &Y)
{
	// Primary checks
	if (A.Size2 != X.Size || X.Size != Y.Size)
		return false;

	for (size_t i = 0; i < A.Size1; i++)
	{
		double YI = static_cast <double> (0);

		for (size_t j = A.CPU_RowIndices[i]; j < A.CPU_RowIndices[i + 1]; j++)
			YI += A.CPU_Values[j] * X.CPU_Values[A.CPU_Columns[j]];

		Y.CPU_Values[i] = YI;
	}

	return true;
}

//
// GPUGPUCSRMatrixVectorMultiply
// Matrix-Vector multiply on GPU

bool GPUGPUCSRMatrixVectorMultiply(GPUCSRMatrix &A, GPUVector &X, GPUVector &Y)
{
	// Primary checks
	if (A.Size2 != X.Size || X.Size != Y.Size || !X.Allocated || !Y.Allocated)
		return false;

#ifdef USE_TEXTURE_CACHING

	// Bind the texture memory to X
	Bind_X(X.GPU_Values);

#endif


	bool UseVectorizedVersion = (A.NNZ / A.Size2) > 10;	// From nVidia forum

	if (UseVectorizedVersion)
	{
		dim3 Grid = Build_Grid(A.Size1 *  HALF_WARP_SIZE, BLOCK_SIZE);
		GPUGPUCSRMatrixVectorMultiply_CSRGPUVectorized_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, X.GPU_Values, Y.GPU_Values);

		if (!GPUSparse::CUDA_Success(hipGetLastError()))
			return false;
	}

	else

	{
		dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
		GPUGPUCSRMatrixVectorMultiply_CSR_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, X.GPU_Values, Y.GPU_Values);

		if (!GPUSparse::CUDA_Success(hipGetLastError()))
			return false;
	}

#ifdef USE_TEXTURE_CACHING

	// Unbind the texture memory
	Unbind_X();
	

#endif

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPUGPUCSRMatrixGetDiagonals
// Extract the diagonal elements of a matrix into a vector on CPU

bool CPUGPUCSRMatrixGetDiagonals(GPUCSRMatrix &A, GPUVector &X)
{
	// Primary checks
	if (A.Size1 != A.Size2 || A.Size2 != X.Size)
		return false;

	for (size_t i = 0; i < A.Size1; i++)
	{
		X.CPU_Values[i] = static_cast <double> (0);

		for (size_t j = A.CPU_RowIndices[i]; j < A.CPU_RowIndices[i + 1]; j++)
			if (A.CPU_Columns[j] == i)
				X.CPU_Values[i] = A.CPU_Values[j];
	}

	return true;
}

//
// GPUGPUCSRMatrixGetDiagonals
// Extract the diagonal elements of a matrix into a vector on GPU

bool GPUGPUCSRMatrixGetDiagonals(GPUCSRMatrix &A, GPUVector &X)
{
	// Primary checks
	if (A.Size1 != A.Size2 || A.Size2 != X.Size || !A.Allocated || !X.Allocated)
		return false;

	dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	GPUGPUCSRMatrixGetDiagonals_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, X.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPUGPUCSRMatrixMatrixDiagonalMultiply
// Multiply a digonal matrix specified with a vector with a matrix on CPU

bool CPUGPUCSRMatrixMatrixDiagonalMultiply(GPUVector &X, GPUCSRMatrix &A)
{
	// Primary checks
	if (X.Size != A.Size1)
		return false;

	for (size_t i = 0; i < X.Size; i++)
	{
		double t = X.CPU_Values[i];

		for (size_t j = A.CPU_RowIndices[i]; j < A.CPU_RowIndices[i + 1]; j++)
				A.CPU_Values[j] *= t;
	}

	return true;
}

//
// GPUGPUCSRMatrixMatrixDiagonalMultiply
// Multiply a digonal matrix specified with a vector with a matrix on GPU

bool GPUGPUCSRMatrixMatrixDiagonalMultiply(GPUVector &X, GPUCSRMatrix &A)
{
	// Primary checks
	if (X.Size != A.Size1 || !X.Allocated || !A.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);
	GPUGPUCSRMatrixMatrixDiagonalMultiply_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, X.GPU_Values, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPUGPUVectorPrepareDiagonalPreconditionerValues
// Prepare diagonal values of the matrix for Diagonal Preconditioner on CPU

bool CPUGPUVectorPrepareDiagonalPreconditionerValues(GPUVector &X)
{
	for (size_t i = 0; i < X.Size; i++)
		if (X.CPU_Values[i] == 0.00)
			X.CPU_Values[i] = 1.00;
		else
			X.CPU_Values[i] = 1.00 / X.CPU_Values[i];
//			X.CPU_Values[i] = 1.00 / sqrt(abs(X.CPU_Values[i]));

	return true;
}

//
// GPUGPUVectorPrepareDiagonalPreconditionerValues
// Prepare diagonal values of the matrix for Diagonal Preconditioner on GPU

bool GPUGPUVectorPrepareDiagonalPreconditionerValues(GPUVector &X)
{
	// Primary check
	if (!X.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);
	GPUGPUVectorPrepareDiagonalPreconditionerValues_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, X.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// GPU_PrepareSPAIPreconditioner
// Prepare SPAI preconditioner on GPU

bool GPU_PrepareSPAIPreconditioner(GPUCSRMatrix &A, GPUCSRMatrix &M)
{
	// Primary checks
	if (A.Size1 != M.Size1 || A.Size2 != M.Size2 || A.NNZ != M.NNZ || !A.Allocated || !M.Allocated)
		return false;

	dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	GPU_SPAIPreconditioner_CSR_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, M.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPUGPUVectorVectorMultiply
// Vector-Vector multiply on CPU

bool CPUGPUVectorVectorMultiply(GPUVector &X, GPUVector &Y, double &Result)
{
	// Primary check
	if (X.Size != Y.Size)
		return false;

	Result = static_cast <double> (0);

	for (size_t i = 0; i < X.Size; i++)
		Result += X.CPU_Values[i] * Y.CPU_Values[i];

	return true;
}

//
// GPUGPUVectorVectorMultiply
// Vector-Vector multiply on GPU

bool GPUGPUVectorVectorMultiply(GPUVector &X, GPUVector &Y, double &Result)
{
	// Primary check
	if (X.Size != Y.Size || !X.Allocated || !Y.Allocated)
		return false;

	Result = hipblasDdot(X.Size, X.GPU_Values, 1, Y.GPU_Values, 1);

	return CUBLAS_Success(cublasGetError());
}

//
// CPUGPUVectorVectorMultiplyElementWise
// Vector-Vector element-wise multiply on CPU

bool CPUGPUVectorVectorMultiplyElementWise(GPUVector &X, GPUVector &Y,  GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size)
		return false;

	for (size_t i = 0; i < X.Size; i++)
		Z.CPU_Values[i] = X.CPU_Values[i] * Y.CPU_Values[i];

	return true;
}

//
// GPUGPUVectorVectorMultiplyElementWise
// Vector-Vector element-wise multiply on GPU

bool GPUGPUVectorVectorMultiplyElementWise(GPUVector &X, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size || !X.Allocated || !Y.Allocated || !Z.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	GPUGPUVectorVectorMultiplyElementWise_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, X.GPU_Values, Y.GPU_Values, Z.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPUGPUVectorNorm2
// Vector norm 2 on CPU

bool CPUGPUVectorNorm2(GPUVector &X, double &Result)
{
	Result = static_cast <double> (0);

	for (size_t i = 0; i < X.Size; i++)
		Result += X.CPU_Values[i] * X.CPU_Values[i];

	Result = sqrt(Result);

	return true;
}

//
// GPUGPUVectorNorm2
// Vector norm 2 on GPU

bool GPUGPUVectorNorm2(GPUVector &X, double &Result)
{
	// Primary check
	if (!X.Allocated)
		return false;

	Result = hipblasDnrm2(X.Size, X.GPU_Values, 1);

	return CUBLAS_Success(cublasGetError());
}

//
// CPUGPUVectorScaleAndAdd
// Vector scale-and-add on CPU

// Variant 1: Z = A * X + B * Y

bool CPUGPUVectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size)
		return false;

	for (size_t i = 0; i < X.Size; i++)
		Z.CPU_Values[i] = A * X.CPU_Values[i] + B * Y.CPU_Values[i];

	return true;
}

// Variant 2: Y = A * X + B * Y

bool CPUGPUVectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y)
{
	// Primary check
	if (X.Size != Y.Size)
		return false;

	for (size_t i = 0; i < X.Size; i++)
		Y.CPU_Values[i] = A * X.CPU_Values[i] + B * Y.CPU_Values[i];

	return true;
}

//
// GPUGPUVectorScaleAndAdd
// Vector scale-and-add on GPU

// Variant 1: Z = A * X + B * Y

bool GPUGPUVectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size || !X.Allocated || !Y.Allocated || !Z.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	if (A == 1.00)
	{
	//printf("A = 1.0\n");
		if (B == 1.00)
			GPUGPUVectorScaleAndAdd_1_A_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else if (B == -1.00)
			GPUGPUVectorScaleAndAdd_1_B_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else
			GPUGPUVectorScaleAndAdd_1_E_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}

	else if (A == -1.00)
	{
	//printf("A = -1.0\n");
		if (B == 1.00)
			GPUGPUVectorScaleAndAdd_1_C_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else if (B == -1.00)
			GPUGPUVectorScaleAndAdd_1_D_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else
			GPUGPUVectorScaleAndAdd_1_F_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}

	else
	{
	//printf("B = 1.0\n");
		if (B == 1.00)
			GPUGPUVectorScaleAndAdd_1_G_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else if (B == -1.00)
			GPUGPUVectorScaleAndAdd_1_H_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else
			GPUGPUVectorScaleAndAdd_1_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}

	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

// Variant 2: Y = A * X + B * Y

bool GPUGPUVectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y)
{
	// Primary check
	if (X.Size != Y.Size || !X.Allocated || !Y.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	if (A == 1.00)
	{
		if (B == 1.00)
			GPUGPUVectorScaleAndAdd_2_A_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else if (B == -1.00)
			GPUGPUVectorScaleAndAdd_2_B_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else
			GPUGPUVectorScaleAndAdd_2_E_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);
	}

	else if (A == -1.00)
	{
		if (B == 1.00)
			GPUGPUVectorScaleAndAdd_2_C_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else if (B == -1.00)
			GPUGPUVectorScaleAndAdd_2_D_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else
			GPUGPUVectorScaleAndAdd_2_F_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);
	}

	else
	{
		if (B == 1.00)
			GPUGPUVectorScaleAndAdd_2_G_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else if (B == -1.00)
			GPUGPUVectorScaleAndAdd_2_H_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else
			GPUGPUVectorScaleAndAdd_2_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);
	}

	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

/*double GPU_dotProduct(size_t numElems, const double *firstVec, int incFirstVec,
    const double *secondVec, int incSecondVec){
    return hipblasDdot(numElems, firstVec, incFirstVec, secondVec, incSecondVec);
}*/


/** ADDED FUNCTIONS **/

void GPU_fillWithZeros(size_t numElems, double* gpuVec){
	dim3 grid = Build_Grid(numElems, BLOCK_SIZE);
	fillWithZeros <<< grid, BLOCK_SIZE >>> (gpuVec, numElems);
}

bool GPUGPUVectorScaleAndAdd_addingVersion(double A, GPUVector &X, double B, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size || !X.Allocated || !Y.Allocated || !Z.Allocated){
		//printf("Falla x la comprovació\n");		
		return false;
	}
		
	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	if (A == 1.00)
	{
	//printf("A = 1.0\n");
		if (B == 1.00)
			GPUGPUVectorScaleAndAdd_1_A_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

			
		else if (B == -1.00)
			GPUGPUVectorScaleAndAdd_1_B_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
				
		else
			GPUGPUVectorScaleAndAdd_1_E_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}
	
	else if (A == -1.00)
	{
	//printf("A = -1.0\n");
		if (B == 1.00)
			GPUGPUVectorScaleAndAdd_1_C_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
			
		else if (B == -1.00)
			GPUGPUVectorScaleAndAdd_1_D_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
			
		else
			GPUGPUVectorScaleAndAdd_1_F_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}
	
	else
	{
	//printf("B = 1.0\n");
		if (B == 1.00)
			GPUGPUVectorScaleAndAdd_1_G_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
			
		else if (B == -1.00)
			GPUGPUVectorScaleAndAdd_1_H_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
			
		else
			GPUGPUVectorScaleAndAdd_1_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}
	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
	
}

/** temp variables for LU decomposition **/
LaVectorLongInt ipiv;
typedef struct{
    size_t Size;
    double* CPU_Values;
    double* GPU_Values;
}_Vector;

/** functions from scipy for mat-mat calculation **/
template <class I>
void csr_matmat_pass1(const I n_row,
                      const I n_col,
                      const I Ap[],
                      const I Aj[],
                      const I Bp[],
                      const I Bj[],
                            I Cp[]){
    int* mask = new int[n_col];
    for(size_t aux = 0; aux < n_col; aux++){
	mask[aux] = -1;
    }
    Cp[0] = 0;

    I nnz = 0;
    for(I i = 0; i < n_row; i++){
        for(I jj = Ap[i]; jj < Ap[i+1]; jj++){
            I j = Aj[jj];
            for(I kk = Bp[j]; kk < Bp[j+1]; kk++){
                I k = Bj[kk];
                if(mask[k] != (int)i){
                    mask[k] = (int)i;
                    nnz++;
                }
            }
        }
        Cp[i+1] = nnz;
    }
    delete[] mask;
}

template <class I, class T>
void csr_matmat_pass2(const I n_row,
      	              const I n_col,
      	              const I Ap[],
      	              const I Aj[],
      	              const T Ax[],
      	              const I Bp[],
      	              const I Bj[],
      	              const T Bx[],
      	                    I Cp[],
      	                    I Cj[],
      	                    T Cx[])
{


    int* next = new int[n_col];
    T* sums = new T[n_col];
    for(size_t aux = 0; aux < n_col; aux++){
	next[aux] = -1;
	sums[aux] = 0;
    }

    I nnz = 0;

    Cp[0] = 0;

    for(I i = 0; i < n_row; i++){
        int head   = -2;
        I length =  0;

        I jj_start = Ap[i];
        I jj_end   = Ap[i+1];
        for(I jj = jj_start; jj < jj_end; jj++){
            I j = Aj[jj];
            T v = Ax[jj];

            I kk_start = Bp[j];
            I kk_end   = Bp[j+1];
            for(I kk = kk_start; kk < kk_end; kk++){
                I k = Bj[kk];

                sums[k] += v*Bx[kk];

                if(next[k] == -1){
                    next[k] = head;
                    head = (int)k;
                    length++;
                }
            }
        }

        for(I jj = 0; jj < length; jj++){

            if(sums[head] != 0){
                Cj[nnz] = (size_t)head;
                Cx[nnz] = sums[head];
                nnz++;
            }

            I temp = (size_t)head;
            head = (I)next[head];

            next[temp] = -1; //clear arrays
            sums[temp] =  0;
        }

        Cp[i+1] = nnz;
    }

	delete[] sums;
	delete[] next;
}

/** maxLevels define the maxLevels of that execution
	G defines the diagonals of each lvl of A, created on previous step**/
void multilevel(GPUCSRMatrix**& A, GPUCSRMatrix**& P, GPUCSRMatrix**& R, GPUCSRMatrix**& G, GPUVector& b, GPUVector& u,
			unsigned short lvl, unsigned short maxLevels, size_t* preSweeps, size_t* postSweeps, bool assumeZeros)
{
    bool vectorized = (A[lvl]->NNZ / A[lvl]->Size2) > 10;

    //calculateInstantVector(u, b, A[lvl], G[lvl]);
    if(lvl < maxLevels){

	double* vecR = new double[R[lvl]->Size1];
    	GPUVector r(R[lvl]->Size1, vecR);
	r.GPU_Allocate();
	
	//clock_t t1 = clock();
        if(assumeZeros) //we receive from the upper level a zero start vector
        {
            if(preSweeps[lvl] != 0)
            {
                //first iteration (does not require computation of residual
                dim3 Grid = Build_Grid(A[lvl]->Size1, BLOCK_SIZE);
                
#ifdef USE_TEXTURE_CACHING

				// Bind the texture memory to X
				Bind_X(b.GPU_Values);

#endif
                
                GPUGPUCSRMatrixVectorMultiply_CSR_Kernel_addingVersion <<< Grid, BLOCK_SIZE >>>(G[lvl]->Size1, G[lvl]->GPU_Columns, G[lvl]->GPU_RowIndices, G[lvl]->GPU_Values, b.GPU_Values, u.GPU_Values);
                if(!CUDA_Success(hipDeviceSynchronize())){
                    printf("Error en linea 130\n");
                }
                
#ifdef USE_TEXTURE_CACHING

				// Unbind the texture memory
				Unbind_X();
	
#endif

                //from the second sweel on we need to recompute the residual
                for(size_t i = 1; i < preSweeps[lvl]; i++){
		    if(!vectorized)
			calculateInstantVector(u, b, *A[lvl], *G[lvl]);
		    else
			calculateInstantVectorGPUVectorized(u, b, *A[lvl], *G[lvl]);
		}

                if(!vectorized)
                    generateResidual(*R[lvl], b, *A[lvl], u, r);
                else
                    generateResidualGPUVectorized(*R[lvl], b, *A[lvl], u, r);


            }
            else //preSweeps[0] == 0 case
            {
                //inefficient! -- in this case we do not need to recompute the residual
                if(!vectorized)
                    generateResidual(*R[lvl], b, *A[lvl], u, r);
                else
                    generateResidualGPUVectorized(*R[lvl], b, *A[lvl], u, r);

            }
        }
        else
        {
          //from the second sweel on we need to recompute the residual
                for(size_t i = 0; i < preSweeps[lvl]; i++){
		    if(!vectorized)
			calculateInstantVector(u, b, *A[lvl], *G[lvl]);
		    else
			calculateInstantVectorGPUVectorized(u, b, *A[lvl], *G[lvl]);
		}


                if(!vectorized)
                    generateResidual(*R[lvl], b, *A[lvl], u, r);
                else
                    generateResidualGPUVectorized(*R[lvl], b, *A[lvl], u, r);
        }
	double *vecV = new double[r.Size];
        GPUVector v(r.Size, vecV);
        v.GPU_Allocate();
        dim3 Grid = Build_Grid(v.Size, BLOCK_SIZE);
        fillWithZeros <<< Grid, BLOCK_SIZE >>>(v.GPU_Values, v.Size);
        if(!CUDA_Success(hipDeviceSynchronize())){
                printf("Error en linea 160\n");
        }

        multilevel(A, P, R, G, r, v, lvl+1, maxLevels, preSweeps, postSweeps, assumeZeros);

        GPUVector pv(P[lvl]->Size1);
	pv.GPU_Allocate();
        Grid = Build_Grid(pv.Size, BLOCK_SIZE);
        // here product matrix P with vector v
        
#ifdef USE_TEXTURE_CACHING

		// Bind the texture memory to X
		Bind_X(v.GPU_Values);

#endif
        
        GPUGPUCSRMatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(P[lvl]->Size1, P[lvl]->GPU_Columns, P[lvl]->GPU_RowIndices, P[lvl]->GPU_Values, v.GPU_Values, pv.GPU_Values);
        if(!CUDA_Success(hipDeviceSynchronize())){
            printf("Error en linea 173\n");
        }
        
#ifdef USE_TEXTURE_CACHING

		// Unbind the texture memory
		Unbind_X();
	
#endif

        // here the addition of pv to u
        Grid = Build_Grid(u.Size, BLOCK_SIZE);
        sumVectorVector <<< Grid, BLOCK_SIZE >>> (pv.GPU_Values, u.GPU_Values, u.Size);
        if(!CUDA_Success(hipDeviceSynchronize())){
            printf("Error in the line 180");
        }
        //delete de pv, v i r
        delete[] vecV;
        delete[] vecR;

        //double norm2 = checkResidual(u, b, A[lvl]);

	for(size_t i = 0; i < postSweeps[lvl]; i++){
	    if(!vectorized)
		calculateInstantVector(u, b, *A[lvl], *G[lvl]);
	    else
		calculateInstantVectorGPUVectorized(u, b, *A[lvl], *G[lvl]);
	}

    }else{
	//clock_t t1 = clock();
        //here lapack direct solver

	u.Copy(GPU_CPU);
	b.Copy(GPU_CPU);


        LaGenMatDouble _A(A[lvl]->matAuxValues, A[lvl]->Size1, A[lvl]->Size2);
        LaGenMatDouble _b(b.CPU_Values, b.Size, 1);
        LaGenMatDouble _x(u.CPU_Values, u.Size, 1);
//	LaLinearSolve( _A, _x, _b );

    	_x.inject(_b);            // will throw exception if not conformant

	integer info = 0;
	int M = _A.size(0);
	integer Ml = M;
	integer lda = _A.inc(0) * _A.gdim(0);

	integer K = _x.size(1);
	integer ldx = _x.inc(0) * _x.gdim(0);
	F77NAME(dgetrs) ("No transpose", &Ml, &K, &_A(0,0), &lda, &ipiv(0), &_x(0,0), &ldx, &info);

	//int res = clapack_dgetrs(CblasRowMajor, CblasNoTrans, &Ml, &K, &_A(0,0), &lda, &ipiv(0), &_x(0,0), &ldx);

	//copyMem(u.CPU_Values, u.GPU_Values, u.Size, 0);
	u.Copy(CPU_GPU);

	//clock_t t2 = clock();

   }

}
/** This function is a wrapper for u += G ( A, b, u) **/
void calculateInstantVector(GPUVector& u, const GPUVector& b, const GPUCSRMatrix& A, const GPUCSRMatrix& G)
{
		/** Au **/
	GPUVector auxAU(A.Size1);
	auxAU.GPU_Allocate();
	dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	
#ifdef USE_TEXTURE_CACHING

	// Bind the texture memory to X
	Bind_X(u.GPU_Values);

#endif
	
	GPUGPUCSRMatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, u.GPU_Values, auxAU.GPU_Values);
	if(!CUDA_Success(hipDeviceSynchronize())){
	printf("Error in the line 238");
	}

#ifdef USE_TEXTURE_CACHING

	// Unbind the texture memory
	Unbind_X();

#endif

		/** b - AU **/
	GPUVector auxABU(A.Size1);
	auxABU.GPU_Allocate();
	Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	subVectorVector <<< Grid, BLOCK_SIZE >>>(b.GPU_Values, auxAU.GPU_Values, auxABU.GPU_Values, auxABU.Size);
	if(!CUDA_Success(hipDeviceSynchronize())){
	printf("Error in the line 249");
	}
		/** u += G ( b - Au ) **/
	Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	
#ifdef USE_TEXTURE_CACHING

	// Bind the texture memory to X
	Bind_X(auxABU.GPU_Values);

#endif
	
	GPUGPUCSRMatrixVectorMultiply_CSR_Kernel_addingVersion <<< Grid, BLOCK_SIZE >>>(G.Size1, G.GPU_Columns, G.GPU_RowIndices, G.GPU_Values, auxABU.GPU_Values, u.GPU_Values);
	if(!CUDA_Success(hipDeviceSynchronize())){
	printf("Error in the line 255");
	}

#ifdef USE_TEXTURE_CACHING

	// Unbind the texture memory
	Unbind_X();

#endif
	
	//deleting structures
}

void calculateInstantVectorGPUVectorized(GPUVector& u, const GPUVector& b, const GPUCSRMatrix& A, const GPUCSRMatrix& G)
{
		/** Au **/
	GPUVector auxAU(A.Size1);
	auxAU.GPU_Allocate();
	dim3 Grid = Build_Grid(A.Size1 * HWS, BS);
	GPUGPUCSRMatrixVectorMultiply_CSRGPUVectorized_Kernel <<< Grid, BS >>>(A.Size1, A.GPU_Columns,
	    A.GPU_RowIndices, A.GPU_Values, u.GPU_Values, auxAU.GPU_Values);
	if(!CUDA_Success(hipDeviceSynchronize())){
	printf("Error in the line 272");
	}
		/** b - AU **/
	GPUVector auxABU(A.Size1);
	auxABU.GPU_Allocate();
	Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	subVectorVector <<< Grid, BLOCK_SIZE >>>(b.GPU_Values, auxAU.GPU_Values, auxABU.GPU_Values, auxABU.Size);
	if(!CUDA_Success(hipDeviceSynchronize())){
	printf("Error in the line 282");
	}
		/** u += G ( b - Au ) **/
	Grid = Build_Grid(A.Size1, BS);

#ifdef USE_TEXTURE_CACHING

	// Bind the texture memory to X
	Bind_X(auxABU.GPU_Values);

#endif

	GPUGPUCSRMatrixVectorMultiply_CSR_Kernel_addingVersion <<< Grid, BS >>>(G.Size1, G.GPU_Columns, G.GPU_RowIndices, G.GPU_Values, auxABU.GPU_Values, u.GPU_Values);
	if(!CUDA_Success(hipDeviceSynchronize())){
		printf("Error in the line 288");
	}
	
#ifdef USE_TEXTURE_CACHING

	// Unbind the texture memory
	Unbind_X();
	
#endif
	
	//deleting structures
}

void generateResidual(const GPUCSRMatrix& R, const GPUVector& b, const GPUCSRMatrix& A, const GPUVector& u, GPUVector& r){
		/** Au **/
	GPUVector auxAU(A.Size1);
	auxAU.GPU_Allocate();
	dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	
#ifdef USE_TEXTURE_CACHING

	// Bind the texture memory to X
	Bind_X(u.GPU_Values);

#endif
	
	GPUGPUCSRMatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, u.GPU_Values, auxAU.GPU_Values);
	if(!CUDA_Success(hipDeviceSynchronize())){
		printf("Error in the line 305");
	}

#ifdef USE_TEXTURE_CACHING

	// Unbind the texture memory
	Unbind_X();

#endif

		/** b - AU **/
	GPUVector auxABU(A.Size1);
	auxABU.GPU_Allocate();
	Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	subVectorVector <<< Grid, BLOCK_SIZE >>>(b.GPU_Values, auxAU.GPU_Values,
	    auxABU.GPU_Values, auxABU.Size);
	if(!CUDA_Success(hipDeviceSynchronize())){
		printf("Error in the line 315");
	}
		/** r = R ( b - Au ) **/
	Grid = Build_Grid(R.Size1, BLOCK_SIZE);
	
#ifdef USE_TEXTURE_CACHING

	// Bind the texture memory to X
	Bind_X(auxABU.GPU_Values);

#endif
	
	GPUGPUCSRMatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(R.Size1, R.GPU_Columns, R.GPU_RowIndices, R.GPU_Values, auxABU.GPU_Values, r.GPU_Values);
	if(!CUDA_Success(hipDeviceSynchronize())){
		printf("Error in the line 325");
	}
	
#ifdef USE_TEXTURE_CACHING

	// Unbind the texture memory
	Unbind_X();

#endif	

    //deleting structures
}

void generateResidualGPUVectorized(const GPUCSRMatrix& R, const GPUVector& b, const GPUCSRMatrix& A, const GPUVector& u, GPUVector& r){
		/** Au **/
		GPUVector auxAU(A.Size1);
	auxAU.GPU_Allocate();
	dim3 Grid = Build_Grid(A.Size1 * HWS, BS);
	GPUGPUCSRMatrixVectorMultiply_CSRGPUVectorized_Kernel <<< Grid, BS >>>(A.Size1, A.GPU_Columns,
	    A.GPU_RowIndices, A.GPU_Values, u.GPU_Values, auxAU.GPU_Values);
	if(!CUDA_Success(hipDeviceSynchronize())){
	printf("Error in the line 340");
	}
		/** b - AU **/
	GPUVector auxABU(A.Size1);
	auxABU.GPU_Allocate();
	Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	subVectorVector <<< Grid, BLOCK_SIZE >>>(b.GPU_Values, auxAU.GPU_Values,
	    auxABU.GPU_Values, auxABU.Size);
	if(!CUDA_Success(hipDeviceSynchronize())){
	printf("Error in the line 350");
	}
		/** r = R ( b - Au ) **/
	Grid = Build_Grid(R.Size1 *  HWS, BS);
	GPUGPUCSRMatrixVectorMultiply_CSRGPUVectorized_Kernel <<< Grid, BS >>>(R.Size1, R.GPU_Columns,
	    R.GPU_RowIndices, R.GPU_Values, auxABU.GPU_Values, r.GPU_Values);
	if(!CUDA_Success(hipDeviceSynchronize())){
	printf("Error in the line 360");
	}
	//deleting structures

}

/** This function will return the hierarchy reconstructed of A and b;
    Additionally return the number of real hierarchy levels.
    All matrices and vectors are returned as GPU structures**/
size_t generateHierarchy(GPUCSRMatrix**& Matrices, GPUCSRMatrix**& Pmat, GPUCSRMatrix**& Qmat,
        GPUCSRMatrix**& Gmat, double W, size_t numLevelsRoh, size_t max_levels, size_t min_system_size)
{
    size_t i = 0;
    for(i = 0; i < max_levels; i++){
        /** This condition controls MAX_SYSTEM_SIZE for the last matrix in hierarchy **/
        if(Matrices[i]->Size1 < min_system_size || i == max_levels-1){
	    Matrices[i]->GenerateDenseRepresentation();
            LaGenMatDouble A(Matrices[i]->matAuxValues, Matrices[i]->Size1, Matrices[i]->Size2);

            int M = A.size(0);
            integer Ml = M;
            integer lda = A.inc(0) * A.gdim(0);

            LaVectorLongInt ipiv_( M);
            ipiv = ipiv_;
            integer info = 0;
            int max = M > 1 ? M : 1;
            if(M < 0){
                info = -1;
            }else if(M < 0){
                //este caso lo saltamos porque tratamos matrices cuadradas y por tanto se asume en el caso anterior
            }else if(lda < max){
                info  = -4;
            }
            F77NAME(dgetrf) (&Ml, &Ml, &A(0,0), &lda, &ipiv(0), &info);

		
/*	    if( i > 0 ){ //A matrix is not the first one in hierarchy
		    if(Matrices[i].CPU_RowIndices != NULL) delete[] Matrices[i].CPU_Columns;
		    if(Matrices[i].CPU_Values != NULL) delete[] Matrices[i].CPU_Values;
		    if(Matrices[i].CPU_Columns != NULL) delete[] Matrices[i].CPU_RowIndices;
	    }*/
            Matrices[i]->numValuesDenseRep = A.inc(0) * A.gdim(0);

            break;
        }

	double *values, *values_2;
	size_t *ptr, *indices, *ptr_2, *indices_2;
	size_t NNZ, NNZ_2, Size1, Size2;
	double *vecDiag;
        createDiagonal_vCPU(*Matrices[i], NNZ, Size1, Size2, indices, ptr, values, vecDiag);
        GPUVector diag(Matrices[i]->Size1, vecDiag);
	Gmat[i] = new GPUCSRMatrix(NNZ, Size1, Size2, indices, ptr, values, false);
	Gmat[i]->GPU_Allocate();
	Gmat[i]->Copy(CPU_GPU, false);
	delete[] values;
	delete[] ptr;
	delete[] indices;
	
        /** Generating P and Q for the current A level **/
	
        generateP_vCPU(*Matrices[i], diag, W, numLevelsRoh, NNZ, Size1, Size2, indices, ptr, values);
	Pmat[i] = new GPUCSRMatrix(NNZ, Size1, Size2, indices, ptr, values, false);
	Pmat[i]->GPU_Allocate();
	Pmat[i]->Copy(CPU_GPU, false);
	delete[] values;
	delete[] ptr;
	delete[] indices;
        generateQ(*Pmat[i], NNZ, Size1, Size2, indices, ptr, values);
        Qmat[i] = new GPUCSRMatrix(NNZ, Size1, Size2, indices, ptr, values, false);
	Qmat[i]->GPU_Allocate();
	Qmat[i]->Copy(CPU_GPU, false);
	delete[] values;
	delete[] ptr;
	delete[] indices;
        //CUDA_CHECK(hipFree(diag.GPU_Values));
	delete[] diag.CPU_Values;

        /** Allocating result matrix, and partialResult matrix **/

	//R * A = AUX
	ptr = new size_t[Qmat[i]->Size1+1];

        csr_matmat_pass1(Qmat[i]->Size1,
                      Matrices[i]->Size2,
                      Qmat[i]->CPU_RowIndices,
                      Qmat[i]->CPU_Columns,
                      Matrices[i]->CPU_RowIndices,
                      Matrices[i]->CPU_Columns,
                            ptr);
        NNZ = ptr[Qmat[i]->Size1];
        indices = new size_t[NNZ];
        values = new double[NNZ];
        csr_matmat_pass2(Qmat[i]->Size1,
      	              Matrices[i]->Size2,
      	              Qmat[i]->CPU_RowIndices,
                      Qmat[i]->CPU_Columns,
      	              Qmat[i]->CPU_Values,
      	              Matrices[i]->CPU_RowIndices,
                      Matrices[i]->CPU_Columns,
      	              Matrices[i]->CPU_Values,
      	                    ptr,
      	                    indices,
                            values);
        NNZ = ptr[Qmat[i]->Size1];
	//newA = AUX * P
        ptr_2 = new size_t[Qmat[i]->Size1+1];
        csr_matmat_pass1(Qmat[i]->Size1,
                      Pmat[i]->Size2,
                      ptr,
                      indices,
                      Pmat[i]->CPU_RowIndices,
                      Pmat[i]->CPU_Columns,
                            ptr_2);
        NNZ_2 = ptr_2[Qmat[i]->Size1];
        indices_2 = new size_t[NNZ_2];
        values_2 = new double[NNZ_2];
        csr_matmat_pass2(Qmat[i]->Size1,
                      Pmat[i]->Size2,
                      ptr,
                      indices,
      	              values,
      	              Pmat[i]->CPU_RowIndices,
                      Pmat[i]->CPU_Columns,
      	              Pmat[i]->CPU_Values,
      	                    ptr_2,
      	                    indices_2,
      	                    values_2);
        NNZ_2 = ptr_2[Qmat[i]->Size1];

	/** Store new matrix in the next lvl of hierarchy, i+1 **/
	Matrices[i+1] = new GPUCSRMatrix(NNZ_2, Qmat[i]->Size1, Pmat[i]->Size2, indices_2, ptr_2, values_2, false);

        sortMatrix(Matrices[i+1]->CPU_RowIndices, Matrices[i+1]->CPU_Columns, Matrices[i+1]->CPU_Values, Matrices[i+1]->Size1, Matrices[i+1]->NNZ, true);
        if(Matrices[i+1]->Size1 >= min_system_size || i+1 == max_levels-1 ){
		Matrices[i+1]->GPU_Allocate();
		Matrices[i+1]->Copy(CPU_GPU, false);
        }

        /** Free memory from useless structures **/
	delete[] values;
	delete[] ptr;
	delete[] indices;
	delete[] values_2;
	delete[] ptr_2;
	delete[] indices_2;

    }
    return i;
}

void generateP_vCPU(const GPUCSRMatrix& A, const GPUVector& diag, double W, size_t numLevelsRoh, size_t &pNNZ, size_t &pSize1, size_t &pSize2, size_t *&pindices, size_t *&pptr, double *&pvalues){
    size_t NNZ_ptent, Size1_ptent, Size2_ptent, *ptr_ptent, *indices_ptent;
    double *values_ptent;
    createPTent(A, NNZ_ptent, Size1_ptent, Size2_ptent, ptr_ptent, indices_ptent, values_ptent);

    /** Create P from Ptent **/
    //wDA

    size_t NNZ_prod, *ptr_prod, *indices_prod;
    double *values_prod;

    indices_prod = new size_t[A.NNZ];
    values_prod = new double[A.NNZ];
    ptr_prod = new size_t[A.Size1+1];

    ptr_prod[0] = 0;
    size_t currentIndice = 0;
    for(size_t i = 0; i < A.Size1; i++){
        ptr_prod[i+1] = ptr_prod[i];
        if(diag.CPU_Values[i] != 0.0){
            for(size_t r = A.CPU_RowIndices[i]; r < A.CPU_RowIndices[i+1]; r++){
                ptr_prod[i+1]++;
                indices_prod[currentIndice] = A.CPU_Columns[r];
                values_prod[currentIndice] = A.CPU_Values[r] * diag.CPU_Values[i];
                currentIndice++;
            }
        }
    }

    NNZ_prod = ptr_prod[A.Size1];
    GPUCSRMatrix prodMat(NNZ_prod, A.Size1, A.Size2, indices_prod, ptr_prod, values_prod);
    delete[] indices_prod;
    delete[] values_prod;
    delete[] ptr_prod;
    prodMat.GPU_Allocate();
    prodMat.Copy(CPU_GPU, false);

    double roh_ = roh(prodMat, numLevelsRoh);
    double W_ = W / roh_;

    for(size_t i = 0; i < prodMat.NNZ; i++){
        prodMat.CPU_Values[i] *= W_;
    }

    //I - wDA

    size_t subNNZ = prodMat.Size1 + prodMat.NNZ;
    size_t* ptr_sub = new size_t[prodMat.Size1+1];
    size_t* indices_sub = new size_t[subNNZ];
    double* values_sub = new double[subNNZ];

    subIdentityMatrix_cpu(prodMat, ptr_sub, indices_sub, values_sub);

    //(I - wDA) * PTent
    pSize1 = prodMat.Size1;
    pSize2 = Size2_ptent;
    pptr = new size_t[pSize1+1];
    
    csr_matmat_pass1(prodMat.Size1,
                      Size2_ptent,
                      ptr_sub,
                      indices_sub,
                      ptr_ptent,
                      indices_ptent,
                            pptr);
    pNNZ = pptr[pSize1];
    pindices = new size_t[pNNZ];
    pvalues = new double[pNNZ];
    csr_matmat_pass2(prodMat.Size1,
                      Size2_ptent,
                      ptr_sub,
                      indices_sub,
      	              values_sub,
      	              ptr_ptent,
                      indices_ptent,
      	              values_ptent,
      	                    pptr,
                            pindices,
                            pvalues);
    pNNZ = pptr[pSize1];

    sortMatrix(pptr, pindices, pvalues, pSize1, pNNZ, true);

    /** Free resources **/
    delete[] values_ptent;
    delete[] indices_ptent;
    delete[] ptr_ptent;

    delete[] indices_sub;
    delete[] values_sub;
    delete[] ptr_sub;

}

/** This is a simple function that transposes P assuming symmetric matrix
 *  Soon it will be need to implementate the non-symmetric construction of Q*/
void generateQ(const GPUCSRMatrix &P, size_t &NNZ, size_t &Size1, size_t &Size2, size_t *&indices, size_t *&ptr, double *&values){
    NNZ = P.NNZ;
    Size2 = P.Size1;
    Size1 = P.Size2;
    indices = new size_t[NNZ];
    values = new double[NNZ];
    ptr = new size_t[Size1+1];
    //ens es suficient amb utilitzar la funcio csr_tocsc i prendre-ho com csr
    csr_tocsc(P.Size1,
	           P.Size2,
	           P.CPU_RowIndices,
	           P.CPU_Columns,
	           P.CPU_Values,
	                 ptr,
	                 indices,
	                 values);
}

/** This function will create G from diagonal of A **/
void createDiagonal(const GPUCSRMatrix &A, size_t &Size, double *&values){
    Size = A.Size1;
    values = new double[Size];

    for(size_t i = 0; i < A.Size1; i++){
        values[i] = 0.0;
        for(size_t r = A.CPU_RowIndices[i]; r < A.CPU_RowIndices[i+1]; r++){
            if(A.CPU_Columns[r] == i){
		if(fabs(A.CPU_Values[r]) > 1e-30)
                	values[i] =1.0/A.CPU_Values[r];
		else{
			values[i]  =1.0;
		}
	
                break;
            }
        }
    }

}

void createDiagonal_vCPU(const GPUCSRMatrix& A, size_t &NNZ, size_t &Size1, size_t &Size2, size_t *&indices, size_t *&ptr, double *&values, double *&vecDiag){

    vecDiag = new double[A.Size1];

    Size1 = A.Size1;
    Size2 = A.Size2;

    ptr = new size_t[Size1+1];
    indices = new size_t[Size1];
    values = new double[Size1];

    size_t currentIndice = 0;
    ptr[0] = 0;
    for(size_t i = 0; i < A.Size1; i++){
        ptr[i+1] = ptr[i];
        vecDiag[i] = 0.0;
        for(size_t r = A.CPU_RowIndices[i]; r < A.CPU_RowIndices[i+1]; r++){
            if(A.CPU_Columns[r] == i){
                indices[currentIndice] = A.CPU_Columns[r];
                vecDiag[i] = values[currentIndice] = 1.0/A.CPU_Values[r];
                ptr[i+1]++;
                currentIndice++;
                break;
            }
        }
    }
    NNZ = ptr[Size1];
}
/** Aggregation of A to generate the colored graph **/
template <class I>
I standardAggregation(const I n_row,
                       const I Ap[],
                       const I Aj[],
                             I  x[])
{
    // Bj[n] == -1 means i-th node has not been aggregated
    //std::fill(x, x + n_row, 0);
    for(long q = 0; q < n_row; q++)
		x[q] = 0;

    I next_aggregate = 1; // number of aggregates + 1

    //Pass #1
    for(I i = 0; i < n_row; i++){
        if(x[i]){ continue; } //already marked

        const I row_start = Ap[i];
        const I row_end   = Ap[i+1];

        //Determine whether all neighbors of this node are free (not already aggregates)
        bool has_aggregated_neighbors = false;
        bool has_neighbors            = false;
        for(I jj = row_start; jj < row_end; jj++){
            const I j = Aj[jj];
            if( i != j ){
                has_neighbors = true;
                if( x[j] ){
                    has_aggregated_neighbors = true;
                    break;
                }
            }
        }

        if(!has_neighbors){
            //isolated node, do not aggregate
            x[i] = -n_row;
        }
        else if (!has_aggregated_neighbors){
            //Make an aggregate out of this node and its neighbors
            x[i] = next_aggregate;
            for(I jj = row_start; jj < row_end; jj++){
                x[Aj[jj]] = next_aggregate;
            }
            next_aggregate++;
        }
    }


    //Pass #2
    // Add unaggregated nodes to any neighboring aggregate
    for(I i = 0; i < n_row; i++){
        if(x[i]){ continue; } //already marked

        for(I jj = Ap[i]; jj < Ap[i+1]; jj++){
            const I j = Aj[jj];

            const I xj = x[j];
            if(xj > 0){
                x[i] = -xj;
                break;
            }
        }
    }

    next_aggregate--;

    //Pass #3
    for(I i = 0; i < n_row; i++){
        const I xi = x[i];

        if(xi != 0){
            // node i has been aggregated
            if(xi > 0)
                x[i] = xi - 1;
            else if(xi == -n_row)
                x[i] = -1;
            else
                x[i] = -xi - 1;
            continue;
        }

        // node i has not been aggregated
        const I row_start = Ap[i];
        const I row_end   = Ap[i+1];

        x[i] = next_aggregate;

        for(I jj = row_start; jj < row_end; jj++){
            const I j = Aj[jj];

            if(x[j] == 0){ //unmarked neighbors
                x[j] = next_aggregate;
            }
        }
        next_aggregate++;
    }

    return next_aggregate; //number of aggregates
}

/** This function will create Ptent from a given A **/
void createPTent(const GPUCSRMatrix &A, size_t &NNZ_ptent, size_t &Size1_ptent, size_t &Size2_ptent, size_t *&ptr_ptent, size_t *&indices_ptent, double *&values_ptent){
    long *x;
    long size = A.Size1;
    x = new long[size];
    size_t maxColumn = standardAggregation(size, (long*)A.CPU_RowIndices, (long*)A.CPU_Columns, x);

    ptr_ptent = new size_t[size+1];
    indices_ptent = new size_t[size];
    values_ptent = new double[size];

    long* auxValues = new long[size];
    //ini auxValues to 0
    for(long i = 0; i < size; i++){
        auxValues[i] = 0;
    }
    //calculate each value group num
    for(long i = 0; i < size; i++){
        auxValues[x[i]]++;
    }

    //assign indices and right values
    ptr_ptent[0] = 0;
    for(long i = 0; i < size; i++){
        size_t j = x[i];
        ptr_ptent[i+1] = ptr_ptent[i] + 1;
        indices_ptent[i] = j;
        values_ptent[i] = 1.0/sqrt((double)auxValues[j]);
    }

    Size2_ptent = maxColumn;
    Size1_ptent = size;
    NNZ_ptent = size;

    delete[] x;
    delete[] auxValues;
}

void csr_tocsc(const size_t n_row,
	           const size_t n_col,
	           const size_t Ap[],
	           const size_t Aj[],
	           const double Ax[],
	                 size_t*& Bp,
	                 size_t* Bi,
	                 double* Bx)
{
    const size_t nnz = Ap[n_row];
    Bp = new size_t[n_col+1];
    //compute number of non-zero entries per column of A
    //std::fill(Bp, Bp + n_col, 0);
	for(size_t q = 0; q < n_col; q++)
		Bp[q] = 0;

    for (size_t n = 0; n < nnz; n++){
        Bp[Aj[n]]++;
    }

    //cumsum the nnz per column to get Bp[]
    for(size_t col = 0, cumsum = 0; col < n_col; col++){
        size_t temp  = Bp[col];
        Bp[col] = cumsum;
        cumsum += temp;
    }
    Bp[n_col] = nnz;

    for(size_t row = 0; row < n_row; row++){
        for(size_t jj = Ap[row]; jj < Ap[row+1]; jj++){
            size_t col  = Aj[jj];
            size_t dest = Bp[col];

            Bi[dest] = row;
            Bx[dest] = Ax[jj];

            Bp[col]++;
        }
    }

    for(size_t col = 0, last = 0; col <= n_col; col++){
        size_t temp  = Bp[col];
        Bp[col] = last;
        last    = temp;
    }
}

double calculateNorm(_Vector &b){
    double finalNum = 0.0;
    for(size_t i = 0; i < b.Size; i++){
        finalNum += pow(b.CPU_Values[i], 2);
    }
    return sqrt(finalNum);
}

double calculateNorm_GPU(GPUVector &b){
    return hipblasDnrm2(b.Size, b.GPU_Values, 1);
}

double checkResidual(const GPUVector &u, const GPUVector &b, const GPUCSRMatrix &A){
    /** Au **/
    GPUVector auxAU(A.Size1);
    auxAU.GPU_Allocate();
    dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
    
#ifdef USE_TEXTURE_CACHING

	// Bind the texture memory to X
	Bind_X(u.GPU_Values);

#endif
    
    GPUGPUCSRMatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, u.GPU_Values, auxAU.GPU_Values);
    if(!CUDA_Success(hipDeviceSynchronize())){
        printf("Error in the line 1208");
    }
    
#ifdef USE_TEXTURE_CACHING

	// Unbind the texture memory
	Unbind_X();

#endif

    /** b - AU **/
    GPUVector auxABU(auxAU.Size);
    auxABU.GPU_Allocate();
    Grid = Build_Grid(A.Size1, BLOCK_SIZE);
    subVectorVector <<< Grid, BLOCK_SIZE >>>(b.GPU_Values, auxAU.GPU_Values,
            auxABU.GPU_Values, auxABU.Size);
    if(!CUDA_Success(hipDeviceSynchronize())){
        printf("Error in the line 1219");
    }
    double finalNum = calculateNorm_GPU(auxABU);

    return finalNum;
}

bool checkConvergence(const GPUVector& u, const GPUVector& b, const GPUCSRMatrix& A, const double lastResidual, const double threshold){
    double newResidual = checkResidual(u, b, A);

    if(newResidual/lastResidual < threshold){
        return true;
    }
    return false;

}

void computeDenseMatrix(const GPUCSRMatrix& A, double *& vec){
    vec = new double[A.Size2 * A.Size1];
    size_t pointer = 0;
    size_t currentIndice = 0;
    for(size_t i = 0; i < A.Size1; i++){
        size_t nonZeros = A.CPU_RowIndices[i+1] - A.CPU_RowIndices[i];
        size_t columnPointer = 0;
        for(size_t j = 0; j < A.Size2; j++){
            if(nonZeros > 0  && A.CPU_Columns[currentIndice] == columnPointer){
                vec[pointer] = (double)A.CPU_Values[currentIndice];
                currentIndice++;
                nonZeros--;
            }else{
                vec[pointer] = 0.0;
            }
            pointer++;
            columnPointer++;
        }
    }
}

void subIdentityMatrix_cpu(const GPUCSRMatrix& A, size_t* CPU_RowIndices, size_t* CPU_Columns, double* CPU_Values){
    size_t currentIndex = 0;
    CPU_RowIndices[0] = 0;
    for(size_t i = 0; i < A.Size1; i++){
        bool haveDiagonal = false;
        size_t numElems = 0;
        long lastIndex = -1;
        for(size_t r = A.CPU_RowIndices[i]; r < A.CPU_RowIndices[i+1]; r++){
            lastIndex = A.CPU_Columns[r];
            if(A.CPU_Columns[r] > i && !haveDiagonal){
                CPU_Columns[currentIndex] = i;
                CPU_Values[currentIndex] = 1.0;
                currentIndex++;
                haveDiagonal = true;
                numElems++;
            }
            if(A.CPU_Columns[r] == i){
                haveDiagonal = true;
                if(A.CPU_Values[r] != 1.0){
                    CPU_Columns[currentIndex] = A.CPU_Columns[r];
                    CPU_Values[currentIndex] = 1.0 - A.CPU_Values[r];
                    currentIndex++;
                    numElems++;
                }
            }else{
                CPU_Columns[currentIndex] = A.CPU_Columns[r];
                CPU_Values[currentIndex] = -A.CPU_Values[r];
                currentIndex++;
                numElems++;
            }
        }
        if(lastIndex < (long)i){
            CPU_Columns[currentIndex] = i;
            CPU_Values[currentIndex] = 1.0;
            currentIndex++;
            numElems++;
        }
        CPU_RowIndices[i+1] = CPU_RowIndices[i] + numElems;
    }
}

double eigVals(_Vector& H, size_t finalIters){
    LaGenMatDouble A( H.CPU_Values, finalIters, finalIters);
    LaVectorDouble eigvals_real(finalIters);
    LaVectorDouble eigvals_imag(finalIters);
    LaGenMatDouble VR(finalIters, finalIters);

    LaEigSolve(A, eigvals_real,
		eigvals_imag, VR);
    double *real = eigvals_real.addr();
    double *imag = eigvals_imag.addr();

    double max = real[0];
    if(max < 0)
        max = -max;
    for(size_t i = 1; i < finalIters; i++){
        if(real[i] < 0)
            real[i] = -real[i];
        if(real[i] > max)
            max = real[i];
    }
    return max;
}

double roh(const GPUCSRMatrix& A, size_t iter){
    double threshold = 1e-10;

    size_t maxIter;
    maxIter = A.Size2 < iter ?  A.Size2 : iter;

    _Vector *V = new _Vector[maxIter+1];

    V[0].Size = A.Size2;
    V[0].CPU_Values = new double[V[0].Size];
    srand(0);
    for(size_t i = 0; i < V[0].Size; i++){
        V[0].CPU_Values[i] = (double)(((int)rand())%100000000)/100000000.0;

    }
    double v0Norm = calculateNorm(V[0]);
    for(size_t i = 0; i < V[0].Size; i++){
        V[0].CPU_Values[i] /= v0Norm;
    }
    mallocAndCopyMem(V[0].CPU_Values, V[0].GPU_Values, V[0].Size);
    //delete[] V[0].CPU_Values;

    _Vector H;
    H.Size = (maxIter+1) * (maxIter+1);
    H.CPU_Values = new double[H.Size];
    for(size_t q = 0; q < H.Size; q++){
        H.CPU_Values[q] = 0.0;
    }
    size_t numCurrentV = 1;

    size_t j;

    for(j = 0; j < maxIter; j++){
        V[numCurrentV].Size = A.Size1;
        malloc_(V[numCurrentV].GPU_Values, V[numCurrentV].Size);
        V[numCurrentV].CPU_Values = new double[V[numCurrentV].Size];
        dim3 grid = Build_Grid(V[numCurrentV].Size, BLOCK_SIZE);
        
#ifdef USE_TEXTURE_CACHING

		// Bind the texture memory to X
		Bind_X(V[numCurrentV-1].GPU_Values);

#endif
        
        GPUGPUCSRMatrixVectorMultiply_CSR_Kernel <<< grid, BLOCK_SIZE >>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, V[numCurrentV-1].GPU_Values, V[numCurrentV].GPU_Values);
        if(!CUDA_Success(hipDeviceSynchronize())){
            printf("Error in line 1379\n");
        }

 #ifdef USE_TEXTURE_CACHING

		// Unbind the texture memory
		Unbind_X();

#endif

        copyMem(V[numCurrentV].GPU_Values, V[numCurrentV].CPU_Values, V[numCurrentV].Size, 1);

        _Vector auxVec;
        auxVec.Size = V[numCurrentV].Size;
        malloc_(auxVec.GPU_Values, auxVec.Size);

        grid = Build_Grid(V[numCurrentV-1].Size, BLOCK_SIZE);
        for(size_t i = 0; i < numCurrentV; i++){
            size_t matrixIndice = (i*(maxIter+1))+j;
            double auxVal = H.CPU_Values[matrixIndice] = hipblasDdot(V[i].Size, V[i].GPU_Values, 1, V[numCurrentV].GPU_Values, 1);
            subVectorConstantValue <<< grid, BLOCK_SIZE>>> (V[numCurrentV].GPU_Values, H.CPU_Values[matrixIndice], V[i].GPU_Values, V[numCurrentV].Size);
            if(!CUDA_Success(hipDeviceSynchronize())){
                printf("Error in line 1394\n");
            }
        }
        size_t matrixIndice = ((j+1) * (maxIter+1)) + j;
        copyMem(V[numCurrentV].GPU_Values, V[numCurrentV].CPU_Values, V[numCurrentV].Size, 1);
        //copyMem(H.GPU_Values, H.CPU_Values, H.Size, 1);
        H.CPU_Values[matrixIndice] = calculateNorm(V[numCurrentV]);
        if(H.CPU_Values[matrixIndice] < threshold)
            break;
        //copyMem(H.CPU_Values, H.GPU_Values, H.Size, 0);

        divideVectorConstantValue <<< grid, BLOCK_SIZE >>> (V[numCurrentV].GPU_Values, H.CPU_Values[matrixIndice], V[numCurrentV].Size);
        if(!CUDA_Success(hipDeviceSynchronize())){
            printf("Error in line 1407\n");
        }
        CUDA_CHECK(hipFree(auxVec.GPU_Values));

        numCurrentV++;

    }
    for(size_t i = 0; i < numCurrentV; i++){
        delete[] V[i].CPU_Values;
        CUDA_CHECK(hipFree(V[i].GPU_Values));
    }
    delete[] V;

    double max = eigVals(H, maxIter+1);

    delete[] H.CPU_Values;

    return max;
}

/**     Memory management functions     **/
/*template <class Q>
void mallocAndCopyMem(Q*& CPU, Q*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(Q)));
    CUDA_CHECK(hipMemcpy(GPU, CPU, size*sizeof(Q), hipMemcpyHostToDevice));
}*/

void mallocAndCopyMem(double*& CPU, double*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(double)));
    CUDA_CHECK(hipMemcpy(GPU, CPU, size*sizeof(double), hipMemcpyHostToDevice));
}

void mallocAndCopyMem(size_t*& CPU, size_t*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(size_t)));
    CUDA_CHECK(hipMemcpy(GPU, CPU, size*sizeof(size_t), hipMemcpyHostToDevice));
}

/*template <class Q>
void malloc_(Q*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(Q)));
}*/

void malloc_(double*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(double)));
}

void malloc_(size_t*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(size_t)));
}

template <class Q>
void copyMem(Q*& source, Q*& destiny, size_t size, unsigned short way){
    switch(way){
        case 0:
            CUDA_CHECK(hipMemcpy(destiny, source, size*sizeof(Q), hipMemcpyHostToDevice));
            break;
        case 1:
            CUDA_CHECK(hipMemcpy(destiny, source, size*sizeof(Q), hipMemcpyDeviceToHost));
            break;
        case 2:
            CUDA_CHECK(hipMemcpy(destiny, source, size*sizeof(Q), hipMemcpyDeviceToDevice));
            break;
    }
}

void deletingStuff(size_t* stuff){
    CUDA_CHECK(hipFree(stuff));
}

void deletingStuff(double* stuff){
    CUDA_CHECK(hipFree(stuff));
}

void GPUGPUVectorMultiply(double* sourceVec, double* destinyVec, size_t N){
	dim3 Grid = Build_Grid(N, BLOCK_SIZE);
	GPUGPUVectorVectorMultiplyElementWise_Kernel <<<Grid, BLOCK_SIZE>>> (N, sourceVec, destinyVec, destinyVec);
	GPUSparse::CUDA_Success(hipDeviceSynchronize());
}





}

}

//
// Compilation command
// make
