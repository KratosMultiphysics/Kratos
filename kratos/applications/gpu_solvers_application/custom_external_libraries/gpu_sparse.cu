#include "hip/hip_runtime.h"
/*
==============================================================================
Kratos
A General Purpose Software for Multi-Physics Finite Element Analysis
Version 1.0 (Released on march 05, 2007).

Copyright 2007
Pooyan Dadvand, Riccardo Rossi, Farshid Mossaiby
pooyan@cimne.upc.edu
rrossi@cimne.upc.edu
mossaiby@yahoo.com
CIMNE (International Center for Numerical Methods in Engineering),
Gran Capita' s/n, 08034 Barcelona, Spain

Permission is hereby granted, free  of charge, to any person obtaining
a  copy  of this  software  and  associated  documentation files  (the
"Software"), to  deal in  the Software without  restriction, including
without limitation  the rights to  use, copy, modify,  merge, publish,
distribute,  sublicense and/or  sell copies  of the  Software,  and to
permit persons to whom the Software  is furnished to do so, subject to
the following condition:

Distribution of this code for  any  commercial purpose  is permissible
ONLY BY DIRECT ARRANGEMENT WITH THE COPYRIGHT OWNER.

The  above  copyright  notice  and  this permission  notice  shall  be
included in all copies or substantial portions of the Software.

THE  SOFTWARE IS  PROVIDED  "AS  IS", WITHOUT  WARRANTY  OF ANY  KIND,
EXPRESS OR  IMPLIED, INCLUDING  BUT NOT LIMITED  TO THE  WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT  SHALL THE AUTHORS OR COPYRIGHT HOLDERS  BE LIABLE FOR ANY
CLAIM, DAMAGES OR  OTHER LIABILITY, WHETHER IN AN  ACTION OF CONTRACT,
TORT  OR OTHERWISE, ARISING  FROM, OUT  OF OR  IN CONNECTION  WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

==============================================================================
*/

//
// Sparse matrix and vector operations on GPU

//
// Notes!
//
//   * In case of VectorVectorMultiply and VectorNorm2, cuBlas library has been used, as it is much faster than a code without proper optimizations.
//
//   * In case of VectorScaleAndAdd, as there is no direct way of using cuBlas for this, we had to write our own kernel. It was 2 or 3 times faster
//     than using cuBlas indirectly.
//
//   * For compilation command, simply issue a 'make' command.
//
//   * Removed templates to be able to link to Kratos.
//
//   * Modified GPUCSRMatrix to keep no. of non-zeros per row constant and equal to HALF_WARP_SIZE (16) and used same no. of threads to multiply a row

// More notes!
//
//	* Checks for error in Bind_X() and Unbind_X() have been removed for more consistent error checking after kernel calls; can they fail?
//
//	* Added an optional parameter to GPUCSRMatrix constructor, so that user can optionally avoid making non-zeros in a row a multiple of HALF_WARP_SIZE (16)
//
//	* Added a make file; in the command line use emu=1 for emulation mode and dbg=1 for a debug version

// Includes, system

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdio>

// Includes, project

#include "gpu_sparse.h"
#include "gpu_sparse_utils.h"
#include "gpu_sparse_kernels.h"

//Includes, preconditioner
#include <lapackd.h>
#include <laslv.h>
#include <gmd.h>
#include <gmf.h>
#include <vector>

namespace Kratos
{

namespace GPUSparse
{

// GPUVector class definition

GPUVector::GPUVector(size_t _Size, double *_CPU_Values): Size(_Size), CPU_Values(_CPU_Values), GPU_Values(0), Allocated(false)
{
	// Nothing to do!
}

GPUVector::GPUVector(size_t _Size): Size(_Size), CPU_Values(0), GPU_Values(0), Allocated(false)
{
	// Nothing to do!
}

GPUVector::~GPUVector()
{
	if (Allocated)
		GPU_Free();
}

bool GPUVector::GPU_Allocate()
{
	if (Allocated)
		return false;

	Allocated = true;

	return CUDA_Success(hipMalloc(reinterpret_cast <void **> (&GPU_Values), Size * sizeof(double)));
}

bool GPUVector::GPU_Free()
{
	if (!Allocated)
		return false;

	Allocated = false;

	return CUDA_Success(hipFree(GPU_Values));
}

bool GPUVector::Copy(CopyDirection Direction)
{
	if (!Allocated)
		return false;

	switch (Direction)
	{
		case CPU_GPU:

			return CUDA_Success(hipMemcpy(GPU_Values, CPU_Values, Size * sizeof(double), hipMemcpyHostToDevice));

		case GPU_CPU:

			return CUDA_Success(hipMemcpy(CPU_Values, GPU_Values, Size * sizeof(double), hipMemcpyDeviceToHost));
	}

	// We should never get here!
	return false;
}

bool GPUVector::CopyFromGPU(GPUVector &V)
{
	if (V.Size != Size || !Allocated || !V.Allocated)
		return false;

	return CUDA_Success(hipMemcpy(GPU_Values, V.GPU_Values, Size * sizeof(double), hipMemcpyDeviceToDevice));
}

// GPUCSRMatrix class definition

GPUCSRMatrix::GPUCSRMatrix(size_t _NNZ, size_t _Size1, size_t _Size2, size_t *_CPU_Columns, size_t *_CPU_RowIndices, double *_CPU_Values, bool _NZMultiple16): NNZ(_NNZ), Size1(_Size1), Size2(_Size2), CPU_Columns(0), CPU_RowIndices(0), CPU_Values(0), GPU_Columns(0), GPU_RowIndices(0), GPU_Values(0), Allocated(false)

{

	if (_NZMultiple16)
	{

		NNZ = 0;
		// Temporary RowIndices vector
		size_t *Temp_CPU_RowIndices = new size_t[Size1 + 1];

		Temp_CPU_RowIndices[0] = 0;

		// Find out how many non-zeros are needed to pad all rows to 16 while building the RowIndices
		for (size_t i = 0; i < Size1; i++)
		{
			size_t NZ = _CPU_RowIndices[i + 1] - _CPU_RowIndices[i];

			size_t R = NZ & HALF_WARP_SIZE_MASK;

			if (R != 0)
				NZ += HALF_WARP_SIZE - R;

			NNZ += NZ;
			Temp_CPU_RowIndices[i + 1] = Temp_CPU_RowIndices[i] + NZ;
		}

		// Allocate CPU memory for CSR structure using only one chunk of page-locked memory to speed up data transfer between CPU and GPU
		void *CSR_Data;


		if (!CUDA_Success(hipHostMalloc(&CSR_Data, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t))))  // TODO: What should be done?!
			CSR_Data = 0;

		// We are sure that using this order, the memory alignment conditions will be satisfied as NNZ is a multiple of HALF_WARP_SIZE (16) and sizeof(double) = 8
		// TODO: Check this!
/*		CPU_Values = reinterpret_cast <double *> (CSR_Data);  // Size: NNZ * sizeof(double)
		CPU_Columns = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * sizeof(double));  // Size: NNZ * sizeof(size_t)
		CPU_RowIndices = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * (sizeof(double) + sizeof(size_t)));  // Size: (Size1 + 1) * sizeof(size_t)*/
CPU_Values = (double*)CSR_Data;
CPU_Columns = (size_t*)((double*) CSR_Data + NNZ);
CPU_RowIndices = (size_t*)((size_t*)((double*) CSR_Data + NNZ) + NNZ);
		// Move temporary data

		memcpy(CPU_RowIndices, Temp_CPU_RowIndices, (Size1 + 1) * sizeof(size_t));


		delete[] Temp_CPU_RowIndices;

		// Build ECSR structure from given CSR
		for (size_t i = 0; i < Size1; i++)
		{
			size_t _Start = _CPU_RowIndices[i], Start = CPU_RowIndices[i];

			for (size_t j = 0; j < _CPU_RowIndices[i + 1] - _CPU_RowIndices[i]; j++)
			{
				CPU_Columns[Start + j] = _CPU_Columns[_Start + j];
				CPU_Values[Start + j] = _CPU_Values[_Start + j];
			}

			size_t LastCol = _CPU_Columns[_CPU_RowIndices[i + 1] - 1];

			for (size_t j = _CPU_RowIndices[i + 1] - _CPU_RowIndices[i]; j < CPU_RowIndices[i + 1] - CPU_RowIndices[i]; j++)
			{
				CPU_Columns[Start + j] = LastCol;  // To maintain coalescing as much as possible
				CPU_Values[Start + j] = 0.00;
			}
		}
	}

	else
	{
		// Allocate CPU memory for CSR structure using only one chunk of page-locked memory to speed up data transfer between CPU and GPU
		void *CSR_Data;

		if (!CUDA_Success(hipHostMalloc(&CSR_Data, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t))))  // TODO: What should be done?!
			CSR_Data = 0;

		// We are sure that using this order, the memory alignment conditions will be satisfied as NNZ is a multiple of HALF_WARP_SIZE (16) and sizeof(double) = 8
		// TODO: Check this!
/*		CPU_Values = reinterpret_cast <double *> (CSR_Data);  // Size: NNZ * sizeof(double)
		CPU_Columns = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * sizeof(double));  // Size: NNZ * sizeof(size_t)
		CPU_RowIndices = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * (sizeof(double) + sizeof(size_t)));  // Size: (Size1 + 1) * sizeof(size_t)*/
CPU_Values = (double*)CSR_Data;
CPU_Columns = (size_t*)((double*) CSR_Data + NNZ);
CPU_RowIndices = (size_t*)((size_t*)((double*) CSR_Data + NNZ) + NNZ);

		// Move temporary data
		memcpy(CPU_Values, _CPU_Values, NNZ * sizeof(double));
		memcpy(CPU_Columns, _CPU_Columns, NNZ * sizeof(size_t));
		memcpy(CPU_RowIndices, _CPU_RowIndices, (Size1 + 1) * sizeof(size_t));
	}
}

GPUCSRMatrix::GPUCSRMatrix(size_t _NNZ, size_t _Size1, size_t _Size2): NNZ(_NNZ), Size1(_Size1), Size2(_Size2), CPU_Columns(0), CPU_RowIndices(0), CPU_Values(0), GPU_Columns(0), GPU_RowIndices(0), GPU_Values(0), Allocated(false)
{
	// Nothing to do!
}

GPUCSRMatrix::~GPUCSRMatrix()
{
	// Free CSR data; as it is allocated in one chunk of memory, we need only to free the begining address
	hipHostFree(CPU_Values);

	if (Allocated)
		GPU_Free();
}

bool GPUCSRMatrix::GPU_Allocate()
{
	if (Allocated)
		return false;

	Allocated = true;

	// Allocate GPU memory for CSR structure using only one chunk of memory to speed up data transfer between CPU and GPU

	void *CSR_Data;


	if (CUDA_Success(hipMalloc(&CSR_Data, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t))))
	{
/*		GPU_Values = reinterpret_cast <double *> (CSR_Data);  // Size: NNZ * sizeof(double)
		GPU_Columns = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * sizeof(double));  // Size: NNZ * sizeof(size_t)
		GPU_RowIndices = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * (sizeof(double) + sizeof(size_t)));  // Size: (Size1 + 1) * sizeof(size_t)*/
GPU_Values = (double*)CSR_Data;
		GPU_Columns = (size_t*)((double*) CSR_Data + NNZ);
		GPU_RowIndices = (size_t*)((size_t*)((double*) CSR_Data + NNZ) + NNZ);

		return true;
	}

	else
		return false;
}

bool GPUCSRMatrix::GPU_Free()
{
	if (!Allocated)
		return false;

	Allocated = false;
	
	return CUDA_Success(hipFree(GPU_Values));
}

bool GPUCSRMatrix::Copy(CopyDirection Direction, bool CopyValuesOnly)
{
	if (!Allocated)
		return false;

	switch (Direction)
	{
		case CPU_GPU:
			if (CopyValuesOnly)
				return CUDA_Success(hipMemcpy(GPU_Values, CPU_Values, NNZ * sizeof(double), hipMemcpyHostToDevice));
			else
				return CUDA_Success(hipMemcpy(GPU_Values, CPU_Values, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t), hipMemcpyHostToDevice));

		case GPU_CPU: 
			if (CopyValuesOnly)
				return CUDA_Success(hipMemcpy(CPU_Values, GPU_Values, NNZ * sizeof(double), hipMemcpyDeviceToHost));
			else
				return CUDA_Success(hipMemcpy(CPU_Values, GPU_Values, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t), hipMemcpyDeviceToHost));

	}

	// We should never get here!
	return false;
}

bool GPUCSRMatrix::CopyFromGPU(GPUCSRMatrix &M, bool CopyStructure, bool CopyValues)
{
	if (M.Size1 != Size1 || M.Size2 != Size2 || M.NNZ != NNZ || !Allocated || !M.Allocated)
		return false;

	size_t CopyLength;
	void *CopyFrom, *CopyTo;

	if (CopyStructure && CopyValues)
	{
		CopyTo = GPU_Values;
		CopyFrom = M.GPU_Values;
		CopyLength = NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t);
	}
	else if (CopyStructure && !CopyValues)
	{
		CopyTo = reinterpret_cast <void *> (reinterpret_cast <size_t> (GPU_Values) + NNZ * sizeof(double));
		CopyFrom = reinterpret_cast <void *> (reinterpret_cast <size_t> (M.GPU_Values) + NNZ * sizeof(double));
		CopyLength = (NNZ + Size1 + 1) * sizeof(size_t);
	}
	else if (!CopyStructure && CopyValues)
	{
		CopyTo = GPU_Values;
		CopyFrom = M.GPU_Values;
		CopyLength = NNZ * sizeof(double);
	}
	else if (!CopyStructure && !CopyValues)
	{
		CopyTo = 0;
		CopyFrom = 0;
		CopyLength = 0;
	}

	if (CopyLength != 0)
		return CUDA_Success(hipMemcpy(CopyTo, CopyFrom, CopyLength, hipMemcpyDeviceToDevice));
	else
		return true;
}

// Operations defined on GPUCSRMatrix and GPUVector

//
// CPU_MatrixVectorMultiply
// Matrix-Vector multiply on CPU

bool CPU_MatrixVectorMultiply(GPUCSRMatrix &A, GPUVector &X, GPUVector &Y)
{
	// Primary checks
	if (A.Size2 != X.Size || X.Size != Y.Size)
		return false;

	for (size_t i = 0; i < A.Size1; i++)
	{
		double YI = static_cast <double> (0);

		for (size_t j = A.CPU_RowIndices[i]; j < A.CPU_RowIndices[i + 1]; j++)
			YI += A.CPU_Values[j] * X.CPU_Values[A.CPU_Columns[j]];

		Y.CPU_Values[i] = YI;
	}

	return true;
}

//
// GPU_MatrixVectorMultiply
// Matrix-Vector multiply on GPU

bool GPU_MatrixVectorMultiply(GPUCSRMatrix &A, GPUVector &X, GPUVector &Y)
{
	// Primary checks
	if (A.Size2 != X.Size || X.Size != Y.Size || !X.Allocated || !Y.Allocated)
		return false;

#ifdef USE_TEXTURE_CACHING

	// Bind the texture memory to X
	Bind_X(X.GPU_Values);

#endif


	bool UseVectorizedVersion = (A.NNZ / A.Size2) > 10;	// From nVidia forum

	if (UseVectorizedVersion)
	{
		dim3 Grid = Build_Grid(A.Size1 *  HALF_WARP_SIZE, BLOCK_SIZE);
		GPU_MatrixVectorMultiply_CSR_Vectorized_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, X.GPU_Values, Y.GPU_Values);

		if (!GPUSparse::CUDA_Success(hipGetLastError()))
			return false;
	}

	else

	{
		dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
		GPU_MatrixVectorMultiply_CSR_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, X.GPU_Values, Y.GPU_Values);

		if (!GPUSparse::CUDA_Success(hipGetLastError()))
			return false;
	}

#ifdef USE_TEXTURE_CACHING

	// Unbind the texture memory
	Unbind_X();
	}

#endif

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_MatrixGetDiagonals
// Extract the diagonal elements of a matrix into a vector on CPU

bool CPU_MatrixGetDiagonals(GPUCSRMatrix &A, GPUVector &X)
{
	// Primary checks
	if (A.Size1 != A.Size2 || A.Size2 != X.Size)
		return false;

	for (size_t i = 0; i < A.Size1; i++)
	{
		X.CPU_Values[i] = static_cast <double> (0);

		for (size_t j = A.CPU_RowIndices[i]; j < A.CPU_RowIndices[i + 1]; j++)
			if (A.CPU_Columns[j] == i)
				X.CPU_Values[i] = A.CPU_Values[j];
	}

	return true;
}

//
// GPU_MatrixGetDiagonals
// Extract the diagonal elements of a matrix into a vector on GPU

bool GPU_MatrixGetDiagonals(GPUCSRMatrix &A, GPUVector &X)
{
	// Primary checks
	if (A.Size1 != A.Size2 || A.Size2 != X.Size || !A.Allocated || !X.Allocated)
		return false;

	dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	GPU_MatrixGetDiagonals_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, X.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_MatrixMatrixDiagonalMultiply
// Multiply a digonal matrix specified with a vector with a matrix on CPU

bool CPU_MatrixMatrixDiagonalMultiply(GPUVector &X, GPUCSRMatrix &A)
{
	// Primary checks
	if (X.Size != A.Size1)
		return false;

	for (size_t i = 0; i < X.Size; i++)
	{
		double t = X.CPU_Values[i];

		for (size_t j = A.CPU_RowIndices[i]; j < A.CPU_RowIndices[i + 1]; j++)
				A.CPU_Values[j] *= t;
	}

	return true;
}

//
// GPU_MatrixMatrixDiagonalMultiply
// Multiply a digonal matrix specified with a vector with a matrix on GPU

bool GPU_MatrixMatrixDiagonalMultiply(GPUVector &X, GPUCSRMatrix &A)
{
	// Primary checks
	if (X.Size != A.Size1 || !X.Allocated || !A.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);
	GPU_MatrixMatrixDiagonalMultiply_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, X.GPU_Values, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_VectorPrepareDiagonalPreconditionerValues
// Prepare diagonal values of the matrix for Diagonal Preconditioner on CPU

bool CPU_VectorPrepareDiagonalPreconditionerValues(GPUVector &X)
{
	for (size_t i = 0; i < X.Size; i++)
		if (X.CPU_Values[i] == 0.00)
			X.CPU_Values[i] = 1.00;
		else
			X.CPU_Values[i] = 1.00 / X.CPU_Values[i];
//			X.CPU_Values[i] = 1.00 / sqrt(abs(X.CPU_Values[i]));

	return true;
}

//
// GPU_VectorPrepareDiagonalPreconditionerValues
// Prepare diagonal values of the matrix for Diagonal Preconditioner on GPU

bool GPU_VectorPrepareDiagonalPreconditionerValues(GPUVector &X)
{
	// Primary check
	if (!X.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);
	GPU_VectorPrepareDiagonalPreconditionerValues_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, X.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// GPU_PrepareSPAIPreconditioner
// Prepare SPAI preconditioner on GPU

bool GPU_PrepareSPAIPreconditioner(GPUCSRMatrix &A, GPUCSRMatrix &M)
{
	// Primary checks
	if (A.Size1 != M.Size1 || A.Size2 != M.Size2 || A.NNZ != M.NNZ || !A.Allocated || !M.Allocated)
		return false;

	dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	GPU_SPAIPreconditioner_CSR_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, M.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_VectorVectorMultiply
// Vector-Vector multiply on CPU

bool CPU_VectorVectorMultiply(GPUVector &X, GPUVector &Y, double &Result)
{
	// Primary check
	if (X.Size != Y.Size)
		return false;

	Result = static_cast <double> (0);

	for (size_t i = 0; i < X.Size; i++)
		Result += X.CPU_Values[i] * Y.CPU_Values[i];

	return true;
}

//
// GPU_VectorVectorMultiply
// Vector-Vector multiply on GPU

bool GPU_VectorVectorMultiply(GPUVector &X, GPUVector &Y, double &Result)
{
	// Primary check
	if (X.Size != Y.Size || !X.Allocated || !Y.Allocated)
		return false;

	Result = hipblasDdot(X.Size, X.GPU_Values, 1, Y.GPU_Values, 1);

	return CUBLAS_Success(cublasGetError());
}

//
// CPU_VectorVectorMultiplyElementWise
// Vector-Vector element-wise multiply on CPU

bool CPU_VectorVectorMultiplyElementWise(GPUVector &X, GPUVector &Y,  GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size)
		return false;

	for (size_t i = 0; i < X.Size; i++)
		Z.CPU_Values[i] = X.CPU_Values[i] * Y.CPU_Values[i];

	return true;
}

//
// GPU_VectorVectorMultiplyElementWise
// Vector-Vector element-wise multiply on GPU

bool GPU_VectorVectorMultiplyElementWise(GPUVector &X, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size || !X.Allocated || !Y.Allocated || !Z.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	GPU_VectorVectorMultiplyElementWise_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, X.GPU_Values, Y.GPU_Values, Z.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_VectorNorm2
// Vector norm 2 on CPU

bool CPU_VectorNorm2(GPUVector &X, double &Result)
{
	Result = static_cast <double> (0);

	for (size_t i = 0; i < X.Size; i++)
		Result += X.CPU_Values[i] * X.CPU_Values[i];

	Result = sqrt(Result);

	return true;
}

//
// GPU_VectorNorm2
// Vector norm 2 on GPU

bool GPU_VectorNorm2(GPUVector &X, double &Result)
{
	// Primary check
	if (!X.Allocated)
		return false;

	Result = hipblasDnrm2(X.Size, X.GPU_Values, 1);

	return CUBLAS_Success(cublasGetError());
}

//
// CPU_VectorScaleAndAdd
// Vector scale-and-add on CPU

// Variant 1: Z = A * X + B * Y

bool CPU_VectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size)
		return false;

	for (size_t i = 0; i < X.Size; i++)
		Z.CPU_Values[i] = A * X.CPU_Values[i] + B * Y.CPU_Values[i];

	return true;
}

// Variant 2: Y = A * X + B * Y

bool CPU_VectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y)
{
	// Primary check
	if (X.Size != Y.Size)
		return false;

	for (size_t i = 0; i < X.Size; i++)
		Y.CPU_Values[i] = A * X.CPU_Values[i] + B * Y.CPU_Values[i];

	return true;
}

//
// GPU_VectorScaleAndAdd
// Vector scale-and-add on GPU

// Variant 1: Z = A * X + B * Y

bool GPU_VectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size || !X.Allocated || !Y.Allocated || !Z.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	if (A == 1.00)
	{
	//printf("A = 1.0\n");
		if (B == 1.00)
			GPU_VectorScaleAndAdd_1_A_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_1_B_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else
			GPU_VectorScaleAndAdd_1_E_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}

	else if (A == -1.00)
	{
	//printf("A = -1.0\n");
		if (B == 1.00)
			GPU_VectorScaleAndAdd_1_C_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_1_D_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else
			GPU_VectorScaleAndAdd_1_F_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}

	else
	{
	//printf("B = 1.0\n");
		if (B == 1.00)
			GPU_VectorScaleAndAdd_1_G_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_1_H_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else
			GPU_VectorScaleAndAdd_1_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}

	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

// Variant 2: Y = A * X + B * Y

bool GPU_VectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y)
{
	// Primary check
	if (X.Size != Y.Size || !X.Allocated || !Y.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	if (A == 1.00)
	{
		if (B == 1.00)
			GPU_VectorScaleAndAdd_2_A_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_2_B_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else
			GPU_VectorScaleAndAdd_2_E_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);
	}

	else if (A == -1.00)
	{
		if (B == 1.00)
			GPU_VectorScaleAndAdd_2_C_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_2_D_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else
			GPU_VectorScaleAndAdd_2_F_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);
	}

	else
	{
		if (B == 1.00)
			GPU_VectorScaleAndAdd_2_G_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_2_H_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else
			GPU_VectorScaleAndAdd_2_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);
	}

	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

/*double GPU_dotProduct(size_t numElems, const double *firstVec, int incFirstVec,
    const double *secondVec, int incSecondVec){
    return hipblasDdot(numElems, firstVec, incFirstVec, secondVec, incSecondVec);
}*/


/** ADDED FUNCTIONS **/

void GPU_fillWithZeros(size_t numElems, double* gpuVec){
	dim3 grid = Build_Grid(numElems, BLOCK_SIZE);
	fillWithZeros <<< grid, BLOCK_SIZE >>> (gpuVec, numElems);
}

bool GPU_VectorScaleAndAdd_addingVersion(double A, GPUVector &X, double B, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size || !X.Allocated || !Y.Allocated || !Z.Allocated){
		//printf("Falla x la comprovació\n");		
		return false;
	}
		
	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	if (A == 1.00)
	{
	//printf("A = 1.0\n");
		if (B == 1.00)
			GPU_VectorScaleAndAdd_1_A_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

			
		else if (B == -1.00)
			GPU_VectorScaleAndAdd_1_B_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
				
		else
			GPU_VectorScaleAndAdd_1_E_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}
	
	else if (A == -1.00)
	{
	//printf("A = -1.0\n");
		if (B == 1.00)
			GPU_VectorScaleAndAdd_1_C_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
			
		else if (B == -1.00)
			GPU_VectorScaleAndAdd_1_D_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
			
		else
			GPU_VectorScaleAndAdd_1_F_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}
	
	else
	{
	//printf("B = 1.0\n");
		if (B == 1.00)
			GPU_VectorScaleAndAdd_1_G_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
			
		else if (B == -1.00)
			GPU_VectorScaleAndAdd_1_H_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
			
		else
			GPU_VectorScaleAndAdd_1_Kernel_addingVersion <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}
	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
	
}

/** temp variables for LU decomposition **/
LaVectorLongInt ipiv;

/** functions from scipy for mat-mat calculation **/
template <class I>
void csr_matmat_pass1(const I n_row,
                      const I n_col,
                      const I Ap[],
                      const I Aj[],
                      const I Bp[],
                      const I Bj[],
                            I Cp[]){
    //std::vector<I> mask(n_col,-1);
    int* mask = new int[n_col];
    for(size_t aux = 0; aux < n_col; aux++){
	mask[aux] = -1;
    }
    Cp[0] = 0;

    I nnz = 0;
    for(I i = 0; i < n_row; i++){
        for(I jj = Ap[i]; jj < Ap[i+1]; jj++){
            I j = Aj[jj];
            for(I kk = Bp[j]; kk < Bp[j+1]; kk++){
                I k = Bj[kk];
                if(mask[k] != (int)i){
                    mask[k] = (int)i;
                    nnz++;
                }
            }
        }
        Cp[i+1] = nnz;
    }
    delete[] mask;
}

template <class I, class T>
void csr_matmat_pass2(const I n_row,
      	              const I n_col,
      	              const I Ap[],
      	              const I Aj[],
      	              const T Ax[],
      	              const I Bp[],
      	              const I Bj[],
      	              const T Bx[],
      	                    I Cp[],
      	                    I Cj[],
      	                    T Cx[])
{
//    std::vector<I> next(n_col,-1);
 //   std::vector<T> sums(n_col, 0);

    int* next = new int[n_col];
    T* sums = new T[n_col];
    for(size_t aux = 0; aux < n_col; aux++){
	next[aux] = -1;
	sums[aux] = 0;
    }

    I nnz = 0;

    Cp[0] = 0;

    for(I i = 0; i < n_row; i++){
        int head   = -2;
        I length =  0;

        I jj_start = Ap[i];
        I jj_end   = Ap[i+1];
        for(I jj = jj_start; jj < jj_end; jj++){
            I j = Aj[jj];
            T v = Ax[jj];

            I kk_start = Bp[j];
            I kk_end   = Bp[j+1];
            for(I kk = kk_start; kk < kk_end; kk++){
                I k = Bj[kk];

                sums[k] += v*Bx[kk];

                if(next[k] == -1){
                    next[k] = head;
                    head = (int)k;
                    length++;
                }
            }
        }

        for(I jj = 0; jj < length; jj++){

            if(sums[head] != 0){
                Cj[nnz] = (size_t)head;
                Cx[nnz] = sums[head];
                nnz++;
            }

            I temp = (size_t)head;
            head = (I)next[head];

            next[temp] = -1; //clear arrays
            sums[temp] =  0;
        }

        Cp[i+1] = nnz;
    }

	delete[] sums;
	delete[] next;
}

/** maxLevels define the maxLevels of that execution
	G defines the diagonals of each lvl of A, created on previous step**/
void multilevel(_Matrix*& A, _Matrix*& P, _Matrix*& R, _Matrix*& G, _Vector& b, _Vector& u,
			unsigned short lvl, unsigned short maxLevels, size_t* preSweeps, size_t* postSweeps, bool assumeZeros)
{
    bool vectorized = (A[lvl].numNNZ / A[lvl].numCols) > 10;
    _Vector r;
    //calculateInstantVector(u, b, A[lvl], G[lvl]);
    if(lvl < maxLevels){
	//clock_t t1 = clock();
        if(assumeZeros) //we receive from the upper level a zero start vector
        {
            if(preSweeps[lvl] != 0)
            {
                //first iteration (does not require computation of residual
                dim3 Grid = Build_Grid(A[lvl].numRows, BLOCK_SIZE);
                GPU_MatrixVectorMultiply_CSR_Kernel_addingVersion <<< Grid, BLOCK_SIZE >>>(G[lvl].numRows, G[lvl].indices_gpu,
		    G[lvl].ptr_gpu, G[lvl].values_gpu, b.values_gpu, u.values_gpu);
                if(!CUDA_Success(hipDeviceSynchronize())){
                    cout << "Error en linea 130" << endl;
                }
                //from the second sweel on we need to recompute the residual
                for(size_t i = 1; i < preSweeps[lvl]; i++){
		    if(!vectorized)
			calculateInstantVector(u, b, A[lvl], G[lvl]);
		    else
			calculateInstantVector_vectorized(u, b, A[lvl], G[lvl]);
		}

                if(!vectorized)
                    generateResidual(R[lvl], b, A[lvl], u, r);
                else
                    generateResidual_vectorized(R[lvl], b, A[lvl], u, r);


            }
            else //preSweeps[0] == 0 case
            {
                //inefficient! -- in this case we do not need to recompute the residual
                if(!vectorized)
                    generateResidual(R[lvl], b, A[lvl], u, r);
                else
                    generateResidual_vectorized(R[lvl], b, A[lvl], u, r);

            }
        }
        else
        {
          //from the second sweel on we need to recompute the residual
                for(size_t i = 0; i < preSweeps[lvl]; i++){
		    if(!vectorized)
			calculateInstantVector(u, b, A[lvl], G[lvl]);
		    else
			calculateInstantVector_vectorized(u, b, A[lvl], G[lvl]);
		}


                if(!vectorized)
                    generateResidual(R[lvl], b, A[lvl], u, r);
                else
                    generateResidual_vectorized(R[lvl], b, A[lvl], u, r);
        }
        _Vector v;
        v.numElems = r.numElems;
        v.values_cpu = new double[v.numElems];
        malloc_(v.values_gpu, v.numElems);
        dim3 Grid = Build_Grid(v.numElems, BLOCK_SIZE);
        fillWithZeros <<< Grid, BLOCK_SIZE >>>(v.values_gpu, v.numElems);
        if(!CUDA_Success(hipDeviceSynchronize())){
                cout << "Error en linea 160" << endl;
        }

        multilevel(A, P, R, G, r, v, lvl+1, maxLevels, preSweeps, postSweeps, assumeZeros);

        _Vector pv;
        pv.numElems = P[lvl].numRows;
        // malloc de pv.values_gpu
        malloc_(pv.values_gpu, pv.numElems);
        Grid = Build_Grid(pv.numElems, BLOCK_SIZE);
        // here product matrix P with vector v
        GPU_MatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(P[lvl].numRows,
                P[lvl].indices_gpu, P[lvl].ptr_gpu, P[lvl].values_gpu, v.values_gpu, pv.values_gpu);
        if(!CUDA_Success(hipDeviceSynchronize())){
            cout << "Error en linea 173" << endl;
        }
        // here the addition of pv to u
        Grid = Build_Grid(u.numElems, BLOCK_SIZE);
        sumVectorVector <<< Grid, BLOCK_SIZE >>> (pv.values_gpu, u.values_gpu, u.numElems);
        if(!CUDA_Success(hipDeviceSynchronize())){
            cout << "Error en linea 180" << endl;
        }
        //delete de pv, v i r
        CUDA_CHECK(hipFree(pv.values_gpu));
        CUDA_CHECK(hipFree(v.values_gpu));
        CUDA_CHECK(hipFree(r.values_gpu));
        delete[] r.values_cpu;
        delete[] v.values_cpu;

        //double norm2 = checkResidual(u, b, A[lvl]);

	for(size_t i = 0; i < postSweeps[lvl]; i++){
	    if(!vectorized)
		calculateInstantVector(u, b, A[lvl], G[lvl]);
	    else
		calculateInstantVector_vectorized(u, b, A[lvl], G[lvl]);
	}

    }else{
	//clock_t t1 = clock();
        //here lapack direct solver

        copyMem(u.values_gpu, u.values_cpu, u.numElems, 1);
        copyMem(b.values_gpu, b.values_cpu, b.numElems, 1);


        LaGenMatDouble _A(A[lvl].matAuxValues, A[lvl].numRows, A[lvl].numCols);
        LaGenMatDouble _b(b.values_cpu, b.numElems, 1);
        LaGenMatDouble _x(u.values_cpu, u.numElems, 1);

    	_x.inject(_b);            // will throw exception if not conformant

	integer info = 0;
	int M = _A.size(0);
	integer Ml = M;
	integer lda = _A.inc(0) * _A.gdim(0);

	integer K = _x.size(1);
	integer ldx = _x.inc(0) * _x.gdim(0);
	F77NAME(dgetrs) ("No transpose", &Ml, &K, &_A(0,0), &lda, &ipiv(0), &_x(0,0), &ldx, &info);

	//int res = clapack_dgetrs(CblasRowMajor, CblasNoTrans, &Ml, &K, &_A(0,0), &lda, &ipiv(0), &_x(0,0), &ldx);
	//std::cout << "Problem with lapack, num " << res << std:endl;
	copyMem(u.values_cpu, u.values_gpu, u.numElems, 0);

	//clock_t t2 = clock();
	//cout << "Lower lvl timing " << double(t2-t1) / CLOCKS_PER_SEC << "s" << endl;
   }

}
/** This function is a wrapper for u += G ( A, b, u) **/
void calculateInstantVector(_Vector& u, const _Vector& b, const _Matrix& A, const _Matrix& G)
{
    /** Au **/
    _Vector auxAU;
    auxAU.numElems = A.numRows;
    malloc_(auxAU.values_gpu, auxAU.numElems);
    dim3 Grid = Build_Grid(A.numRows, BLOCK_SIZE);
    GPU_MatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(A.numRows, A.indices_gpu,
            A.ptr_gpu, A.values_gpu, u.values_gpu, auxAU.values_gpu);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 238" << endl;
    }

    /** b - AU **/
    _Vector auxABU;
    auxABU.numElems = auxAU.numElems;
    malloc_(auxABU.values_gpu, auxABU.numElems);
    Grid = Build_Grid(A.numRows, BLOCK_SIZE);
    subVectorVector <<< Grid, BLOCK_SIZE >>>(b.values_gpu, auxAU.values_gpu, auxABU.values_gpu, auxABU.numElems);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 249" << endl;
    }
    /** u += G ( b - Au ) **/
    Grid = Build_Grid(A.numRows, BLOCK_SIZE);
    GPU_MatrixVectorMultiply_CSR_Kernel_addingVersion <<< Grid, BLOCK_SIZE >>>(G.numRows, G.indices_gpu,
            G.ptr_gpu, G.values_gpu, auxABU.values_gpu, u.values_gpu);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 255" << endl;
    }
    //deleting structures
    CUDA_CHECK(hipFree(auxABU.values_gpu));
    CUDA_CHECK(hipFree(auxAU.values_gpu));

}
void calculateInstantVector_vectorized(_Vector& u, const _Vector& b, const _Matrix& A, const _Matrix& G)
{
    /** Au **/
    _Vector auxAU;
    auxAU.numElems = A.numRows;
    malloc_(auxAU.values_gpu, auxAU.numElems);
    dim3 Grid = Build_Grid(A.numRows * HWS, BS);
    GPU_MatrixVectorMultiply_CSR_Vectorized_Kernel <<< Grid, BS >>>(A.numRows, A.indices_gpu,
            A.ptr_gpu, A.values_gpu, u.values_gpu, auxAU.values_gpu);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 272" << endl;
    }
    /** b - AU **/
    _Vector auxABU;
    auxABU.numElems = auxAU.numElems;
    malloc_(auxABU.values_gpu, auxABU.numElems);
    Grid = Build_Grid(A.numRows, BLOCK_SIZE);
    subVectorVector <<< Grid, BLOCK_SIZE >>>(b.values_gpu, auxAU.values_gpu, auxABU.values_gpu, auxABU.numElems);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 282" << endl;
    }
    /** u += G ( b - Au ) **/
    Grid = Build_Grid(A.numRows, BS);
    GPU_MatrixVectorMultiply_CSR_Kernel_addingVersion <<< Grid, BS >>>(G.numRows, G.indices_gpu,
            G.ptr_gpu, G.values_gpu, auxABU.values_gpu, u.values_gpu);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 288" << endl;
    }
    //deleting structures
    CUDA_CHECK(hipFree(auxABU.values_gpu));
    CUDA_CHECK(hipFree(auxAU.values_gpu));

}

void generateResidual(const _Matrix& R, const _Vector& b, const _Matrix& A, const _Vector& u, _Vector& r){
    /** Au **/
    _Vector auxAU;
    auxAU.numElems = A.numRows;
    malloc_(auxAU.values_gpu, auxAU.numElems);
    dim3 Grid = Build_Grid(A.numRows, BLOCK_SIZE);
    GPU_MatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(A.numRows, A.indices_gpu,
            A.ptr_gpu, A.values_gpu, u.values_gpu, auxAU.values_gpu);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 305" << endl;
    }
    /** b - AU **/
    _Vector auxABU;
    auxABU.numElems = auxAU.numElems;
    malloc_(auxABU.values_gpu, auxABU.numElems);
    Grid = Build_Grid(A.numRows, BLOCK_SIZE);
    subVectorVector <<< Grid, BLOCK_SIZE >>>(b.values_gpu, auxAU.values_gpu,
            auxABU.values_gpu, auxABU.numElems);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 315" << endl;
    }
    /** r = R ( b - Au ) **/
    r.numElems = R.numRows;
    malloc_(r.values_gpu, r.numElems);
    r.values_cpu = new double[r.numElems];
    Grid = Build_Grid(R.numRows, BLOCK_SIZE);
    GPU_MatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(R.numRows, R.indices_gpu,
            R.ptr_gpu, R.values_gpu, auxABU.values_gpu, r.values_gpu);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 325" << endl;
    }
    //deleting structures
    CUDA_CHECK(hipFree(auxABU.values_gpu));
    CUDA_CHECK(hipFree(auxAU.values_gpu));
}
void generateResidual_vectorized(const _Matrix& R, const _Vector& b, const _Matrix& A, const _Vector& u, _Vector& r){
    /** Au **/
    _Vector auxAU;
    auxAU.numElems = A.numRows;
    malloc_(auxAU.values_gpu, auxAU.numElems);
    dim3 Grid = Build_Grid(A.numRows * HWS, BS);
    GPU_MatrixVectorMultiply_CSR_Vectorized_Kernel <<< Grid, BS >>>(A.numRows, A.indices_gpu,
            A.ptr_gpu, A.values_gpu, u.values_gpu, auxAU.values_gpu);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 340" << endl;
    }
    /** b - AU **/
    _Vector auxABU;
    auxABU.numElems = auxAU.numElems;
    malloc_(auxABU.values_gpu, auxABU.numElems);
    Grid = Build_Grid(A.numRows, BLOCK_SIZE);
    subVectorVector <<< Grid, BLOCK_SIZE >>>(b.values_gpu, auxAU.values_gpu,
            auxABU.values_gpu, auxABU.numElems);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 350" << endl;
    }
    /** r = R ( b - Au ) **/
    r.numElems = R.numRows;
    malloc_(r.values_gpu, r.numElems);
    r.values_cpu = new double[r.numElems];
    Grid = Build_Grid(R.numRows *  HWS, BS);
    GPU_MatrixVectorMultiply_CSR_Vectorized_Kernel <<< Grid, BS >>>(R.numRows, R.indices_gpu,
            R.ptr_gpu, R.values_gpu, auxABU.values_gpu, r.values_gpu);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 360" << endl;
    }
    //deleting structures
    CUDA_CHECK(hipFree(auxABU.values_gpu));
    CUDA_CHECK(hipFree(auxAU.values_gpu));
}

/** This function will return the hierarchy reconstructed of A and b;
    Additionally return the number of real hierarchy levels.
    All matrices and vectors are returned as GPU structures**/
size_t generateHierarchy(_Matrix*& Matrices, _Matrix*& Pmat, _Matrix*& Qmat,
        _Matrix*& Gmat, double W, size_t numLevelsRoh, size_t max_levels, size_t min_system_size)
{
    size_t i = 0;
    for(i = 0; i < max_levels; i++){
        /** This condition controls MAX_SYSTEM_SIZE for the last matrix in hierarchy **/
        if(Matrices[i].numRows < min_system_size || i == max_levels-1){
/*		cout << "BEFORE" << endl;
		for(size_t j = 0; j < Matrices[i].numNNZ; j++){
			cout << Matrices[i].values_cpu[j] << " ";
		}
		cout << endl << endl;*/
            computeDenseMatrix(Matrices[i], Matrices[i].matAuxValues);
            LaGenMatDouble A(Matrices[i].matAuxValues, Matrices[i].numRows, Matrices[i].numCols);

            int M = A.size(0);
            integer Ml = M;
            integer lda = A.inc(0) * A.gdim(0);

            LaVectorLongInt ipiv_( M);
            ipiv = ipiv_;
            integer info = 0;
            int max = M > 1 ? M : 1;
            if(M < 0){
                info = -1;
            }else if(M < 0){
                //este caso lo saltamos porque tratamos matrices cuadradas y por tanto se asume en el caso anterior
            }else if(lda < max){
                info  = -4;
            }
            F77NAME(dgetrf) (&Ml, &Ml, &A(0,0), &lda, &ipiv(0), &info);

		
	    if( i > 0 ){ //A matrix is not the first one in hierarchy
		    if(Matrices[i].ptr_cpu != NULL) delete[] Matrices[i].indices_cpu;
		    if(Matrices[i].values_cpu != NULL) delete[] Matrices[i].values_cpu;
		    if(Matrices[i].indices_cpu != NULL) delete[] Matrices[i].ptr_cpu;
	    }
            Matrices[i].numValuesDenseRep = A.inc(0) * A.gdim(0);
/*		cout << "AFTER" << endl;
		for(size_t j = 0; j < Matrices[i].numValuesDenseRep; j++){
			cout << Matrices[i].matAuxValues[j] << " ";
		}
		cout << endl << endl;
		exit(1);*/
            break;
        }
        _Matrix newDiag;
        _Vector diag;
        clock_t t1 = clock();
        createDiagonal_vCPU(Matrices[i], newDiag, diag);
        clock_t t2 = clock();
        Gmat[i] = newDiag;
        /** Generating P and Q for the current A level **/
        _Matrix P = generateP_vCPU(Matrices[i], diag, W, numLevelsRoh);
        clock_t t3 = clock();
        _Matrix Q = generateQ(P);
        clock_t t4 = clock();
        Pmat[i] = P;
        Qmat[i] = Q;

        //CUDA_CHECK(hipFree(diag.values_gpu));
	delete[] diag.values_cpu;

        /** Allocating result matrix, and partialResult matrix **/
        _Matrix matResult, matPartialResult;

        matPartialResult.numRows = Q.numRows;
        matPartialResult.numCols = Matrices[i].numCols;
        matPartialResult.ptr_cpu = new size_t[matPartialResult.numRows+1];

        csr_matmat_pass1(Q.numRows,
                      Matrices[i].numCols,
                      Q.ptr_cpu,
                      Q.indices_cpu,
                      Matrices[i].ptr_cpu,
                      Matrices[i].indices_cpu,
                            matPartialResult.ptr_cpu);
        matPartialResult.numNNZ = matPartialResult.ptr_cpu[matPartialResult.numRows];
        matPartialResult.indices_cpu = new size_t[matPartialResult.numNNZ];
        matPartialResult.values_cpu = new double[matPartialResult.numNNZ];
        csr_matmat_pass2(Q.numRows,
      	              Matrices[i].numCols,
      	              Q.ptr_cpu,
                      Q.indices_cpu,
      	              Q.values_cpu,
      	              Matrices[i].ptr_cpu,
                      Matrices[i].indices_cpu,
      	              Matrices[i].values_cpu,
      	                    matPartialResult.ptr_cpu,
      	                    matPartialResult.indices_cpu,
                            matPartialResult.values_cpu);
        matPartialResult.numNNZ = matPartialResult.ptr_cpu[matPartialResult.numRows];

        matResult.numRows = matPartialResult.numRows;
        matResult.numCols = P.numCols;
        matResult.ptr_cpu = new size_t[matResult.numRows+1];
        csr_matmat_pass1(matPartialResult.numRows,
                      P.numCols,
                      matPartialResult.ptr_cpu,
                      matPartialResult.indices_cpu,
                      P.ptr_cpu,
                      P.indices_cpu,
                            matResult.ptr_cpu);
        matResult.numNNZ = matResult.ptr_cpu[matResult.numRows];
        matResult.indices_cpu = new size_t[matResult.numNNZ];
        matResult.values_cpu = new double[matResult.numNNZ];
        csr_matmat_pass2(matPartialResult.numRows,
                      P.numCols,
                      matPartialResult.ptr_cpu,
                      matPartialResult.indices_cpu,
      	              matPartialResult.values_cpu,
      	              P.ptr_cpu,
                      P.indices_cpu,
      	              P.values_cpu,
      	                    matResult.ptr_cpu,
      	                    matResult.indices_cpu,
      	                    matResult.values_cpu);
        matResult.numNNZ = matResult.ptr_cpu[matResult.numRows];

        sortMatrix(matResult, true);
        if(matResult.numRows >= min_system_size || i+1 == max_levels-1 ){
            mallocAndCopyMem(matResult.indices_cpu, matResult.indices_gpu, matResult.numNNZ);
            mallocAndCopyMem(matResult.ptr_cpu, matResult.ptr_gpu, matResult.numRows+1);
            mallocAndCopyMem(matResult.values_cpu, matResult.values_gpu, matResult.numNNZ);
        }
        clock_t t5 = clock();
        /** Store new matrix in the next lvl of hierarchy, i+1 **/
        Matrices[i+1] = matResult;

        /** Free memory from useless structures **/
        delete[] matPartialResult.indices_cpu;
        delete[] matPartialResult.ptr_cpu;
        delete[] matPartialResult.values_cpu;

/*        cout << "Level " << i << endl;
        cout << "   Time to create diagonal " << double(t2-t1) / CLOCKS_PER_SEC << "s" << endl;
        cout << "   Time to create P " << double(t3-t2) / CLOCKS_PER_SEC << "s" << endl;
        cout << "   Time to create Q " << double(t4-t3) / CLOCKS_PER_SEC << "s" << endl;
        cout << "   Time to create mat " << double(t5-t4) / CLOCKS_PER_SEC << "s" << endl;*/
    }
    return i;
}

_Matrix generateP_vCPU(const _Matrix& A, const _Vector& diag, double W, size_t numLevelsRoh){
    _Matrix Ptent;
    createPTent(A, Ptent);

    _Matrix P;
    /** Create P from Ptent **/
    //wDA
    _Matrix prodMat;
    prodMat.numNNZ = A.numNNZ;
    prodMat.numRows = A.numRows;
    prodMat.numCols = A.numCols;

    prodMat.indices_cpu = new size_t[prodMat.numNNZ];
    prodMat.values_cpu = new double[prodMat.numNNZ];
    prodMat.ptr_cpu = new size_t[prodMat.numRows+1];

    prodMat.ptr_cpu[0] = 0;
    size_t currentIndice = 0;
    for(size_t i = 0; i < A.numRows; i++){
        prodMat.ptr_cpu[i+1] = prodMat.ptr_cpu[i];
        if(diag.values_cpu[i] != 0.0){
            for(size_t r = A.ptr_cpu[i]; r < A.ptr_cpu[i+1]; r++){
                prodMat.ptr_cpu[i+1]++;
                prodMat.indices_cpu[currentIndice] = A.indices_cpu[r];
                prodMat.values_cpu[currentIndice] = A.values_cpu[r] * diag.values_cpu[i];
                currentIndice++;
            }
        }
    }

    prodMat.numNNZ = prodMat.ptr_cpu[prodMat.numRows];
    mallocAndCopyMem(prodMat.indices_cpu, prodMat.indices_gpu, prodMat.numNNZ);
    mallocAndCopyMem(prodMat.values_cpu, prodMat.values_gpu, prodMat.numNNZ);
    mallocAndCopyMem(prodMat.ptr_cpu, prodMat.ptr_gpu, prodMat.numRows+1);

    double roh_ = roh(prodMat, numLevelsRoh);
    double W_ = W / roh_;
    //cout << "La W queda aixi: " << W_ << ", i la W es: " << W << endl;

    for(size_t i = 0; i < prodMat.numNNZ; i++){
        prodMat.values_cpu[i] *= W_;
    }

    //printMatrix(prodMat);

    //I - wDA
    _Matrix subMat;
    subMat.numNNZ = prodMat.numRows + prodMat.numNNZ;
    subMat.numRows = prodMat.numRows;
    subMat.numCols = prodMat.numCols;

    subMat.ptr_cpu = new size_t[subMat.numRows+1];
    subMat.indices_cpu = new size_t[subMat.numNNZ];
    subMat.values_cpu = new double[subMat.numNNZ];
    subIdentityMatrix_cpu(prodMat, subMat);
    subMat.numNNZ = subMat.ptr_cpu[subMat.numRows];


    //(I - wDA) * PTent
    P.numRows = subMat.numRows;
    P.numCols = Ptent.numCols;
    P.ptr_cpu = new size_t[P.numRows+1];
    
    csr_matmat_pass1(subMat.numRows,
                      Ptent.numCols,
                      subMat.ptr_cpu,
                      subMat.indices_cpu,
                      Ptent.ptr_cpu,
                      Ptent.indices_cpu,
                            P.ptr_cpu);
    P.numNNZ = P.ptr_cpu[P.numRows];
    P.indices_cpu = new size_t[P.numNNZ];
    P.values_cpu = new double[P.numNNZ];
    csr_matmat_pass2(subMat.numRows,
                      Ptent.numCols,
                      subMat.ptr_cpu,
                      subMat.indices_cpu,
      	              subMat.values_cpu,
      	              Ptent.ptr_cpu,
                      Ptent.indices_cpu,
      	              Ptent.values_cpu,
      	                    P.ptr_cpu,
                            P.indices_cpu,
                            P.values_cpu);
    P.numNNZ = P.ptr_cpu[P.numRows];
    sortMatrix(P, true);


    mallocAndCopyMem(P.ptr_cpu, P.ptr_gpu, P.numRows+1);
    mallocAndCopyMem(P.indices_cpu, P.indices_gpu, P.numNNZ);
    mallocAndCopyMem(P.values_cpu, P.values_gpu, P.numNNZ);

    /** Free resources **/
    delete[] prodMat.ptr_cpu;
    delete[] prodMat.indices_cpu;
    delete[] prodMat.values_cpu;
    delete[] Ptent.indices_cpu;
    delete[] Ptent.ptr_cpu;
    delete[] Ptent.values_cpu;


    CUDA_CHECK(hipFree(prodMat.ptr_gpu));
    CUDA_CHECK(hipFree(prodMat.indices_gpu));
    CUDA_CHECK(hipFree(prodMat.values_gpu));

    delete[] subMat.ptr_cpu;
    delete[] subMat.indices_cpu;
    delete[] subMat.values_cpu;

    /** Return P **/
    return P;
}

/** This is a simple function that transposes P assuming symmetric matrix
 *  Soon it will be need to implementate the non-symmetric construction of Q*/
_Matrix generateQ(const _Matrix& P){
    _Matrix Q;
    Q.numNNZ = P.numNNZ;
    Q.numCols = P.numRows;
    Q.numRows = P.numCols;
    Q.indices_cpu = new size_t[Q.numNNZ];
    Q.values_cpu = new double[Q.numNNZ];
    Q.ptr_cpu = new size_t[Q.numRows+1];
    malloc_(Q.indices_gpu, Q.numNNZ);
    malloc_(Q.values_gpu, Q.numNNZ);
    malloc_(Q.ptr_gpu, Q.numRows+1);
    //ens es suficient amb utilitzar la funcio csr_tocsc i prendre-ho com csr
    csr_tocsc(P.numRows,
	           P.numCols,
	           P.ptr_cpu,
	           P.indices_cpu,
	           P.values_cpu,
	                 Q.ptr_cpu,
	                 Q.indices_cpu,
	                 Q.values_cpu);
    //copy back
    copyMem(Q.values_cpu, Q.values_gpu, Q.numNNZ, 0);
    copyMem(Q.indices_cpu, Q.indices_gpu, Q.numNNZ, 0);
    copyMem(Q.ptr_cpu, Q.ptr_gpu, Q.numRows+1, 0);
    return Q;
}

/** This function will create G from diagonal of A **/
void createDiagonal(const _Matrix& A, _Vector& res){
    res.numElems = A.numRows;
    //malloc_(res.values_gpu, res.numElems);
    res.values_cpu = new double[res.numElems];
    malloc_(res.values_gpu, res.numElems);
    
    for(size_t i = 0; i < A.numRows; i++){
        res.values_cpu[i] = 0.0;
        for(size_t r = A.ptr_cpu[i]; r < A.ptr_cpu[i+1]; r++){
            if(A.indices_cpu[r] == i){
                res.values_cpu[i] =1.0/A.values_cpu[r];
                break;
            }
        }
    }
    //copia cpu->gpu
    copyMem(res.values_cpu, res.values_gpu, res.numElems, 0);
}

void createDiagonal_vCPU(const _Matrix& A, _Matrix&G, _Vector& res){
    res.numElems = A.numRows;
    //malloc_(res.values_gpu, res.numElems);
    res.values_cpu = new double[res.numElems];

    G.numRows = A.numRows;
    G.numCols = A.numCols;
    malloc_(G.values_gpu, G.numRows);
    malloc_(G.indices_gpu, G.numRows);
    malloc_(G.ptr_gpu, G.numRows+1);
    G.ptr_cpu = new size_t[G.numRows+1];
    G.indices_cpu = new size_t[G.numRows];
    G.values_cpu = new double[G.numRows];

    size_t currentIndice = 0;
    G.ptr_cpu[0] = 0;
    for(size_t i = 0; i < A.numRows; i++){
        G.ptr_cpu[i+1] = G.ptr_cpu[i];
        res.values_cpu[i] = 0.0;
        for(size_t r = A.ptr_cpu[i]; r < A.ptr_cpu[i+1]; r++){
            if(A.indices_cpu[r] == i){
                G.indices_cpu[currentIndice] = A.indices_cpu[r];
                res.values_cpu[i] = G.values_cpu[currentIndice] = 1.0/A.values_cpu[r];
                G.ptr_cpu[i+1]++;
                currentIndice++;
                break;
            }
        }
    }

    //copia cpu->gpu

    G.numNNZ = G.ptr_cpu[G.numRows];
    copyMem(G.ptr_cpu, G.ptr_gpu, G.numRows+1, 0);
    copyMem(G.values_cpu, G.values_gpu, G.numNNZ, 0);
    copyMem(G.indices_cpu, G.indices_gpu, G.numNNZ, 0);

    //copyMem(res.values_cpu, res.values_gpu, res.numElems, 0);
}
/** Aggregation of A to generate the colored graph **/
template <class I>
I standardAggregation(const I n_row,
                       const I Ap[],
                       const I Aj[],
                             I  x[])
{
    // Bj[n] == -1 means i-th node has not been aggregated
    std::fill(x, x + n_row, 0);

    I next_aggregate = 1; // number of aggregates + 1

    //Pass #1
    for(I i = 0; i < n_row; i++){
        if(x[i]){ continue; } //already marked

        const I row_start = Ap[i];
        const I row_end   = Ap[i+1];

        //Determine whether all neighbors of this node are free (not already aggregates)
        bool has_aggregated_neighbors = false;
        bool has_neighbors            = false;
        for(I jj = row_start; jj < row_end; jj++){
            const I j = Aj[jj];
            if( i != j ){
                has_neighbors = true;
                if( x[j] ){
                    has_aggregated_neighbors = true;
                    break;
                }
            }
        }

        if(!has_neighbors){
            //isolated node, do not aggregate
            x[i] = -n_row;
        }
        else if (!has_aggregated_neighbors){
            //Make an aggregate out of this node and its neighbors
            x[i] = next_aggregate;
            for(I jj = row_start; jj < row_end; jj++){
                x[Aj[jj]] = next_aggregate;
            }
            next_aggregate++;
        }
    }


    //Pass #2
    // Add unaggregated nodes to any neighboring aggregate
    for(I i = 0; i < n_row; i++){
        if(x[i]){ continue; } //already marked

        for(I jj = Ap[i]; jj < Ap[i+1]; jj++){
            const I j = Aj[jj];

            const I xj = x[j];
            if(xj > 0){
                x[i] = -xj;
                break;
            }
        }
    }

    next_aggregate--;

    //Pass #3
    for(I i = 0; i < n_row; i++){
        const I xi = x[i];

        if(xi != 0){
            // node i has been aggregated
            if(xi > 0)
                x[i] = xi - 1;
            else if(xi == -n_row)
                x[i] = -1;
            else
                x[i] = -xi - 1;
            continue;
        }

        // node i has not been aggregated
        const I row_start = Ap[i];
        const I row_end   = Ap[i+1];

        x[i] = next_aggregate;

        for(I jj = row_start; jj < row_end; jj++){
            const I j = Aj[jj];

            if(x[j] == 0){ //unmarked neighbors
                x[j] = next_aggregate;
            }
        }
        next_aggregate++;
    }

    return next_aggregate; //number of aggregates
}

/** This function will create Ptent from a given A **/
void createPTent(const _Matrix& A, _Matrix& P){
    long *x;
    long size = A.numRows;
    x = new long[size];
    size_t maxColumn = standardAggregation(size, (long*)A.ptr_cpu, (long*)A.indices_cpu, x);
    P.ptr_cpu = new size_t[size+1];
    P.indices_cpu = new size_t[size];
    P.values_cpu = new double[size];

    long* auxValues = new long[size];
    //ini auxValues to 0
    for(size_t i = 0; i < size; i++){
        auxValues[i] = 0;
    }
    //calculate each value group num
    for(size_t i = 0; i < size; i++){
        auxValues[x[i]]++;
    }
    for(size_t i = 0; i < size; i++){
        if(x[i] >= size)
            cout << "a l'index " << i << " tenim un valor de " << x[i]  << ", i el size es: " << size << endl;
    }
    //assign indices and right values
    P.ptr_cpu[0] = 0;
    for(size_t i = 0; i < size; i++){
        size_t j = x[i];
        P.ptr_cpu[i+1] = P.ptr_cpu[i] + 1;
        P.indices_cpu[i] = j;
        P.values_cpu[i] = 1.0/sqrt(auxValues[j]);
    }

    P.numCols = maxColumn;
    P.numRows = size;
    P.numNNZ = size;

    delete[] x;
    delete[] auxValues;
}

void csr_tocsc(const size_t n_row,
	           const size_t n_col,
	           const size_t Ap[],
	           const size_t Aj[],
	           const double Ax[],
	                 size_t*& Bp,
	                 size_t* Bi,
	                 double* Bx)
{
    const size_t nnz = Ap[n_row];
    Bp = new size_t[n_col+1];
    //compute number of non-zero entries per column of A
    std::fill(Bp, Bp + n_col, 0);

    for (size_t n = 0; n < nnz; n++){
        Bp[Aj[n]]++;
    }

    //cumsum the nnz per column to get Bp[]
    for(size_t col = 0, cumsum = 0; col < n_col; col++){
        size_t temp  = Bp[col];
        Bp[col] = cumsum;
        cumsum += temp;
    }
    Bp[n_col] = nnz;

    for(size_t row = 0; row < n_row; row++){
        for(size_t jj = Ap[row]; jj < Ap[row+1]; jj++){
            size_t col  = Aj[jj];
            size_t dest = Bp[col];

            Bi[dest] = row;
            Bx[dest] = Ax[jj];

            Bp[col]++;
        }
    }

    for(size_t col = 0, last = 0; col <= n_col; col++){
        size_t temp  = Bp[col];
        Bp[col] = last;
        last    = temp;
    }
}

double calculateNorm(_Vector& b){
    double finalNum = 0.0;
    for(size_t i = 0; i < b.numElems; i++){
        finalNum += pow(b.values_cpu[i], 2);
    }
    return sqrt(finalNum);
}

double calculateNorm_GPU(_Vector&b){
    return hipblasDnrm2(b.numElems, b.values_gpu, 1);
}

double checkResidual(const _Vector& u, const _Vector& b, const _Matrix& A){
    /** Au **/
    _Vector auxAU;
    auxAU.numElems = A.numRows;
    malloc_(auxAU.values_gpu, auxAU.numElems);
    dim3 Grid = Build_Grid(A.numRows, BLOCK_SIZE);
    GPU_MatrixVectorMultiply_CSR_Kernel <<< Grid, BLOCK_SIZE >>>(A.numRows, A.indices_gpu,
            A.ptr_gpu, A.values_gpu, u.values_gpu, auxAU.values_gpu);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 1208" << endl;
    }
    /** b - AU **/
    _Vector auxABU;
    auxABU.numElems = auxAU.numElems;
    auxABU.values_cpu = new double[auxABU.numElems];
    malloc_(auxABU.values_gpu, auxABU.numElems);
    Grid = Build_Grid(A.numRows, BLOCK_SIZE);
    subVectorVector <<< Grid, BLOCK_SIZE >>>(b.values_gpu, auxAU.values_gpu,
            auxABU.values_gpu, auxABU.numElems);
    if(!CUDA_Success(hipDeviceSynchronize())){
        cout << "Error en linea 1219" << endl;
    }
    copyMem(auxABU.values_gpu, auxABU.values_cpu, auxABU.numElems, 1);
    double finalNum = calculateNorm_GPU(auxABU);
    delete[] auxABU.values_cpu;
    CUDA_CHECK(hipFree(auxAU.values_gpu));
    CUDA_CHECK(hipFree(auxABU.values_gpu));

    return finalNum;
}

bool checkConvergence(const _Vector& u, const _Vector& b, const _Matrix& A, const double lastResidual, const double threshold){
    //double threshold = 1e-9;
    double newResidual = checkResidual(u, b, A);
    cout << "New norm: " << newResidual << endl;
    cout << "Current status: " << newResidual/lastResidual << endl;
/*    for(size_t i = 0; i < 5; i++){
        cout << "Element " << i << ": " << u.values_cpu[i] << endl;
    }*/
    cout << endl;
    if(newResidual/lastResidual < threshold){
        return true;
    }
    return false;

}

void computeDenseMatrix(const _Matrix& A, double *& vec){
    vec = new double[A.numCols * A.numRows];
    size_t pointer = 0;
    size_t currentIndice = 0;
    for(size_t i = 0; i < A.numRows; i++){
        size_t nonZeros = A.ptr_cpu[i+1] - A.ptr_cpu[i];
        size_t columnPointer = 0;
        for(size_t j = 0; j < A.numCols; j++){
            if(nonZeros > 0  && A.indices_cpu[currentIndice] == columnPointer){
                vec[pointer] = (double)A.values_cpu[currentIndice];
                currentIndice++;
                nonZeros--;
            }else{
                vec[pointer] = 0.0;
            }
            pointer++;
            columnPointer++;
        }
    }
}

void subIdentityMatrix_cpu(const _Matrix& A, _Matrix& sub){
    size_t currentIndex = 0;
    sub.ptr_cpu[0] = 0;
    for(size_t i = 0; i < A.numRows; i++){
        bool haveDiagonal = false;
        size_t numElems = 0;
        long lastIndex = -1;
        for(size_t r = A.ptr_cpu[i]; r < A.ptr_cpu[i+1]; r++){
            lastIndex = A.indices_cpu[r];
            if(A.indices_cpu[r] > i && !haveDiagonal){
                sub.indices_cpu[currentIndex] = i;
                sub.values_cpu[currentIndex] = 1.0;
                currentIndex++;
                haveDiagonal = true;
                numElems++;
            }
            if(A.indices_cpu[r] == i){
                haveDiagonal = true;
                if(A.values_cpu[r] != 1.0){
                    sub.indices_cpu[currentIndex] = A.indices_cpu[r];
                    sub.values_cpu[currentIndex] = 1.0 - A.values_cpu[r];
                    currentIndex++;
                    numElems++;
                }
            }else{
                sub.indices_cpu[currentIndex] = A.indices_cpu[r];
                sub.values_cpu[currentIndex] = -A.values_cpu[r];
                currentIndex++;
                numElems++;
            }
        }
        if(lastIndex < i){
            sub.indices_cpu[currentIndex] = i;
            sub.values_cpu[currentIndex] = 1.0;
            currentIndex++;
            numElems++;
        }
        sub.ptr_cpu[i+1] = sub.ptr_cpu[i] + numElems;
    }
}

double eigVals(_Vector& H, size_t finalIters){
    LaGenMatDouble A( H.values_cpu, finalIters, finalIters);
    LaVectorDouble eigvals_real(finalIters);
    LaVectorDouble eigvals_imag(finalIters);
    LaGenMatDouble VR(finalIters, finalIters);

    LaEigSolve(A, eigvals_real,
		eigvals_imag, VR);
    double *real = eigvals_real.addr();
    double *imag = eigvals_imag.addr();

    double max = real[0];
    if(max < 0)
        max = -max;
    for(size_t i = 1; i < finalIters; i++){
        if(real[i] < 0)
            real[i] = -real[i];
        if(real[i] > max)
            max = real[i];
    }

 /*   cout << "ROH VALUES " << endl;
    for(size_t i = 0; i < finalIters; i++)
        cout << real[i] << " ";
    cout << endl;*/
    return max;
}

double roh(const _Matrix& A, size_t iter){
    double threshold = 1e-10;

    size_t maxIter;
    maxIter = A.numCols < iter ?  A.numCols : iter;

    _Vector *V = new _Vector[maxIter+1];

    V[0].numElems = A.numCols;
    V[0].values_cpu = new double[V[0].numElems];
    srand(0);
    for(size_t i = 0; i < V[0].numElems; i++){
        V[0].values_cpu[i] = (double)(((int)rand())%100000000)/100000000.0;
        //cout << V[0].values_cpu[i]<< endl;
    }
    double v0Norm = calculateNorm(V[0]);
    for(size_t i = 0; i < V[0].numElems; i++){
        V[0].values_cpu[i] /= v0Norm;
    }
    mallocAndCopyMem(V[0].values_cpu, V[0].values_gpu, V[0].numElems);
    //delete[] V[0].values_cpu;

    _Vector H;
    H.numElems = (maxIter+1) * (maxIter+1);
    H.values_cpu = new double[H.numElems];
    for(size_t q = 0; q < H.numElems; q++){
        H.values_cpu[q] = 0.0;
    }
    size_t numCurrentV = 1;

    size_t j;

    for(j = 0; j < maxIter; j++){
        V[numCurrentV].numElems = A.numRows;
        malloc_(V[numCurrentV].values_gpu, V[numCurrentV].numElems);
        V[numCurrentV].values_cpu = new double[V[numCurrentV].numElems];
        dim3 grid = Build_Grid(V[numCurrentV].numElems, BLOCK_SIZE);
        GPU_MatrixVectorMultiply_CSR_Kernel <<< grid, BLOCK_SIZE >>> (A.numRows, A.indices_gpu, A.ptr_gpu,
            A.values_gpu, V[numCurrentV-1].values_gpu, V[numCurrentV].values_gpu);
        if(!CUDA_Success(hipDeviceSynchronize())){
            cout << "Error en linea 1379" << endl;
        }
        copyMem(V[numCurrentV].values_gpu, V[numCurrentV].values_cpu, V[numCurrentV].numElems, 1);

        _Vector auxVec;
        auxVec.numElems = V[numCurrentV].numElems;
        malloc_(auxVec.values_gpu, auxVec.numElems);

        grid = Build_Grid(V[numCurrentV-1].numElems, BLOCK_SIZE);
        for(size_t i = 0; i < numCurrentV; i++){
            size_t matrixIndice = (i*(maxIter+1))+j;
            double auxVal = H.values_cpu[matrixIndice] = hipblasDdot(V[i].numElems, V[i].values_gpu, 1, V[numCurrentV].values_gpu, 1);
            subVectorConstantValue <<< grid, BLOCK_SIZE>>> (V[numCurrentV].values_gpu, H.values_cpu[matrixIndice], V[i].values_gpu, V[numCurrentV].numElems);
            if(!CUDA_Success(hipDeviceSynchronize())){
                cout << "Error en linea 1394" << endl;
            }
        }
        size_t matrixIndice = ((j+1) * (maxIter+1)) + j;
        copyMem(V[numCurrentV].values_gpu, V[numCurrentV].values_cpu, V[numCurrentV].numElems, 1);
        //copyMem(H.values_gpu, H.values_cpu, H.numElems, 1);
        H.values_cpu[matrixIndice] = calculateNorm(V[numCurrentV]);
        if(H.values_cpu[matrixIndice] < threshold)
            break;
        //copyMem(H.values_cpu, H.values_gpu, H.numElems, 0);

        divideVectorConstantValue <<< grid, BLOCK_SIZE >>> (V[numCurrentV].values_gpu, H.values_cpu[matrixIndice], V[numCurrentV].numElems);
        if(!CUDA_Success(hipDeviceSynchronize())){
            cout << "Error en linea 1407" << endl;
        }
        CUDA_CHECK(hipFree(auxVec.values_gpu));

        numCurrentV++;

    }
    for(size_t i = 0; i < numCurrentV; i++){
        delete[] V[i].values_cpu;
        CUDA_CHECK(hipFree(V[i].values_gpu));
    }
    delete[] V;

/*    for(size_t q = 0; q < maxIter+1; q++){
        for(size_t w = 0; w < maxIter+1; w++){
            cout << H.values_cpu[(q*(maxIter+1)) + w] << " ";
        }
        cout << endl;
    }
    cout << endl;*/

    double max = eigVals(H, maxIter+1);

    delete[] H.values_cpu;

    return max;
}

/**     Memory management functions     **/
/*template <class Q>
void mallocAndCopyMem(Q*& CPU, Q*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(Q)));
    CUDA_CHECK(hipMemcpy(GPU, CPU, size*sizeof(Q), hipMemcpyHostToDevice));
}*/

void mallocAndCopyMem(double*& CPU, double*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(double)));
    CUDA_CHECK(hipMemcpy(GPU, CPU, size*sizeof(double), hipMemcpyHostToDevice));
}

void mallocAndCopyMem(size_t*& CPU, size_t*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(size_t)));
    CUDA_CHECK(hipMemcpy(GPU, CPU, size*sizeof(size_t), hipMemcpyHostToDevice));
}

/*template <class Q>
void malloc_(Q*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(Q)));
}*/

void malloc_(double*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(double)));
}

void malloc_(size_t*& GPU, size_t size){
    CUDA_CHECK(hipMalloc((void**) &GPU, size*sizeof(size_t)));
}

template <class Q>
void copyMem(Q*& source, Q*& destiny, size_t size, unsigned short way){
    switch(way){
        case 0:
            CUDA_CHECK(hipMemcpy(destiny, source, size*sizeof(Q), hipMemcpyHostToDevice));
            break;
        case 1:
            CUDA_CHECK(hipMemcpy(destiny, source, size*sizeof(Q), hipMemcpyDeviceToHost));
            break;
        case 2:
            CUDA_CHECK(hipMemcpy(destiny, source, size*sizeof(Q), hipMemcpyDeviceToDevice));
            break;
    }
}

void deletingStuff(size_t* stuff){
    CUDA_CHECK(hipFree(stuff));
}

void deletingStuff(double* stuff){
    CUDA_CHECK(hipFree(stuff));
}

void GPU_VectorMultiply(double* sourceVec, double* destinyVec, size_t N){
	dim3 Grid = Build_Grid(N, BLOCK_SIZE);
	GPU_VectorVectorMultiplyElementWise_Kernel <<<Grid, BLOCK_SIZE>>> (N, sourceVec, destinyVec, destinyVec);
	GPUSparse::CUDA_Success(hipDeviceSynchronize());
}





}

}

//
// Compilation command
// make
