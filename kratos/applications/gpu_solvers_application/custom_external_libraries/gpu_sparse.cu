#include "hip/hip_runtime.h"
/*
==============================================================================
Kratos
A General Purpose Software for Multi-Physics Finite Element Analysis
Version 1.0 (Released on march 05, 2007).

Copyright 2007
Pooyan Dadvand, Riccardo Rossi, Farshid Mossaiby
pooyan@cimne.upc.edu
rrossi@cimne.upc.edu
mossaiby@yahoo.com
CIMNE (International Center for Numerical Methods in Engineering),
Gran Capita' s/n, 08034 Barcelona, Spain

Permission is hereby granted, free  of charge, to any person obtaining
a  copy  of this  software  and  associated  documentation files  (the
"Software"), to  deal in  the Software without  restriction, including
without limitation  the rights to  use, copy, modify,  merge, publish,
distribute,  sublicense and/or  sell copies  of the  Software,  and to
permit persons to whom the Software  is furnished to do so, subject to
the following condition:

Distribution of this code for  any  commercial purpose  is permissible
ONLY BY DIRECT ARRANGEMENT WITH THE COPYRIGHT OWNER.

The  above  copyright  notice  and  this permission  notice  shall  be
included in all copies or substantial portions of the Software.

THE  SOFTWARE IS  PROVIDED  "AS  IS", WITHOUT  WARRANTY  OF ANY  KIND,
EXPRESS OR  IMPLIED, INCLUDING  BUT NOT LIMITED  TO THE  WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT  SHALL THE AUTHORS OR COPYRIGHT HOLDERS  BE LIABLE FOR ANY
CLAIM, DAMAGES OR  OTHER LIABILITY, WHETHER IN AN  ACTION OF CONTRACT,
TORT  OR OTHERWISE, ARISING  FROM, OUT  OF OR  IN CONNECTION  WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

==============================================================================
*/

//
// Sparse matrix and vector operations on GPU

//
// Notes!
//
//   * In case of VectorVectorMultiply and VectorNorm2, cuBlas library has been used, as it is much faster than a code without proper optimizations.
//
//   * In case of VectorScaleAndAdd, as there is no direct way of using cuBlas for this, we had to write our own kernel. It was 2 or 3 times faster
//     than using cuBlas indirectly.
//
//   * For compilation command, simply issue a 'make' command.
//
//   * Removed templates to be able to link to Kratos.
//
//   * Modified GPUCSRMatrix to keep no. of non-zeros per row constant and equal to HALF_WARP_SIZE (16) and used same no. of threads to multiply a row

// More notes!
//
//	* Checks for error in Bind_X() and Unbind_X() have been removed for more consistent error checking after kernel calls; can they fail?
//
//	* Added an optional parameter to GPUCSRMatrix constructor, so that user can optionally avoid making non-zeros in a row a multiple of HALF_WARP_SIZE (16)
//
//	* Added a make file; in the command line use emu=1 for emulation mode and dbg=1 for a debug version

// Includes, system

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Includes, project

#include "gpu_sparse.h"
#include "gpu_sparse_utils.h"
#include "gpu_sparse_kernels.h"

namespace Kratos
{

namespace GPUSparse
{

// GPUVector class definition

GPUVector::GPUVector(size_t _Size, double *_CPU_Values): Size(_Size), CPU_Values(_CPU_Values), GPU_Values(0), Allocated(false)
{
	// Nothing to do!
}

GPUVector::GPUVector(size_t _Size): Size(_Size), CPU_Values(0), GPU_Values(0), Allocated(false)
{
	// Nothing to do!
}

GPUVector::~GPUVector()
{
	if (Allocated)
		GPU_Free();
}

bool GPUVector::GPU_Allocate()
{
	if (Allocated)
		return false;

	Allocated = true;

	return CUDA_Success(hipMalloc(reinterpret_cast <void **> (&GPU_Values), Size * sizeof(double)));
}

bool GPUVector::GPU_Free()
{
	if (!Allocated)
		return false;

	Allocated = false;

	return CUDA_Success(hipFree(GPU_Values));
}

bool GPUVector::Copy(CopyDirection Direction)
{
	if (!Allocated)
		return false;

	switch (Direction)
	{
		case CPU_GPU:

			return CUDA_Success(hipMemcpy(GPU_Values, CPU_Values, Size * sizeof(double), hipMemcpyHostToDevice));

		case GPU_CPU:

			return CUDA_Success(hipMemcpy(CPU_Values, GPU_Values, Size * sizeof(double), hipMemcpyDeviceToHost));
	}

	// We should never get here!
	return false;
}

bool GPUVector::CopyFromGPU(GPUVector &V)
{
	if (V.Size != Size || !Allocated || !V.Allocated)
		return false;

	return CUDA_Success(hipMemcpy(GPU_Values, V.GPU_Values, Size * sizeof(double), hipMemcpyDeviceToDevice));
}

// GPUCSRMatrix class definition

GPUCSRMatrix::GPUCSRMatrix(size_t _NNZ, size_t _Size1, size_t _Size2, size_t *_CPU_Columns, size_t *_CPU_RowIndices, double *_CPU_Values, bool _NZMultiple16): NNZ(_NNZ), Size1(_Size1), Size2(_Size2), CPU_Columns(0), CPU_RowIndices(0), CPU_Values(0), GPU_Columns(0), GPU_RowIndices(0), GPU_Values(0), Allocated(false)
{
	if (_NZMultiple16)
	{
		NNZ = 0;

		// Temporary RowIndices vector
		size_t *Temp_CPU_RowIndices = new size_t[Size1 + 1];

		Temp_CPU_RowIndices[0] = 0;

		// Find out how many non-zeros are needed to pad all rows to 16 while building the RowIndices
		for (size_t i = 0; i < Size1; i++)
		{
			size_t NZ = _CPU_RowIndices[i + 1] - _CPU_RowIndices[i];

			size_t R = NZ & HALF_WARP_SIZE_MASK;

			if (R != 0)
				NZ += HALF_WARP_SIZE - R;

			NNZ += NZ;
			Temp_CPU_RowIndices[i + 1] = Temp_CPU_RowIndices[i] + NZ;
		}

		// Allocate CPU memory for CSR structure using only one chunk of page-locked memory to speed up data transfer between CPU and GPU
		void *CSR_Data;

		if (!CUDA_Success(hipHostMalloc(&CSR_Data, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t))))  // TODO: What should be done?!
			CSR_Data = 0;

		// We are sure that using this order, the memory alignment conditions will be satisfied as NNZ is a multiple of HALF_WARP_SIZE (16) and sizeof(double) = 8
		// TODO: Check this!
		CPU_Values = reinterpret_cast <double *> (CSR_Data);  // Size: NNZ * sizeof(double)
		CPU_Columns = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * sizeof(double));  // Size: NNZ * sizeof(size_t)
		CPU_RowIndices = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * (sizeof(double) + sizeof(size_t)));  // Size: (Size1 + 1) * sizeof(size_t)

		// Move temporary data
		memcpy(CPU_RowIndices, Temp_CPU_RowIndices, (Size1 + 1) * sizeof(size_t));

		delete[] Temp_CPU_RowIndices;

		// Build ECSR structure from given CSR
		for (size_t i = 0; i < Size1; i++)
		{
			size_t _Start = _CPU_RowIndices[i], Start = CPU_RowIndices[i];

			for (size_t j = 0; j < _CPU_RowIndices[i + 1] - _CPU_RowIndices[i]; j++)
			{
				CPU_Columns[Start + j] = _CPU_Columns[_Start + j];
				CPU_Values[Start + j] = _CPU_Values[_Start + j];
			}

			size_t LastCol = _CPU_Columns[_CPU_RowIndices[i + 1] - 1];

			for (size_t j = _CPU_RowIndices[i + 1] - _CPU_RowIndices[i]; j < CPU_RowIndices[i + 1] - CPU_RowIndices[i]; j++)
			{
				CPU_Columns[Start + j] = LastCol;  // To maintain coalescing as much as possible
				CPU_Values[Start + j] = 0.00;
			}
		}
	}
	else
	{
		// Allocate CPU memory for CSR structure using only one chunk of page-locked memory to speed up data transfer between CPU and GPU
		void *CSR_Data;

		if (!CUDA_Success(hipHostMalloc(&CSR_Data, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t))))  // TODO: What should be done?!
			CSR_Data = 0;

		// We are sure that using this order, the memory alignment conditions will be satisfied as NNZ is a multiple of HALF_WARP_SIZE (16) and sizeof(double) = 8
		// TODO: Check this!
		CPU_Values = reinterpret_cast <double *> (CSR_Data);  // Size: NNZ * sizeof(double)
		CPU_Columns = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * sizeof(double));  // Size: NNZ * sizeof(size_t)
		CPU_RowIndices = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * (sizeof(double) + sizeof(size_t)));  // Size: (Size1 + 1) * sizeof(size_t)

		// Move temporary data
		memcpy(CPU_Values, _CPU_Values, NNZ * sizeof(double));
		memcpy(CPU_Columns, _CPU_Columns, NNZ * sizeof(size_t));
		memcpy(CPU_RowIndices, _CPU_RowIndices, (Size1 + 1) * sizeof(size_t));
	}
}

GPUCSRMatrix::GPUCSRMatrix(size_t _NNZ, size_t _Size1, size_t _Size2): NNZ(_NNZ), Size1(_Size1), Size2(_Size2), CPU_Columns(0), CPU_RowIndices(0), CPU_Values(0), GPU_Columns(0), GPU_RowIndices(0), GPU_Values(0), Allocated(false)
{
	// Nothing to do!
}

GPUCSRMatrix::~GPUCSRMatrix()
{
	// Free CSR data; as it is allocated in one chunk of memory, we need only to free the begining address
	hipHostFree(CPU_Values);

	if (Allocated)
		GPU_Free();
}

bool GPUCSRMatrix::GPU_Allocate()
{
	if (Allocated)
		return false;

	Allocated = true;

	// Allocate GPU memory for CSR structure using only one chunk of memory to speed up data transfer between CPU and GPU

	void *CSR_Data;

	if (CUDA_Success(hipMalloc(&CSR_Data, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t))))
	{
		GPU_Values = reinterpret_cast <double *> (CSR_Data);  // Size: NNZ * sizeof(double)
		GPU_Columns = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * sizeof(double));  // Size: NNZ * sizeof(size_t)
		GPU_RowIndices = reinterpret_cast <size_t *> (reinterpret_cast <size_t> (CSR_Data) + NNZ * (sizeof(double) + sizeof(size_t)));  // Size: (Size1 + 1) * sizeof(size_t)

		return true;
	}

	else
		return false;
}

bool GPUCSRMatrix::GPU_Free()
{
	if (!Allocated)
		return false;

	Allocated = false;

	// Free CSR data; as it is allocated in one chunk of memory, we need only to free the begining address
	return CUDA_Success(hipFree(GPU_Values));
}

bool GPUCSRMatrix::Copy(CopyDirection Direction, bool CopyValuesOnly)
{
	if (!Allocated)
		return false;

	switch (Direction)
	{
		case CPU_GPU:

			if (CopyValuesOnly)
				return CUDA_Success(hipMemcpy(GPU_Values, CPU_Values, NNZ * sizeof(double), hipMemcpyHostToDevice));
			else
				return CUDA_Success(hipMemcpy(GPU_Values, CPU_Values, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t), hipMemcpyHostToDevice));

		case GPU_CPU:

			if (CopyValuesOnly)
				return CUDA_Success(hipMemcpy(CPU_Values, GPU_Values, NNZ * sizeof(double), hipMemcpyDeviceToHost));
			else
				return CUDA_Success(hipMemcpy(CPU_Values, GPU_Values, NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t), hipMemcpyDeviceToHost));

	}

	// We should never get here!
	return false;
}

bool GPUCSRMatrix::CopyFromGPU(GPUCSRMatrix &M, bool CopyStructure, bool CopyValues)
{
	if (M.Size1 != Size1 || M.Size2 != Size2 || M.NNZ != NNZ || !Allocated || !M.Allocated)
		return false;

	size_t CopyLength;
	void *CopyFrom, *CopyTo;

	if (CopyStructure && CopyValues)
	{
		CopyTo = GPU_Values;
		CopyFrom = M.GPU_Values;
		CopyLength = NNZ * (sizeof(double) + sizeof(size_t)) + (Size1 + 1) * sizeof(size_t);
	}
	else if (CopyStructure && !CopyValues)
	{
		CopyTo = reinterpret_cast <void *> (reinterpret_cast <size_t> (GPU_Values) + NNZ * sizeof(double));
		CopyFrom = reinterpret_cast <void *> (reinterpret_cast <size_t> (M.GPU_Values) + NNZ * sizeof(double));
		CopyLength = (NNZ + Size1 + 1) * sizeof(size_t);
	}
	else if (!CopyStructure && CopyValues)
	{
		CopyTo = GPU_Values;
		CopyFrom = M.GPU_Values;
		CopyLength = NNZ * sizeof(double);
	}
	else if (!CopyStructure && !CopyValues)
	{
		CopyTo = 0;
		CopyFrom = 0;
		CopyLength = 0;
	}

	if (CopyLength != 0)
		return CUDA_Success(hipMemcpy(CopyTo, CopyFrom, CopyLength, hipMemcpyDeviceToDevice));
	else
		return true;
}

// Operations defined on GPUCSRMatrix and GPUVector

//
// CPU_MatrixVectorMultiply
// Matrix-Vector multiply on CPU

bool CPU_MatrixVectorMultiply(GPUCSRMatrix &A, GPUVector &X, GPUVector &Y)
{
	// Primary checks
	if (A.Size2 != X.Size || X.Size != Y.Size)
		return false;

	for (size_t i = 0; i < A.Size1; i++)
	{
		double YI = static_cast <double> (0);

		for (size_t j = A.CPU_RowIndices[i]; j < A.CPU_RowIndices[i + 1]; j++)
			YI += A.CPU_Values[j] * X.CPU_Values[A.CPU_Columns[j]];

		Y.CPU_Values[i] = YI;
	}

	return true;
}

//
// GPU_MatrixVectorMultiply
// Matrix-Vector multiply on GPU

bool GPU_MatrixVectorMultiply(GPUCSRMatrix &A, GPUVector &X, GPUVector &Y)
{
	// Primary checks
	if (A.Size2 != X.Size || X.Size != Y.Size || !X.Allocated || !Y.Allocated)
		return false;

#ifdef USE_TEXTURE_CACHING

	// Bind the texture memory to X
	Bind_X(X.GPU_Values);

#endif

	bool UseVectorizedVersion = (A.NNZ / A.Size2) > 10;	// From nVidia forum

	if (UseVectorizedVersion)
	{
		dim3 Grid = Build_Grid(A.Size1 *  HALF_WARP_SIZE, BLOCK_SIZE);
		GPU_MatrixVectorMultiply_CSR_Vectorized_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, X.GPU_Values, Y.GPU_Values);
		if (!GPUSparse::CUDA_Success(hipGetLastError()))
			return false;
	}

	else

	{
		dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
		GPU_MatrixVectorMultiply_CSR_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, X.GPU_Values, Y.GPU_Values);
		if (!GPUSparse::CUDA_Success(hipGetLastError()))
			return false;
	}

#ifdef USE_TEXTURE_CACHING

	// Unbind the texture memory
	Unbind_X();

#endif

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_MatrixGetDiagonals
// Extract the diagonal elements of a matrix into a vector on CPU

bool CPU_MatrixGetDiagonals(GPUCSRMatrix &A, GPUVector &X)
{
	// Primary checks
	if (A.Size1 != A.Size2 || A.Size2 != X.Size)
		return false;

	for (size_t i = 0; i < A.Size1; i++)
	{
		X.CPU_Values[i] = static_cast <double> (0);

		for (size_t j = A.CPU_RowIndices[i]; j < A.CPU_RowIndices[i + 1]; j++)
			if (A.CPU_Columns[j] == i)
				X.CPU_Values[i] = A.CPU_Values[j];
	}

	return true;
}

//
// GPU_MatrixGetDiagonals
// Extract the diagonal elements of a matrix into a vector on GPU

bool GPU_MatrixGetDiagonals(GPUCSRMatrix &A, GPUVector &X)
{
	// Primary checks
	if (A.Size1 != A.Size2 || A.Size2 != X.Size || !A.Allocated || !X.Allocated)
		return false;

	dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	GPU_MatrixGetDiagonals_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, X.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_MatrixMatrixDiagonalMultiply
// Multiply a digonal matrix specified with a vector with a matrix on CPU

bool CPU_MatrixMatrixDiagonalMultiply(GPUVector &X, GPUCSRMatrix &A)
{
	// Primary checks
	if (X.Size != A.Size1)
		return false;

	for (size_t i = 0; i < X.Size; i++)
	{
		double t = X.CPU_Values[i];

		for (size_t j = A.CPU_RowIndices[i]; j < A.CPU_RowIndices[i + 1]; j++)
				A.CPU_Values[j] *= t;
	}

	return true;
}

//
// GPU_MatrixMatrixDiagonalMultiply
// Multiply a digonal matrix specified with a vector with a matrix on GPU

bool GPU_MatrixMatrixDiagonalMultiply(GPUVector &X, GPUCSRMatrix &A)
{
	// Primary checks
	if (X.Size != A.Size1 || !X.Allocated || !A.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);
	GPU_MatrixMatrixDiagonalMultiply_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, X.GPU_Values, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_VectorPrepareDiagonalPreconditionerValues
// Prepare diagonal values of the matrix for Diagonal Preconditioner on CPU

bool CPU_VectorPrepareDiagonalPreconditionerValues(GPUVector &X)
{
	for (size_t i = 0; i < X.Size; i++)
		if (X.CPU_Values[i] == 0.00)
			X.CPU_Values[i] = 1.00;
		else
			X.CPU_Values[i] = 1.00 / X.CPU_Values[i];
//			X.CPU_Values[i] = 1.00 / sqrt(abs(X.CPU_Values[i]));

	return true;
}

//
// GPU_VectorPrepareDiagonalPreconditionerValues
// Prepare diagonal values of the matrix for Diagonal Preconditioner on GPU

bool GPU_VectorPrepareDiagonalPreconditionerValues(GPUVector &X)
{
	// Primary check
	if (!X.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);
	GPU_VectorPrepareDiagonalPreconditionerValues_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, X.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// GPU_PrepareSPAIPreconditioner
// Prepare SPAI preconditioner on GPU

bool GPU_PrepareSPAIPreconditioner(GPUCSRMatrix &A, GPUCSRMatrix &M)
{
	// Primary checks
	if (A.Size1 != M.Size1 || A.Size2 != M.Size2 || A.NNZ != M.NNZ || !A.Allocated || !M.Allocated)
		return false;

	dim3 Grid = Build_Grid(A.Size1, BLOCK_SIZE);
	GPU_SPAIPreconditioner_CSR_Kernel <<<Grid, BLOCK_SIZE>>> (A.Size1, A.GPU_Columns, A.GPU_RowIndices, A.GPU_Values, M.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_VectorVectorMultiply
// Vector-Vector multiply on CPU

bool CPU_VectorVectorMultiply(GPUVector &X, GPUVector &Y, double &Result)
{
	// Primary check
	if (X.Size != Y.Size)
		return false;

	Result = static_cast <double> (0);

	for (size_t i = 0; i < X.Size; i++)
		Result += X.CPU_Values[i] * Y.CPU_Values[i];

	return true;
}

//
// GPU_VectorVectorMultiply
// Vector-Vector multiply on GPU

bool GPU_VectorVectorMultiply(GPUVector &X, GPUVector &Y, double &Result)
{
	// Primary check
	if (X.Size != Y.Size || !X.Allocated || !Y.Allocated)
		return false;

	Result = hipblasDdot(X.Size, X.GPU_Values, 1, Y.GPU_Values, 1);

	return CUBLAS_Success(cublasGetError());
}

//
// CPU_VectorVectorMultiplyElementWise
// Vector-Vector element-wise multiply on CPU

bool CPU_VectorVectorMultiplyElementWise(GPUVector &X, GPUVector &Y,  GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size)
		return false;

	for (size_t i = 0; i < X.Size; i++)
		Z.CPU_Values[i] = X.CPU_Values[i] * Y.CPU_Values[i];

	return true;
}

//
// GPU_VectorVectorMultiplyElementWise
// Vector-Vector element-wise multiply on GPU

bool GPU_VectorVectorMultiplyElementWise(GPUVector &X, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size || !X.Allocated || !Y.Allocated || !Z.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	GPU_VectorVectorMultiplyElementWise_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, X.GPU_Values, Y.GPU_Values, Z.GPU_Values);
	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

//
// CPU_VectorNorm2
// Vector norm 2 on CPU

bool CPU_VectorNorm2(GPUVector &X, double &Result)
{
	Result = static_cast <double> (0);

	for (size_t i = 0; i < X.Size; i++)
		Result += X.CPU_Values[i] * X.CPU_Values[i];

	Result = sqrt(Result);

	return true;
}

//
// GPU_VectorNorm2
// Vector norm 2 on GPU

bool GPU_VectorNorm2(GPUVector &X, double &Result)
{
	// Primary check
	if (!X.Allocated)
		return false;

	Result = hipblasDnrm2(X.Size, X.GPU_Values, 1);

	return CUBLAS_Success(cublasGetError());
}

//
// CPU_VectorScaleAndAdd
// Vector scale-and-add on CPU

// Variant 1: Z = A * X + B * Y

bool CPU_VectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size)
		return false;

	for (size_t i = 0; i < X.Size; i++)
		Z.CPU_Values[i] = A * X.CPU_Values[i] + B * Y.CPU_Values[i];

	return true;
}

// Variant 2: Y = A * X + B * Y

bool CPU_VectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y)
{
	// Primary check
	if (X.Size != Y.Size)
		return false;

	for (size_t i = 0; i < X.Size; i++)
		Y.CPU_Values[i] = A * X.CPU_Values[i] + B * Y.CPU_Values[i];

	return true;
}

//
// GPU_VectorScaleAndAdd
// Vector scale-and-add on GPU

// Variant 1: Z = A * X + B * Y

bool GPU_VectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y, GPUVector &Z)
{
	// Primary check
	if (X.Size != Y.Size || Y.Size != Z.Size || !X.Allocated || !Y.Allocated || !Z.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	if (A == 1.00)
	{
		if (B == 1.00)
			GPU_VectorScaleAndAdd_1_A_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_1_B_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else
			GPU_VectorScaleAndAdd_1_E_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}

	else if (A == -1.00)
	{
		if (B == 1.00)
			GPU_VectorScaleAndAdd_1_C_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_1_D_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else
			GPU_VectorScaleAndAdd_1_F_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}

	else
	{
		if (B == 1.00)
			GPU_VectorScaleAndAdd_1_G_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_1_H_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);

		else
			GPU_VectorScaleAndAdd_1_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values, Z.GPU_Values);
	}

	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

// Variant 2: Y = A * X + B * Y

bool GPU_VectorScaleAndAdd(double A, GPUVector &X, double B, GPUVector &Y)
{
	// Primary check
	if (X.Size != Y.Size || !X.Allocated || !Y.Allocated)
		return false;

	dim3 Grid = Build_Grid(X.Size, BLOCK_SIZE);

	if (A == 1.00)
	{
		if (B == 1.00)
			GPU_VectorScaleAndAdd_2_A_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_2_B_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else
			GPU_VectorScaleAndAdd_2_E_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);
	}

	else if (A == -1.00)
	{
		if (B == 1.00)
			GPU_VectorScaleAndAdd_2_C_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_2_D_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else
			GPU_VectorScaleAndAdd_2_F_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);
	}

	else
	{
		if (B == 1.00)
			GPU_VectorScaleAndAdd_2_G_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else if (B == -1.00)
			GPU_VectorScaleAndAdd_2_H_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);

		else
			GPU_VectorScaleAndAdd_2_Kernel <<<Grid, BLOCK_SIZE>>> (X.Size, A, X.GPU_Values, B, Y.GPU_Values);
	}

	if (!GPUSparse::CUDA_Success(hipGetLastError()))
		return false;

	return GPUSparse::CUDA_Success(hipDeviceSynchronize());
}

}

}

//
// Compilation command
// make
