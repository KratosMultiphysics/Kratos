#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c z

       @author Jakub Kurzak
       @author Stan Tomov
       @author Mark Gates
       @author Azzam Haidar
       @author Ahmad Abdelfattah

*/

//#include "common.h"
#include <stdio.h>
#include <hipblas.h>
#include "pastix/api.h"
#include "pastix/datatypes.h"

#define PastixComplex64_PRECISION

#include "gemm_template_kernel_vbatched.cuh"

#include "gemm_config/zgemm_param_nn.h"
#include "gemm_config/zgemm_param_nt.h"
#include "gemm_config/zgemm_param_tn.h"
#include "gemm_config/zgemm_param_tt.h"

#define version(s,v) s ## _V_ ## v
/**
    Purpose
    -------
    ZGEMM performs one of the matrix-matrix operations

        C = alpha*op( A )*op( B ) + beta*C,

    where op( X ) is one of

        op( X ) = X   or   op( X ) = X**T   or   op( X ) = X**H,

    alpha and beta are scalars, and A, B and C are matrices, with
    op( A ) an m by k matrix, op( B ) a k by n matrix and C an m by n matrix.

    Parameters
    ----------
    @param[in]
    transA  CHARACTER*1.
            On entry, transA specifies the form of op( A ) to be used in
            the matrix multiplication as follows:
      -     = 'N':  op( A ) = A.
      -     = 'T':  op( A ) = A**T.
      -     = 'C':  op( A ) = A**H.

    @param[in]
    transB  CHARACTER*1.
            On entry, transB specifies the form of op( B ) to be used in
            the matrix multiplication as follows:
      -     = 'N':  op( B ) = B.
      -     = 'T':  op( B ) = B**T.
      -     = 'C':  op( B ) = B**H.

    @param[in]
    m       INTEGER.
            On entry,  M  specifies  the number  of rows  of the  matrix
            op( dA )  and of the  matrix dC.  M  must  be at least  zero.

    @param[in]
    n       INTEGER.
            On entry,  N  specifies the number  of columns of the matrix
            op( dB ) and the number of columns of the matrix dC. N must be
            at least zero.

    @param[in]
    k       INTEGER.
            On entry,  K  specifies  the number of columns of the matrix
            op( dA ) and the number of rows of the matrix op( dB ). K must
            be at least  zero.

    @param[in]
    alpha   COMPLEX_16
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX_16 array of DIMENSION ( LDA, ka ), where ka is
            k  when  transA = PastixNoTrans,  and is  m  otherwise.
            Before entry with  transA = PastixNoTrans,  the leading  m by k
            part of the array dA must contain the matrix dA, otherwise
            the leading  k by m  part of the array dA must contain  the
            matrix dA.

    @param[in]
    ldda    INTEGER.
            On entry, LDA specifies the first dimension of A as declared
            in the calling (sub) program. When  transA = PastixNoTrans then
            LDA must be at least  max( 1, m ), otherwise  LDA must be at
            least  max( 1, k ).

    @param[in]
    dB      COMPLEX_16 array of DIMENSION ( LDB, kb ), where kb is
            n  when  transB = PastixNoTrans,  and is  k  otherwise.
            Before entry with  transB = PastixNoTrans,  the leading  k by n
            part of the array dB must contain the matrix dB, otherwise
            the leading  n by k  part of the array dB must contain  the
            matrix dB.

    @param[in]
    lddb    INTEGER.
            On entry, LDB specifies the first dimension of dB as declared
            in the calling (sub) program. When  transB = PastixNoTrans then
            LDB must be at least  max( 1, k ), otherwise  LDB must be at
            least  max( 1, n ).

    @param[in]
    beta    COMPLEX_16.
            On entry,  BETA  specifies the scalar  beta.  When  BETA  is
            supplied as zero then dC need not be set on input.

    @param[inout]
    dC      COMPLEX_16 array of DIMENSION ( LDC, n ).
            Before entry, the leading  m by n  part of the array  dC must
            contain the matrix  dC,  except when  beta  is zero, in which
            case dC need not be set on entry.
            On exit, the array  dC  is overwritten by the  m by n  matrix
            ( alpha*op( dA )*op( dB ) + beta*dC ).

    @param[in]
    lddc    INTEGER.
            On entry, LDC specifies the first dimension of dC as declared
            in  the  calling  (sub)  program.   LDC  must  be  at  least
            max( 1, m ).

    @ingroup pastix_zblas3
    ********************************************************************/
#ifdef __cplusplus
extern "C"
#endif
void
pastix_zgemm_vbatched(
    pastix_trans_t transA, pastix_trans_t transB,
    pastix_int_t* m, pastix_int_t* n, pastix_int_t* k,
    hipDoubleComplex alpha,
    hipDoubleComplex const * const * dA_array, pastix_int_t* ldda,
    hipDoubleComplex const * const * dB_array, pastix_int_t* lddb,
    hipDoubleComplex beta,
    hipDoubleComplex **dC_array, pastix_int_t* lddc,
    pastix_int_t max_m, pastix_int_t max_n, pastix_int_t max_k,
    pastix_int_t batchCount, hipStream_t stream )
{
    pastix_int_t info = 0;
    // TODO: the checker needs modification for variable size: for example, checking lda, and checking m,n,k
    // Shall we do it on CPU or on GPU?
    /*if      ( transA != PastixNoTrans && transA != PastixTrans && transA != PastixConjTrans )
        info = -1;
    else if ( transB != PastixNoTrans && transB != PastixTrans && transB != PastixConjTrans )
        info = -2;
    else if ( m < 0 )
        info = -3;
    else if ( n < 0 )
        info = -4;
    else if ( k < 0 )
        info = -5;
    else if ( transA == PastixNoTrans ? ldda < m : ldda < k )
        info = -8;
    else if ( transB == PastixNoTrans ? lddb < k : lddb < n )
        info = -10;
    else if ( lddc < m )
        info = -13;
    */
    if (info != 0) {
        //pastix_xerbla( __func__, -(info) );
        return;  //info;
    }
    /*
    pastix_int_t arch = pastix_getdevice_arch();
    if ( arch < 200  ) {
        printf("arch < 200 not supported \n"); // TODO call cublas
        return;
    }

    if ( m <= 0 || n <= 0 || k <= 0 )
        return;
    */

    pastix_int_t shape;
    if      (transA == PastixNoTrans   && transB == PastixNoTrans)   {shape = 0;} // nn
    else if (transA == PastixNoTrans   && transB == PastixTrans)     {shape = 1;} // nt
    else if (transA == PastixNoTrans   && transB == PastixConjTrans) {shape = 2;} // nc
    else if (transA == PastixTrans     && transB == PastixNoTrans)   {shape = 3;} // tn
    else if (transA == PastixTrans     && transB == PastixTrans)     {shape = 4;} // tt
    else if (transA == PastixTrans     && transB == PastixConjTrans) {shape = 5;} // tc
    else if (transA == PastixConjTrans && transB == PastixNoTrans)   {shape = 6;} // cn
    else if (transA == PastixConjTrans && transB == PastixTrans)     {shape = 7;} // ct
    else if (transA == PastixConjTrans && transB == PastixConjTrans) {shape = 8;} // cc

    //TODO: probably the texture init code should be placed here

    size_t offsetA = 0;
    size_t offsetB = 0;
    offsetA = offsetA/sizeof(hipDoubleComplex);
    offsetB = offsetB/sizeof(hipDoubleComplex);

    switch(shape)
    {
        case 0: // nn
            {
                gemm_template_vbatched_nn<hipDoubleComplex, version(NN,18), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
            }
            break;
        case 1: // nt
            {
                if(max_k <= 8)
                {
                    // version 58
                    gemm_template_vbatched_nt<hipDoubleComplex, version(NT,58), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
                }
                else
                {
                    // version 29
                    gemm_template_vbatched_nt<hipDoubleComplex, version(NT,29), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
                }
            }
            break;
        case 2: // nc
            {
                if(max_k <= 8)
                {
                    // version 58
                    gemm_template_vbatched_nt<hipDoubleComplex, version(NT,58), 0, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
                }
                else
                {
                    // version 29
                    gemm_template_vbatched_nt<hipDoubleComplex, version(NT,29), 0, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
                }
            }
            break;
        case 3: // tn
            {
                // version 72
                gemm_template_vbatched_tn<hipDoubleComplex, version(TN,72), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
            }
            break;
        case 6: // cn
            {
                // version 72
                gemm_template_vbatched_tn<hipDoubleComplex, version(TN,72), 1, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
            }
            break;
        case 4: // tt
            {
                // version 13
                gemm_template_vbatched_tt<hipDoubleComplex, version(TT,13), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
            }
            break;
        case 5: // tc
            {
                // version 13
                gemm_template_vbatched_tt<hipDoubleComplex, version(TT,13), 0, 1>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
            }
            break;
        case 7: // ct
            {
                // version 13
                gemm_template_vbatched_tt<hipDoubleComplex, version(TT,13), 1, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
            }
            break;
        case 8: // cc
            {
                // version 13
                gemm_template_vbatched_tt<hipDoubleComplex, version(TT,13), 1, 1>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, batchCount, stream, max_m, max_n);
            }
            break;
        default:; // propose something
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////
