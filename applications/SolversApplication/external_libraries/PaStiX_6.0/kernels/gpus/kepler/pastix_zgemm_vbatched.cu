#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c z

       @author Jakub Kurzak
       @author Stan Tomov
       @author Mark Gates
       @author Azzam Haidar
       @author Ahmad Abdelfattah

*/

//#include "common.h"
#include <stdio.h>
#include <assert.h>
#include <hipblas.h>
#include "pastix.h"
#include "pastix/datatypes.h"
#include "kernels/pastix_cuda.h"
//#include "kernels/pastix_zcores.h"

#define PastixComplex64_PRECISION

#include "gemm_template_kernel_vbatched.cuh"

#include "gemm_config/zgemm_param_nn.h"
#include "gemm_config/zgemm_param_nt.h"
#include "gemm_config/zgemm_param_tn.h"
#include "gemm_config/zgemm_param_tt.h"

#define version(s,v) s ## _V_ ## v
/**
    Purpose
    -------
    ZGEMM performs one of the matrix-matrix operations

        C = alpha*op( A )*op( B ) + beta*C,

    where op( X ) is one of

        op( X ) = X   or   op( X ) = X**T   or   op( X ) = X**H,

    alpha and beta are scalars, and A, B and C are matrices, with
    op( A ) an m by k matrix, op( B ) a k by n matrix and C an m by n matrix.

    Parameters
    ----------
    @param[in]
    transA  CHARACTER*1.
            On entry, transA specifies the form of op( A ) to be used in
            the matrix multiplication as follows:
      -     = 'N':  op( A ) = A.
      -     = 'T':  op( A ) = A**T.
      -     = 'C':  op( A ) = A**H.

    @param[in]
    transB  CHARACTER*1.
            On entry, transB specifies the form of op( B ) to be used in
            the matrix multiplication as follows:
      -     = 'N':  op( B ) = B.
      -     = 'T':  op( B ) = B**T.
      -     = 'C':  op( B ) = B**H.

    @param[in]
    m       INTEGER.
            On entry,  M  specifies  the number  of rows  of the  matrix
            op( dA )  and of the  matrix dC.  M  must  be at least  zero.

    @param[in]
    n       INTEGER.
            On entry,  N  specifies the number  of columns of the matrix
            op( dB ) and the number of columns of the matrix dC. N must be
            at least zero.

    @param[in]
    k       INTEGER.
            On entry,  K  specifies  the number of columns of the matrix
            op( dA ) and the number of rows of the matrix op( dB ). K must
            be at least  zero.

    @param[in]
    alpha   COMPLEX_16
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX_16 array of DIMENSION ( LDA, ka ), where ka is
            k  when  transA = PastixNoTrans,  and is  m  otherwise.
            Before entry with  transA = PastixNoTrans,  the leading  m by k
            part of the array dA must contain the matrix dA, otherwise
            the leading  k by m  part of the array dA must contain  the
            matrix dA.

    @param[in]
    ldda    INTEGER.
            On entry, LDA specifies the first dimension of A as declared
            in the calling (sub) program. When  transA = PastixNoTrans then
            LDA must be at least  max( 1, m ), otherwise  LDA must be at
            least  max( 1, k ).

    @param[in]
    dB      COMPLEX_16 array of DIMENSION ( LDB, kb ), where kb is
            n  when  transB = PastixNoTrans,  and is  k  otherwise.
            Before entry with  transB = PastixNoTrans,  the leading  k by n
            part of the array dB must contain the matrix dB, otherwise
            the leading  n by k  part of the array dB must contain  the
            matrix dB.

    @param[in]
    lddb    INTEGER.
            On entry, LDB specifies the first dimension of dB as declared
            in the calling (sub) program. When  transB = PastixNoTrans then
            LDB must be at least  max( 1, k ), otherwise  LDB must be at
            least  max( 1, n ).

    @param[in]
    beta    COMPLEX_16.
            On entry,  BETA  specifies the scalar  beta.  When  BETA  is
            supplied as zero then dC need not be set on input.

    @param[inout]
    dC      COMPLEX_16 array of DIMENSION ( LDC, n ).
            Before entry, the leading  m by n  part of the array  dC must
            contain the matrix  dC,  except when  beta  is zero, in which
            case dC need not be set on entry.
            On exit, the array  dC  is overwritten by the  m by n  matrix
            ( alpha*op( dA )*op( dB ) + beta*dC ).

    @param[in]
    lddc    INTEGER.
            On entry, LDC specifies the first dimension of dC as declared
            in  the  calling  (sub)  program.   LDC  must  be  at  least
            max( 1, m ).

    @ingroup pastix_zblas3
    ********************************************************************/
#ifdef __cplusplus
extern "C"
#endif
void
/* pastix_zgemm_vbatched_nt( */
/*     pastix_trans_t transB, */
/*     pastix_int_t m[32], pastix_int_t n, pastix_int_t k, */
/*     hipDoubleComplex alpha, */
/*     hipDoubleComplex const * dA, pastix_int_t ldda, */
/*     hipDoubleComplex const * dB, pastix_int_t lddb, */
/*     hipDoubleComplex beta, */
/*     hipDoubleComplex       * dC_array[32], pastix_int_t lddc, */
/*     pastix_int_t max_m, pastix_int_t batchCount, const pastix_int_t Acoefind[32], hipStream_t stream ) */
pastix_zgemm_vbatched_nt(
    pastix_trans_t transB,
    pastix_int_t n, pastix_int_t k,
    hipDoubleComplex alpha,
    const hipDoubleComplex * dB, pastix_int_t lddb,
    hipDoubleComplex beta,
    pastix_int_t max_m, pastix_int_t batchCount, hipStream_t stream,
    gemm_params_t params )
{
    assert( transB != PastixNoTrans );

    size_t offsetA = 0;
    size_t offsetB = 0;
    offsetA = offsetA / sizeof(hipDoubleComplex);
    offsetB = offsetB / sizeof(hipDoubleComplex);

    if (transB == PastixTrans)
    {
        if(k <= 8)
        {
            // version 58
            pastix_gemm_template_vbatched_nt<hipDoubleComplex, version(NT,58), 0, 0>
                //(m, n, k, dA, ldda, dB, lddb, dC_array, lddc, alpha, beta, batchCount, Acoefind, stream, max_m);
                (n, k, dB, lddb, alpha, beta, batchCount, stream, max_m, params);
        }
        else
        {
            // version 29
            pastix_gemm_template_vbatched_nt<hipDoubleComplex, version(NT,29), 0, 0>
                (n, k, dB, lddb, alpha, beta, batchCount, stream, max_m, params);
        }
    }
    else if (transB == PastixConjTrans)
    {
        if(k <= 8)
        {
            // version 58
            pastix_gemm_template_vbatched_nt<hipDoubleComplex, version(NT,58), 0, 1>
                (n, k, dB, lddb, alpha, beta, batchCount, stream, max_m, params);
        }
        else
        {
            // version 29
            pastix_gemm_template_vbatched_nt<hipDoubleComplex, version(NT,29), 0, 1>
                (n, k, dB, lddb, alpha, beta, batchCount, stream, max_m, params);
        }
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////
