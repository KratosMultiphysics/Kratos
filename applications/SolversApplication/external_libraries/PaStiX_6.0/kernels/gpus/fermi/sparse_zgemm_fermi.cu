#include "hip/hip_runtime.h"
/**
 * @version 6.0.1
 * @author Mathieu Faverge
 * @author Pierre Ramet
 * @author Xavier Lacoste
 * @date 2018-07-16
 * @precisions normal z -> c d s
 *
 **/
/*
  -- MAGMA (version 1.1) --
  Univ. of Tennessee, Knoxville
  Univ. of California, Berkeley
  Univ. of Colorado, Denver
  November 2011


  @precisions normal z -> z c d s

*/
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdio.h>
#include <assert.h>
#include "pastix.h"
#include "pastix/datatypes.h"
#include "kernels/pastix_cuda.h"

#define PastixComplex64_PRECISION

#include "gemm_stencil.h"

/* extern "C" void */
/* GENERATE_SM_VERSION_NAME(gemm)( char TRANSA, char TRANSB, int m , int n , int k , */
/*                                  hipDoubleComplex alpha, const hipDoubleComplex *d_A, int lda, */
/*                                                         const hipDoubleComplex *d_B, int ldb, */
/*                                  hipDoubleComplex beta,        hipDoubleComplex *d_C, int ldc, */
/*                                  int blocknbr, const int *blocktab, int fblocknbr, const int *fblocktab, */
/*                                  hipStream_t stream ) */
extern "C" void
pastix_fermi_zgemmsp( char TRANSA, char TRANSB, int m , int n , int k ,
                      hipDoubleComplex alpha, const hipDoubleComplex *d_A, int lda,
                                             const hipDoubleComplex *d_B, int ldb,
                      hipDoubleComplex beta,        hipDoubleComplex *d_C, int ldc,
                      int blocknbr, const int *blocktab, int fblocknbr, const int *fblocktab,
                      hipStream_t stream )
{
    /*  -- MAGMA (version 1.1) --
        Univ. of Tennessee, Knoxville
        Univ. of California, Berkeley
        Univ. of Colorado, Denver
        November 2011

        Purpose
        =======
        ZGEMM  performs one of the matrix-matrix operations

        C := alpha*op( A )*op( B ) + beta*C,

        where  op( X ) is one of

        op( X ) = X   or   op( X ) = X',

        alpha and beta are scalars, and A, B and C are matrices, with op( A )
        an m by k matrix,  op( B )  a  k by n matrix and  C an m by n matrix.

        Parameters
        ==========
        TRANSA - CHARACTER*1.
        On entry, TRANSA specifies the form of op( A ) to be used in
        the matrix multiplication as follows:
        TRANSA = 'N' or 'n',  op( A ) = A.
        TRANSA = 'T' or 't',  op( A ) = A'.
        TRANSA = 'C' or 'c',  op( A ) = A'.
        Unchanged on exit.

        TRANSB - CHARACTER*1.
        On entry, TRANSB specifies the form of op( B ) to be used in
        the matrix multiplication as follows:
        TRANSB = 'N' or 'n',  op( B ) = B.
        TRANSB = 'T' or 't',  op( B ) = B'.
        TRANSB = 'C' or 'c',  op( B ) = B'.
        Unchanged on exit.

        M      - INTEGER.
        On entry,  M  specifies  the number  of rows  of the  matrix
        op( d_A )  and of the  matrix d_C.  M  must  be at least  zero.
        Unchanged on exit.

        N      - INTEGER.
        On entry,  N  specifies the number  of columns of the matrix
        op( d_B ) and the number of columns of the matrix d_C. N must be
        at least zero.
        Unchanged on exit.

        K      - INTEGER.
        On entry,  K  specifies  the number of columns of the matrix
        op( d_A ) and the number of rows of the matrix op( d_B ). K must
        be at least  zero.
        Unchanged on exit.

        ALPHA  - COMPLEX_16
        On entry, ALPHA specifies the scalar alpha.
        Unchanged on exit.

        d_A    - COMPLEX_16 array of DIMENSION ( LDA, ka ), where ka is
        k  when  TRANSA = 'N' or 'n',  and is  m  otherwise.
        Before entry with  TRANSA = 'N' or 'n',  the leading  m by k
        part of the array d_A must contain the matrix d_A, otherwise
        the leading  k by m  part of the array d_A must contain  the
        matrix d_A.
        Unchanged on exit.

        LDA    - INTEGER.
        On entry, LDA specifies the first dimension of A as declared
        in the calling (sub) program. When  TRANSA = 'N' or 'n' then
        LDA must be at least  max( 1, m ), otherwise  LDA must be at
        least  max( 1, k ).
        Unchanged on exit.

        d_B    - COMPLEX_16 array of DIMENSION ( LDB, kb ), where kb is
        n  when  TRANSB = 'N' or 'n',  and is  k  otherwise.
        Before entry with  TRANSB = 'N' or 'n',  the leading  k by n
        part of the array d_B must contain the matrix d_B, otherwise
        the leading  n by k  part of the array d_B must contain  the
        matrix d_B.
        Unchanged on exit.

        LDB    - INTEGER.
        On entry, LDB specifies the first dimension of d_B as declared
        in the calling (sub) program. When  TRANSB = 'N' or 'n' then
        LDB must be at least  max( 1, k ), otherwise  LDB must be at
        least  max( 1, n ).
        Unchanged on exit.

        BETA   - COMPLEX_16.
        On entry,  BETA  specifies the scalar  beta.  When  BETA  is
        supplied as zero then d_C need not be set on input.
        Unchanged on exit.

        d_C    - COMPLEX_16 array of DIMENSION ( LDC, n ).
        Before entry, the leading  m by n  part of the array  d_C must
        contain the matrix  d_C,  except when  beta  is zero, in which
        case d_C need not be set on entry.
        On exit, the array  d_C  is overwritten by the  m by n  matrix
        ( alpha*op( d_A )*op( d_B ) + beta*d_C ).

        LDC    - INTEGER.
        On entry, LDC specifies the first dimension of d_C as declared
        in  the  calling  (sub)  program.   LDC  must  be  at  least
        max( 1, m ).
        Unchanged on exit.
        =====================================================================    */
    if (m<=0 || n<=0 || k<=0)
        return;

    size_t offsetA = 0;
    size_t offsetB = 0;
#if defined(PRECISION_z) || defined(PRECISION_c)
    int TransA = 2, TransB = 2;
#else
    int TransA = 1, TransB = 1;
#endif
    if (TRANSA == 'T' ||  TRANSA == 't')
        TransA = 1;
    else
        if (TRANSA == 'N' ||  TRANSA == 'n')
            TransA = 0;

    if (TRANSB == 'T' ||  TRANSB == 't')
        TransB = 1;
    else
        if (TRANSB == 'N' ||  TRANSB == 'n')
            TransB = 0;

    size_t sizeA = (size_t) lda * (size_t) (!TransA ? k : m);
    size_t sizeB = (size_t) ldb * (size_t) (!TransB ? n : k);

    /* TODO: Check with Jakub what is this */
    size_t CUBLAS_MAX_1DBUF_SIZE = ((1 << 27) - 512);
#if 0
    if (sizeA>=CUBLAS_MAX_1DBUF_SIZE ||
        sizeB>=CUBLAS_MAX_1DBUF_SIZE )
        {
            hipblasZgemm(TRANSA, TRANSB, m, n, k, alpha,
                        d_A, lda, d_B, ldb,
                        beta, d_C, ldc);
            return;
        }
#else
    if (sizeA>=CUBLAS_MAX_1DBUF_SIZE ||
        sizeB>=CUBLAS_MAX_1DBUF_SIZE )
        {
            fprintf(stderr, "ERROR: The matrix size is too big to use texture\n");
            return;
        }

#endif


    // Set up grids
    // Warning: works because DIM_X and DIM_Y are equals for every cases of one precision
    dim3 dimBlock(DIM_X, DIM_Y);

    offsetA = offsetA/sizeof(d_A[0]);
    offsetB = offsetB/sizeof(d_B[0]);

    // NN
    if (TransA==0 && TransB ==0){
      if((m >= BLK_M_nn) && (n >= BLK_N_nn)){
        dim3 dimGrid(m/BLK_M_nn, n/BLK_N_nn);
        GENERATE_SM_VERSION_NAME(gemm_nn)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                           (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_nn != 0) && (n >= BLK_N_nn)){
        dim3 dimGrid_bottom(1, n/BLK_N_nn);
        GENERATE_SM_VERSION_NAME(gemm_bottom_nn)<<< dimGrid_bottom, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((n%BLK_N_nn != 0) && (m >= BLK_M_nn)){
        dim3 dimGrid_right(m/BLK_M_nn,1);
        GENERATE_SM_VERSION_NAME(gemm_right_nn)<<< dimGrid_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                       (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_nn != 0) && (n%BLK_N_nn != 0)){
        dim3 dimGrid_bottom_right(1, 1);
        GENERATE_SM_VERSION_NAME(gemm_bottom_right_nn)<<< dimGrid_bottom_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
    }
    // NT
    else if (TransA==0 && TransB ==1){
      /*dim3 dimGrid(m/BLK_M_nt + (m%BLK_M_nt != 0),
        n/BLK_N_nt + (n%BLK_N_nt != 0));
        //GENERATE_SM_VERSION_NAME(gemm_nt)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
        //(int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);*/
      ///////////////////////////////////////////////
      if((m >= BLK_M_nt) && (n >= BLK_N_nt)){
        dim3 dimGrid(m/BLK_M_nt, n/BLK_N_nt);
        GENERATE_SM_VERSION_NAME(gemm_nt)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                           (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_nt != 0) && (n >= BLK_N_nt)){
        dim3 dimGrid_bottom(1, n/BLK_N_nt);
        GENERATE_SM_VERSION_NAME(gemm_bottom_nt)<<< dimGrid_bottom, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((n%BLK_N_nt != 0) && (m >= BLK_M_nt)){
        dim3 dimGrid_right(m/BLK_M_nt,1);
        GENERATE_SM_VERSION_NAME(gemm_right_nt)<<< dimGrid_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                       (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_nt != 0) && (n%BLK_N_nt != 0)){
        dim3 dimGrid_bottom_right(1, 1);
        GENERATE_SM_VERSION_NAME(gemm_bottom_right_nt)<<< dimGrid_bottom_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
    }
    // TN
    else if (TransA==1 && TransB ==0){
      if((m >= BLK_M_tn) && (n >= BLK_N_tn)){
        dim3 dimGrid(m/BLK_M_tn, n/BLK_N_tn);
        GENERATE_SM_VERSION_NAME(gemm_tn)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                           (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tn != 0) && (n >= BLK_N_tn)){
        dim3 dimGrid_bottom(1, n/BLK_N_tn);
        GENERATE_SM_VERSION_NAME(gemm_bottom_tn)<<< dimGrid_bottom, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((n%BLK_N_tn != 0) && (m >= BLK_M_tn)){
        dim3 dimGrid_right(m/BLK_M_tn,1);
        GENERATE_SM_VERSION_NAME(gemm_right_tn)<<< dimGrid_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                       (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tn != 0) && (n%BLK_N_tn != 0)){
        dim3 dimGrid_bottom_right(1, 1);
        GENERATE_SM_VERSION_NAME(gemm_bottom_right_tn)<<< dimGrid_bottom_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
    }
    // TT
    else if (TransA==1 && TransB ==1){
      if((m >= BLK_M_tt) && (n >= BLK_N_tt)){
        dim3 dimGrid(m/BLK_M_tt, n/BLK_N_tt);
        GENERATE_SM_VERSION_NAME(gemm_tt)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                           (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tt != 0) && (n >= BLK_N_tt)){
        dim3 dimGrid_bottom(1, n/BLK_N_tt);
        GENERATE_SM_VERSION_NAME(gemm_bottom_tt)<<< dimGrid_bottom, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((n%BLK_N_tt != 0) && (m >= BLK_M_tt)){
        dim3 dimGrid_right(m/BLK_M_tt,1);
        GENERATE_SM_VERSION_NAME(gemm_right_tt)<<< dimGrid_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                       (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tt != 0) && (n%BLK_N_tt != 0)){
        dim3 dimGrid_bottom_right(1, 1);
        GENERATE_SM_VERSION_NAME(gemm_bottom_right_tt)<<< dimGrid_bottom_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
    }
#if defined(PRECISION_z) || defined(PRECISION_c)
    // NC
    else if (TransA==0 && TransB ==2){
      if((m >= BLK_M_nt) && (n >= BLK_N_nt)){
        dim3 dimGrid(m/BLK_M_nt, n/BLK_N_nt);
        GENERATE_SM_VERSION_NAME(gemm_nc)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                           (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_nt != 0) && (n >= BLK_N_nt)){
        dim3 dimGrid_bottom(1, n/BLK_N_nt);
        GENERATE_SM_VERSION_NAME(gemm_bottom_nc)<<< dimGrid_bottom, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((n%BLK_N_nt != 0) && (m >= BLK_M_nt)){
        dim3 dimGrid_right(m/BLK_M_nt,1);
        GENERATE_SM_VERSION_NAME(gemm_right_nc)<<< dimGrid_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                       (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_nt != 0) && (n%BLK_N_nt != 0)){
        dim3 dimGrid_bottom_right(1, 1);
        GENERATE_SM_VERSION_NAME(gemm_bottom_right_nc)<<< dimGrid_bottom_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
    }
    // TC
    else if (TransA==1 && TransB ==2){
      if((m >= BLK_M_tt) && (n >= BLK_N_tt)){
        dim3 dimGrid(m/BLK_M_tt, n/BLK_N_tt);
        GENERATE_SM_VERSION_NAME(gemm_tc)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                           (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tt != 0) && (n >= BLK_N_tt)){
        dim3 dimGrid_bottom(1, n/BLK_N_tt);
        GENERATE_SM_VERSION_NAME(gemm_bottom_tc)<<< dimGrid_bottom, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((n%BLK_N_tt != 0) && (m >= BLK_M_tt)){
        dim3 dimGrid_right(m/BLK_M_tt,1);
        GENERATE_SM_VERSION_NAME(gemm_right_tc)<<< dimGrid_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                       (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tt != 0) && (n%BLK_N_tt != 0)){
        dim3 dimGrid_bottom_right(1, 1);
        GENERATE_SM_VERSION_NAME(gemm_bottom_right_tc)<<< dimGrid_bottom_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
    }
    // CN
    else if (TransA==2 && TransB ==0){
      if((m >= BLK_M_tn) && (n >= BLK_N_tn)){
        dim3 dimGrid(m/BLK_M_tn, n/BLK_N_tn);
        GENERATE_SM_VERSION_NAME(gemm_cn)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                           (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tn != 0) && (n >= BLK_N_tn)){
        dim3 dimGrid_bottom(1, n/BLK_N_tn);
        GENERATE_SM_VERSION_NAME(gemm_bottom_cn)<<< dimGrid_bottom, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((n%BLK_N_tn != 0) && (m >= BLK_M_tn)){
        dim3 dimGrid_right(m/BLK_M_tn,1);
        GENERATE_SM_VERSION_NAME(gemm_right_cn)<<< dimGrid_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                       (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tn != 0) && (n%BLK_N_tn != 0)){
        dim3 dimGrid_bottom_right(1, 1);
        GENERATE_SM_VERSION_NAME(gemm_bottom_right_cn)<<< dimGrid_bottom_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
    }
    // CT
    else if (TransA==2 && TransB ==1){
      if((m >= BLK_M_tt) && (n >= BLK_N_tt)){
        dim3 dimGrid(m/BLK_M_tt, n/BLK_N_tt);
        GENERATE_SM_VERSION_NAME(gemm_ct)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                           (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tt != 0) && (n >= BLK_N_tt)){
        dim3 dimGrid_bottom(1, n/BLK_N_tt);
        GENERATE_SM_VERSION_NAME(gemm_bottom_ct)<<< dimGrid_bottom, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((n%BLK_N_tt != 0) && (m >= BLK_M_tt)){
        dim3 dimGrid_right(m/BLK_M_tt,1);
        GENERATE_SM_VERSION_NAME(gemm_right_ct)<<< dimGrid_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                       (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tt != 0) && (n%BLK_N_tt != 0)){
        dim3 dimGrid_bottom_right(1, 1);
        GENERATE_SM_VERSION_NAME(gemm_bottom_right_ct)<<< dimGrid_bottom_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
    }
    // CC
    else if (TransA==2 && TransB ==2){
      if((m >= BLK_M_tt) && (n >= BLK_N_tt)){
        dim3 dimGrid(m/BLK_M_tt, n/BLK_N_tt);
        GENERATE_SM_VERSION_NAME(gemm_cc)<<< dimGrid, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                           (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tt != 0) && (n >= BLK_N_tt)){
        dim3 dimGrid_bottom(1, n/BLK_N_tt);
        GENERATE_SM_VERSION_NAME(gemm_bottom_cc)<<< dimGrid_bottom, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((n%BLK_N_tt != 0) && (m >= BLK_M_tt)){
        dim3 dimGrid_right(m/BLK_M_tt,1);
        GENERATE_SM_VERSION_NAME(gemm_right_cc)<<< dimGrid_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                       (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
      if((m%BLK_M_tt != 0) && (n%BLK_N_tt != 0)){
        dim3 dimGrid_bottom_right(1, 1);
        GENERATE_SM_VERSION_NAME(gemm_bottom_right_cc)<<< dimGrid_bottom_right, dimBlock, 0, stream >>>(m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc,
                                                                         (int)offsetA, (int)offsetB, blocknbr, blocktab, fblocknbr, fblocktab);
      }
    }
#endif
    else {
      fprintf(stderr, "ERROR: in GEMM kernel");
      assert(0);
    }
#ifdef TEXTURE_1D
    hipUnbindTexture ( tex_ref_A ) ;
    hipUnbindTexture ( tex_ref_B ) ;
#endif
}

///////////////////////////////////////////////////////////////////////////////////////////////////
