#include "hip/hip_runtime.h"
//on garde tous les tests sur M et N

///////////////////////////////////////////////////////////////////////////////////////////////////

// size of work for a thread
#define THR_M ( BLK_M / DIM_X )
#define THR_N ( BLK_N / DIM_Y )

///////////////////////////////////////////////////////////////////////////////////////////////////
#ifdef KERNEL_BOTTOM
#  define KERNEL_BOTTOM_NAME _bottom
#else
#  define KERNEL_BOTTOM_NAME
#endif

#ifdef KERNEL_RIGHT
#  define KERNEL_RIGHT_NAME _right
#else
#  define KERNEL_RIGHT_NAME
#endif

#ifdef KERNEL_LDLT
#  define KERNEL_SHORT_NAME gemdm
#else
#  define KERNEL_SHORT_NAME gemm
#endif

#define CONCAT_NAME2(a,b,c,d) a ## b ## c ## d
#define CONCAT_NAME3(a,b,c,d) CONCAT_NAME2(a,b,c,d)
#define CONCAT_NAME(a) CONCAT_NAME3(KERNEL_SHORT_NAME, KERNEL_BOTTOM_NAME, KERNEL_RIGHT_NAME, a)
#ifndef GENERATE_SM_VERSION_KERNEL_NAME
#  error GENERATE_SM_VERSION_KERNEL_NAME must be defined
#endif
#ifndef version
#  error version must be defined
#endif
#if   (version == trans_nn)
#  define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(CONCAT_NAME(_nn))
#elif (version == trans_nt)
#  define TRANS_B
#  define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(CONCAT_NAME(_nt))
#elif (version == trans_nc)
#  define TRANS_B
#  define CONJ_B
#  define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(CONCAT_NAME(_nc))
#elif (version == trans_tn)
#  define TRANS_A
#  define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(CONCAT_NAME(_tn))
#elif (version == trans_tt)
#  define TRANS_A
#  define TRANS_B
#  define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(CONCAT_NAME(_tt))
#elif (version == trans_tc)
#  define TRANS_A
#  define TRANS_B
#  define CONJ_B
#  define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(CONCAT_NAME(_tc))
#elif (version == trans_cn)
#  define TRANS_A
#  define CONJ_A
#  define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(CONCAT_NAME(_cn))
#elif (version == trans_ct)
#  define TRANS_A
#  define CONJ_A
#  define TRANS_B
#  define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(CONCAT_NAME(_ct))
#elif (version == trans_cc)
#  define TRANS_A
#  define CONJ_A
#  define TRANS_B
#  define CONJ_B
#  define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(CONCAT_NAME(_cc))
#endif
#ifndef kernel_name
#  error "kernel_name must be defined"
#endif
///////////////////////////////////////////////////////////////////////////////////////////////////
extern "C" __global__
void     kernel_name (int M, int N, int K,
                      FloatingPoint_t alpha,
                      const FloatingPoint_t *A, int LDA,
#ifdef KERNEL_LDLT
                      const FloatingPoint_t *D, int LDD,
#endif
                      const FloatingPoint_t *B, int LDB,
                      FloatingPoint_t beta,
                      FloatingPoint_t       *C, int LDC,
                      int offsetA,
#ifdef KERNEL_LDLT
                      int offsetD,
#endif
                      int offsetB,
                      int blocknbr, const int *blocktab,
                      int fblocknbr, const int *fblocktab)
{
    int offset[THR_M+1];

    int idx = threadIdx.x;  // thread's m dimension
    int idy = threadIdx.y;  // thread's n dimension

    int idt = DIM_X * idy + idx;    // thread's global number

    int idxA = idt % DIM_XA;    // idx within A
    int idyA = idt / DIM_XA;    // idy within A

    int idxB = idt % DIM_XB;    // idx within B
    int idyB = idt / DIM_XB;    // idy within B
#ifdef KERNEL_BOTTOM
    int blx = M/BLK_M;      // block's m dimension
#else
    int blx = blockIdx.x;   // block's m dimension
#endif
#ifdef KERNEL_RIGHT
    int bly = N/BLK_N;      // block's n dimension
#else
    int bly = blockIdx.y;   // block's n dimension
#endif

    __shared__ FloatingPoint_t sA[BLK_K][BLK_M+1];      // +1 only required if A is transposed
#ifdef KERNEL_LDLT
    __shared__ FloatingPoint_t sD[BLK_K];
#endif
    __shared__ FloatingPoint_t sB[BLK_N][BLK_K+1];      // +1 always required

    // Registers for the innermost loop
    FloatingPoint_t rC[THR_N][THR_M];
    FloatingPoint_t rA[THR_M];
#ifdef KERNEL_LDLT
    FloatingPoint_t rD;
#endif
    FloatingPoint_t rB[THR_N];

#ifdef TRANS_A
    const FloatingPoint_t *offs_dA = A + blx*BLK_M*LDA + idyA*LDA+idxA;
#else
    const FloatingPoint_t *offs_dA = A + blx*BLK_M     + idyA*LDA+idxA;
#endif
#ifdef KERNEL_LDLT
    const FloatingPoint_t *offs_dD = D + idyA*LDD + idyA;
#endif
#ifdef TRANS_B
    const FloatingPoint_t *offs_dB = B + bly*BLK_N     + idyB*LDB+idxB;
#else
    const FloatingPoint_t *offs_dB = B + bly*BLK_N*LDB + idyB*LDB+idxB;
#endif

    int m, n, k, kk;
    int coordm, coordn;
#ifdef KERNEL_LDLT
    int coordd;
#endif
#if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
    int kk_aux=-1*BLK_K;
#endif

    // Zero C
#pragma unroll
    for (n = 0; n < THR_N; n++)
#pragma unroll
        for (m = 0; m < THR_M; m++)
            rC[n][m] = make_FloatingPoint(0.0, 0.0);

    for (kk = 0; kk < K-BLK_K; kk += BLK_K)
    {
#if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
        kk_aux=kk;
#endif
        // Load A dev->shmem
#ifdef TRANS_A
#pragma unroll
        for (n = 0; n < BLK_M; n += DIM_YA){
#pragma unroll
            for (m = 0; m < BLK_K; m += DIM_XA){
                // TODO : implement this correctly, not require in PaStiX yet
#  if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
                coordn = n + blx*BLK_M + idyA;
                if(coordn < M)
#  endif
                    sA[m+idxA][n+idyA] = fetch(A, m, n);
#  if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
                else
                    sA[m+idxA][n+idyA] = make_FloatingPoint(0.0,0.0);
#  endif
            }
        }
#  ifdef KERNEL_LDLT
        // Load D dev->shmem
#pragma unroll
        for (n = 0; n < BLK_K; n += DIM_XA)
        {
            sD[n+idyA] = fetch(D, n, n);

        }
#  endif /* KERNEL_LDLT */
#else /* TRANS_A */
        /* Not TRANS_A : It's the normal case in PaStiX */
#pragma unroll
        for (n = 0; n < BLK_K; n += DIM_YA){
#pragma unroll
            for (m = 0; m < BLK_M; m += DIM_XA){
#  ifdef KERNEL_BOTTOM
                coordm = m + blx*BLK_M + idxA;
                //coordn = n + idyA + kk;
                if(coordm < M/* && coordn < K*/)
#  endif
                    sA[n+idyA][m+idxA] = fetch(A, m, n);
#  ifdef KERNEL_BOTTOM
                else
                    sA[n+idyA][m+idxA] = make_FloatingPoint(0.0,0.0);
#  endif
            }
        }
#  ifdef KERNEL_LDLT
        // Load D dev->shmem
#pragma unroll
        for (n = 0; n < BLK_K; n += DIM_YA)
        {
            sD[n+idyA] = fetch(D, n, n);
        }
#  endif
#endif /* TRANS_A */

        // Load B dev->shmem
#ifdef TRANS_B
        /* TRANS_B : It's the normal case in PaStiX */

#pragma unroll
        for (n = 0; n < BLK_K; n += DIM_YB){
#pragma unroll
            for (m = 0; m < BLK_N; m += DIM_XB){
#  ifdef KERNEL_RIGHT
                coordm = m + bly*BLK_N + idxB;
                if(coordm < N)
#  endif
                    sB[m+idxB][n+idyB] = fetch(B, m, n);
#  ifdef KERNEL_RIGHT
                else
                    sB[m+idxB][n+idyB] = make_FloatingPoint(0.0,0.0);
#  endif
            }
        }
#else /* TRANS_B */
        /* TODO : Check, not TRANS_B : It's NOT the case in PaStiX */
#pragma unroll
        for (n = 0; n < BLK_N; n += DIM_YB){
#pragma unroll
            for (m = 0; m < BLK_K; m += DIM_XB){
#  ifdef KERNEL_RIGHT
                coordn = n + bly*BLK_N + idyB;
                if(coordn < N)
#  endif
                    sB[n+idyB][m+idxB] = fetch(B, m, n);
#  ifdef KERNEL_RIGHT
                else
                    sB[n+idyB][m+idxB] = make_FloatingPoint(0.0,0.0);
#  endif
            }
        }
#endif

        __syncthreads();

        // Multiply
#pragma unroll
        for (k = 0; k < BLK_K; k++)
        {
#ifdef KERNEL_LDLT
            rD = sD[k];
#endif
            // Load A shmem->regs
#pragma unroll
            for (m = 0; m < THR_M; m++)
            {
#ifdef KERNEL_LDLT
                rA[m] = mul(sA[k][m*DIM_X+idx],rD);
#else
                rA[m] = sA[k][m*DIM_X+idx];
#endif
            }
            // Load B shmem->regs
#pragma unroll
            for (n = 0; n < THR_N; n++)
                rB[n] = sB[n*DIM_Y+idy][k];

            // Compute
#pragma unroll
            for (n = 0; n < THR_N; n++)
#pragma unroll
                for (m = 0; m < THR_M; m++) {
#ifdef CONJ_A
#ifdef CONJ_B
                    fma(conj(rA[m]), conj(rB[n]), rC[n][m]);
#else
                    fma(conj(rA[m]), rB[n], rC[n][m]);
#endif
#else
#ifdef CONJ_B
                    fma(rA[m], conj(rB[n]), rC[n][m]);
#else
                    fma(rA[m], rB[n], rC[n][m]);
#endif
#endif
                }
        }
        __syncthreads();


        //maj offset
#ifdef TRANS_A
        offs_dA += BLK_K;
#else
        offs_dA += BLK_K*LDA;
#endif
#ifdef KERNEL_LDLT
        offs_dD += BLK_K*LDD + BLK_K;
#endif
#ifdef TRANS_B
        offs_dB += BLK_K*LDB;
#else
        offs_dB += BLK_K;
#endif

        __syncthreads();

    }
    /////////////////////////////////////////////////////////////////////



#  if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
    kk_aux+=BLK_K;
#  endif
    // Load A dev->shmem
#ifdef TRANS_A
#pragma unroll
    for (n = 0; n < BLK_M; n += DIM_YA){
#pragma unroll
        for (m = 0; m < BLK_K; m += DIM_XA){
#  ifdef KERNEL_BOTTOM
            coordn = n + blx*BLK_M + idyA;
#  endif
#  if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
            coordm = m + idxA + kk_aux;
#  else
            coordm = m + idxA + kk;
#  endif
            if(coordm < K
#  if (defined KERNEL_BOTTOM)
               && coordn < M
#  endif
               )
                sA[m+idxA][n+idyA] = fetch(A, m, n);
            else
                sA[m+idxA][n+idyA] = make_FloatingPoint(0.0,0.0);
        }
    }
#  ifdef KERNEL_LDLT
    // Load D dev->shmem
#pragma unroll
    for (n = 0; n < BLK_K; n += DIM_XA)
    {
        coordd = idyA + kk + n;
        if(coordd<K)
            sD[n+idyA] = fetch(D, n, n);
        else
            sD[n+idyA] = make_FloatingPoint(1.0,0.0);
    }
#endif
#else /* not TRANS_A */
#pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YA){
#pragma unroll
        for (m = 0; m < BLK_M; m += DIM_XA){
#  ifdef KERNEL_BOTTOM
            coordm = m + blx*BLK_M + idxA;
#  endif
#  if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
            coordn = n + idyA + kk_aux;
#  else
            coordn = n + idyA + kk;
#  endif
            if(
#  if (defined KERNEL_BOTTOM)
                coordm < M &&
#  endif
                coordn < K)
                sA[n+idyA][m+idxA] = fetch(A, m, n);
            else
                sA[n+idyA][m+idxA] = make_FloatingPoint(0.0,0.0);
        }
    }
#ifdef KERNEL_LDLT
    // Load D dev->shmem
#pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YA)
    {
        coordd = idyA + kk + n;
        if(coordd<K)
            sD[n+idyA] = fetch(D, n, n);
        else
            sD[n+idyA] = make_FloatingPoint(1.0,0.0);
    }
#endif
#endif

    // Load B dev->shmem
#ifdef TRANS_B
#pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YB){
#pragma unroll
        for (m = 0; m < BLK_N; m += DIM_XB){
#  ifdef KERNEL_RIGHT
            coordm = m + bly*BLK_N + idxB;
#  endif
#  if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
            coordn = n + idyB + kk_aux;
#  else
            coordn = n + idyB + kk;
#  endif
            if(
#  ifdef KERNEL_RIGHT
                coordm < N &&
#  endif
                coordn < K)
                sB[m+idxB][n+idyB] = fetch(B, m, n);
            else
                sB[m+idxB][n+idyB] = make_FloatingPoint(0.0,0.0);
        }
    }
#else
#pragma unroll
    for (n = 0; n < BLK_N; n += DIM_YB){
#pragma unroll
        for (m = 0; m < BLK_K; m += DIM_XB){
#  if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
            coordm = m + idxB + kk_aux;
#  else
            coordm = m + idxB + kk;
#  endif
#  ifdef KERNEL_RIGHT
            coordn = n + bly*BLK_N + idyB;
#  endif
            if(coordm < K
#  ifdef KERNEL_RIGHT
               && coordn < N
#  endif
               )
                sB[n+idyB][m+idxB] = fetch(B, m, n);
            else
                sB[n+idyB][m+idxB] = make_FloatingPoint(0.0,0.0);
        }
    }
#endif

    __syncthreads();

    // Multiply
#pragma unroll
    for (k = 0; k < BLK_K; k++)
    {
#ifdef KERNEL_LDLT
        rD = sD[k];
#endif
        // Load A shmem->regs
#pragma unroll
        for (m = 0; m < THR_M; m++) {
#ifdef KERNEL_LDLT
            rA[m] = mul(sA[k][m*DIM_X+idx],rD);
#else
            rA[m] = sA[k][m*DIM_X+idx];
#endif
        }
        // Load B shmem->regs
#pragma unroll
        for (n = 0; n < THR_N; n++)
            rB[n] = sB[n*DIM_Y+idy][k];

        // Compute
#pragma unroll
        for (n = 0; n < THR_N; n++)
#pragma unroll
            for (m = 0; m < THR_M; m++) {
#ifdef CONJ_A
#ifdef CONJ_B
                fma(conj(rA[m]), conj(rB[n]), rC[n][m]);
#else
                fma(conj(rA[m]), rB[n], rC[n][m]);
#endif
#else
#ifdef CONJ_B
                fma(rA[m], conj(rB[n]), rC[n][m]);
#else
                fma(rA[m], rB[n], rC[n][m]);
#endif
#endif
            }
    }

    __syncthreads();
#if (defined KERNEL_BOTTOM || defined KERNEL_RIGHT)
    //maj offset
#  ifdef TRANS_A
    offs_dA += BLK_K;
#  else
    offs_dA += BLK_K*LDA;
#  endif
    // WARNING: THIS WASN'T HERE BEFORE GENERIC STENCIL
#  ifdef KERNEL_LDLT
    offs_dD += BLK_K*LDD + BLK_K;
#  endif
#  ifdef TRANS_B
    offs_dB += BLK_K*LDB;
#  else
    offs_dB += BLK_K;
#  endif
#endif
    __syncthreads();
    ////////////////////////////////////////////////////////////////////////

    {
#define FROWNUM(tab, b) tab[2*b]
#define LROWNUM(tab, b) tab[2*b+1]
#define BLOCKSIZE(tab, b) LROWNUM(tab, b) - FROWNUM(tab, b) + 1
        int blocknum = 0, fblocknum = 0;
        size_t totalblocksize = 0;
        size_t blocksize = BLOCKSIZE(blocktab, blocknum);
        int    rownum;

        offset[0] = 0;
        for (m = 0; m < THR_M; m++) {
            int coord_dCm = blx*BLK_M + m*DIM_X+idx;

            if (coord_dCm < M) {

                /*
                 * We should keep blocknum < blocknbr
                 */
                while( totalblocksize + blocksize < coord_dCm + 1)
                {
                    totalblocksize += blocksize;
                    blocknum++;
                    blocksize = BLOCKSIZE(blocktab, blocknum);
                }

                /* Global row index */
                rownum = coord_dCm - totalblocksize + FROWNUM(blocktab, blocknum);

                while (LROWNUM(fblocktab, fblocknum) < rownum) {
                    offset[m] += BLOCKSIZE(fblocktab, fblocknum);
                    fblocknum++;
                }
                offset[m+1] = offset[m];
                offset[m] += rownum - FROWNUM(fblocktab, fblocknum);
            }
        }
        __syncthreads();
#undef FROWNUM
#undef LROWNUM
    }



    // Store C regs->dev
#pragma unroll
    for (n = 0; n < THR_N; n++) {
        int coord_dCn = bly*BLK_N + n*DIM_Y+idy;
#pragma unroll
        for (m = 0; m < THR_M; m++) {
            int coord_dCm = blx*BLK_M + m*DIM_X+idx;
            if (coord_dCm < M && coord_dCn < N) {
                int offsC = coord_dCn*LDC + offset[m]; /*coord_dCm;*/

                FloatingPoint_t &regC = rC[n][m];
                FloatingPoint_t &memC = C[offsC];

                memC = add(mul(alpha, regC), mul(beta, memC));
            }
        }
    }
    (void)coordm; (void)coordn;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#undef TRANS_A
#undef TRANS_B
#undef CONJ_A
#undef CONJ_B
#undef KERNEL_BOTTOM_NAME
#undef KERNEL_RIGHT_NAME
#undef KERNEL_SHORT_NAME
#undef CONCAT_NAME
#undef CONCAT_NAME2
#undef CONCAT_NAME3
#undef THR_M
#undef THR_N

#undef kernel_name
